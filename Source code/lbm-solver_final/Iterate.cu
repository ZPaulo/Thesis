#include "hip/hip_runtime.h"
#include <stdio.h>                      // printf();
#include <math.h>                       // need to compile with -lm
#include <stdlib.h>                     // for calloc();
#include <stdbool.h>                    // Include for bool type variables!
#include <string.h>                     // String operations
#include <time.h>                       // time functions
#include <errno.h>
#include "GpuFunctions.h"       // GPU kernels
#include "ShellFunctions.h"     // For convenience
#include "FilesReading.h"       // For reading files
#include "FilesWriting.h"       // For writing files e.g. tecplot
#include "CellFunctions.h"      // For cell modifications
#include "ComputeResiduals.h"   // residuals
#include "LogWriter.h"
#include "Iterate.h"
#include "ArrayUtils.h"
#include "Check.h"
#include "Multiphase.h"
#include "GpuSum.h"

#define CUDA 1

int Iterate2D(InputFilenames *inFn, Arguments *args) {
	// Time measurement: declaration, begin
	clock_t tStart = clock();

	FILE* logFile;               // file for log
	char autosaveFilename[768];  // autosave filename
	char outputFilename[768];    // initial data will be written to this file
	char finalFilename[768];     // final data will be written to this file
	char logFilename[768];       // path of the .log file
	char residualsFilename[768]; // path of the residuals file
	char timeFilename[768];      // path of time measurement file

	logFilename[0] = '\0';
	residualsFilename[0] = '\0';
	timeFilename[0] = '\0';

	if (strlen(inFn->result)) {
		strcat(logFilename, inFn->result);
		strcat(residualsFilename, inFn->result);
		strcat(timeFilename, inFn->result);
	}
	strcat(logFilename, "lbmsolver.log");
	strcat(residualsFilename, "residuals.dat");
	strcat(timeFilename, "runtimes.dat");

	int autosaveIt = 1; // autosave i variable, will be incremented after every autosave
	int numNodes, numConns; // This will store the number of lines of the read files
	FLOAT_TYPE delta;          // grid spacing
	int n, m;                   // number of nodes in the x and y directions
	FLOAT_TYPE maxInletCoordY; // maximum inlet coordinate in y
	FLOAT_TYPE minInletCoordY; // minimum inlet coordinate in y
	int numInletNodes;         // number of inlet nodes

	int *nodeIdX, *nodeIdY, *nodeType, *bcNodeIdX, *bcNodeIdY, *latticeId,
	*bcType, *bcBoundId,*tempi;
	FLOAT_TYPE *nodeX, *nodeY, *bcX, *bcY,*temp;

	FLOAT_TYPE taskTime[9];
	int i;
	for (i = 0; i < 9; ++i) {
		taskTime[i] = 0.0;
	}

	clock_t tInstant1, tInstant2; // Time measurement points, universal
	clock_t tIterStart, tIterEnd; // Time measurement points: main loop

	// cuda time measurement variables
	hipEvent_t start, stop;
	float cudatime;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));
	numNodes = readNodeFile(inFn->node, &nodeIdX, &nodeIdY, &tempi, &nodeX,
			&nodeY, &temp, &nodeType,args->TypeOfProblem);

	if (numNodes == 0) {
		printf("NODES NOT FOUND in file\n");
		return 2;
	}
	int *fluid_d = createGpuArrayInt(numNodes, ARRAY_COPY, 0, nodeType);
	FLOAT_TYPE *coordX_d = createGpuArrayFlt(numNodes, ARRAY_COPY, 0., nodeX);
	FLOAT_TYPE *coordY_d = createGpuArrayFlt(numNodes, ARRAY_COPY, 0., nodeY);
	numConns = readConnFile(inFn->bc, &bcNodeIdX, &bcNodeIdY, &tempi,
			&latticeId, &bcType, &bcX, &bcY, &temp, &bcBoundId,args->TypeOfProblem);

	if (numConns == 0) {
		printf("NEIGHBOURING NOT FOUND in file\n");
		return 2;
	}

	int *bcNodeIdX_d = createGpuArrayInt(numConns, ARRAY_COPY, 0, bcNodeIdX);
	int *bcNodeIdY_d = createGpuArrayInt(numConns, ARRAY_COPY, 0, bcNodeIdX); //WHATCH OUT IdX???
	int *latticeId_d = createGpuArrayInt(numConns, ARRAY_COPY, 0, latticeId);
	int *bcType_d = createGpuArrayInt(numConns, ARRAY_COPY, 0, bcType);
	int *bcBoundId_d = createGpuArrayInt(numConns, ARRAY_COPY, 0, bcBoundId);
	FLOAT_TYPE *bcX_d = createGpuArrayFlt(numConns, ARRAY_COPY, 0., bcX);
	FLOAT_TYPE *bcY_d = createGpuArrayFlt(numConns, ARRAY_COPY, 0., bcY);
	m = getLastValue(nodeIdY, numNodes);
	n = getLastValue(nodeIdX, numNodes);


	delta = getGridSpacing(nodeIdX, nodeIdY, nodeX, numNodes);
	numInletNodes = getNumInletNodes(bcType, latticeId, numConns,
			args->TypeOfProblem);
	maxInletCoordY = getMaxInletCoordY(bcType, latticeId, bcY, delta, numConns,args->TypeOfProblem);
	minInletCoordY = getMinInletCoordY(bcType, latticeId, bcY, delta, numConns,args->TypeOfProblem);
	FLOAT_TYPE *nodeZ = createHostArrayFlt(m * n, ARRAY_ZERO);
	writeInitLog(logFilename, args, delta, m, n, 1, numInletNodes,
			maxInletCoordY, minInletCoordY, 0.0, 0.0);
	logFile = fopen(logFilename, "a");

	// In case of no autosave
	sprintf(autosaveFilename, "NOWHERE!");
	initConstants2D(args, maxInletCoordY, minInletCoordY, delta, m, n);

	dim3 tpb(THREADS); 					 // THREADS/block
	dim3 bpg1((int) (m * n / THREADS) + 1);     // blocks/grid  MxN
	dim3 bpg8((int) (8 * m * n / THREADS) + 1);     // blocks/grid 8MxN
	dim3 bpg9((int) (9 * m * n / THREADS) + 1);     // blocks/grid 9MxN
	dim3 bpgBC((int) (numConns / THREADS) + 1); // blocks/grid N_BC
	// residuals
	FLOAT_TYPE *norm = createHostArrayFlt(args->iterations, ARRAY_ZERO);
	FLOAT_TYPE *dragSum = createHostArrayFlt(args->iterations, ARRAY_ZERO);
	FLOAT_TYPE *liftSum = createHostArrayFlt(args->iterations, ARRAY_ZERO);

	fprintf(logFile, "\n:::: Initializing ::::\n");
	printf("\n:::: Initializing ::::\n");
	CHECK(hipEventRecord(start, 0));
	FLOAT_TYPE *u = createHostArrayFlt(m * n, ARRAY_ZERO);
	FLOAT_TYPE *v = createHostArrayFlt(m * n, ARRAY_ZERO);
	FLOAT_TYPE *w = createHostArrayFlt(m * n, ARRAY_ZERO);
	FLOAT_TYPE *rho = createHostArrayFlt(m * n, ARRAY_ZERO);

	//Multiphase
	FLOAT_TYPE *r_rho = createHostArrayFlt(m * n, ARRAY_ZERO);
	FLOAT_TYPE *b_rho = createHostArrayFlt(m * n, ARRAY_ZERO);
	FLOAT_TYPE *st_error = createHostArrayFlt(args->iterations, ARRAY_ZERO);
	FLOAT_TYPE *color_gradient = createHostArrayFlt(m * n * 2, ARRAY_ZERO);
	FLOAT_TYPE *r_f = createHostArrayFlt(m * n * 9, ARRAY_ZERO);
	FLOAT_TYPE *b_f = createHostArrayFlt(m * n * 9, ARRAY_ZERO);
	FLOAT_TYPE *r_fColl = createHostArrayFlt(m * n * 9, ARRAY_ZERO);
	FLOAT_TYPE *b_fColl = createHostArrayFlt(m * n * 9, ARRAY_ZERO);
	FLOAT_TYPE r_omega = 1.0/(3.0 * args->r_viscosity+0.5);
	FLOAT_TYPE	b_omega = 1.0/(3.0 * args->b_viscosity+0.5);
	FLOAT_TYPE st_predicted = (2.0/9.0)*(1.0+1.0/args->gamma)/(0.5*(r_omega+b_omega))*0.5*args->r_density*(args->r_A+args->b_A);
	int *cg_directions = createHostArrayInt(n*m, ARRAY_ZERO);

#if !CUDA
	FLOAT_TYPE r_phi[9];
	FLOAT_TYPE b_phi[9];
	FLOAT_TYPE w_pert[9];
	int cx[9] = {0, 1, 0, -1, 0, 1, -1, -1, 1};
	int cy[9] = {0, 0, 1, 0, -1, 1, 1, -1, -1};
	FLOAT_TYPE weight[9] = {4.0/9.0, 1.0/9.0, 1.0/9.0, 1.0/9.0, 1.0/9.0, 1.0/36.0, 1.0/36.0, 1.0/36.0, 1.0/36.0};
	if(args->multiPhase){



		int i;
		w_pert[0] = -4.0/ 27.0;
		for(i = 1; i < 5; i++)
			w_pert[i] = 2.0 / 27.0;
		for(i = 5; i < 9; i++)
			w_pert[i] = 5.0 / 108.0;

		r_phi[0] = args->r_alpha;
		for(i = 1; i < 5; i++)
			r_phi[i] = (1.0 - args->r_alpha) / 5.0;
		for(i = 5; i < 9; i++)
			r_phi[i] = (1.0 - args->r_alpha) / 20.0;

		b_phi[0] = args->b_alpha;
		for(i = 1; i < 5; i++)
			b_phi[i] = (1.0 - args->b_alpha) / 5.0;
		for(i = 5; i < 9; i++)
			b_phi[i] = (1.0 - args->b_alpha) / 20.0;


		createBubble(nodeX, nodeY,n,m,args->bubble_radius, r_f, b_f,r_rho,b_rho, args->r_density, args->b_density, r_phi, b_phi, rho);
	}
#endif
	FLOAT_TYPE *rho_d = createGpuArrayFlt(m * n, ARRAY_FILL, args->rho);
	FLOAT_TYPE *r_rho_d = createGpuArrayFlt(m * n, ARRAY_ZERO);
	FLOAT_TYPE *b_rho_d = createGpuArrayFlt(m * n, ARRAY_ZERO);
	FLOAT_TYPE *r_f_d = createGpuArrayFlt(m * n * 9, ARRAY_ZERO);
	FLOAT_TYPE *b_f_d = createGpuArrayFlt(m * n * 9, ARRAY_ZERO);
	FLOAT_TYPE *r_fColl_d = createGpuArrayFlt(m * n * 9, ARRAY_ZERO);
	FLOAT_TYPE *b_fColl_d = createGpuArrayFlt(m * n * 9, ARRAY_ZERO);
	int *cg_dir_d = createGpuArrayInt(m * n, ARRAY_ZERO);
	FLOAT_TYPE *p_in_d = createGpuArrayFlt(n*m, ARRAY_ZERO);
	FLOAT_TYPE *p_out_d = createGpuArrayFlt(n*m, ARRAY_ZERO);
	FLOAT_TYPE p_in_mean;
	FLOAT_TYPE p_out_mean;
	FLOAT_TYPE ms = n * m;
	int *num_in_d = createGpuArrayInt(n*m, ARRAY_ZERO);
	int *num_out_d = createGpuArrayInt(n*m, ARRAY_ZERO);

	FLOAT_TYPE *u0_d, *v0_d;

	if (args->inletProfile == NO_INLET) {
		u0_d = createGpuArrayFlt(m * n, ARRAY_FILL, args->u);
		v0_d = createGpuArrayFlt(m * n, ARRAY_FILL, args->v);
	} else {
		u0_d = createGpuArrayFlt(m * n, ARRAY_ZERO);
		v0_d = createGpuArrayFlt(m * n, ARRAY_ZERO);
	}
	if (args->inletProfile == INLET) {
		gpuInitInletProfile2D<<<bpg1, tpb>>>(u0_d, v0_d, coordY_d, m * n);
	}
	FLOAT_TYPE *drag_d = createGpuArrayFlt(m * n, ARRAY_ZERO);
	FLOAT_TYPE *lift_d = createGpuArrayFlt(m * n, ARRAY_ZERO);

	FLOAT_TYPE *f_d = createGpuArrayFlt(9 * m * n, ARRAY_ZERO);
	FLOAT_TYPE *fColl_d = createGpuArrayFlt(9 * m * n, ARRAY_ZERO);

	FLOAT_TYPE *temp9a_d = createGpuArrayFlt(9 * m * n, ARRAY_ZERO);
	FLOAT_TYPE *temp9b_d = createGpuArrayFlt(9 * m * n, ARRAY_ZERO);
	FLOAT_TYPE *tempA_d = createGpuArrayFlt(m * n, ARRAY_ZERO);
	FLOAT_TYPE *tempB_d = createGpuArrayFlt(m * n, ARRAY_ZERO);

#if CUDA
	if(args->multiPhase){
		initColorGradient(cg_directions, n, m);
		CHECK(hipMemcpy(cg_dir_d, cg_directions, SIZEINT(m*n), hipMemcpyHostToDevice));
		initCGBubble<<<bpg1,tpb>>>(coordX_d,coordY_d,r_rho_d, b_rho_d, rho_d, r_f_d, b_f_d);
	}
#endif
	int *mask = createHostArrayInt(m * n, ARRAY_ZERO);
	int *bcMask = createHostArrayInt(m * n, ARRAY_ZERO);
	int *bcIdx = createHostArrayInt(m * n, ARRAY_ZERO);

	FLOAT_TYPE *u_d = createGpuArrayFlt(m * n, ARRAY_CPYD, 0, u0_d);
	FLOAT_TYPE *v_d = createGpuArrayFlt(m * n, ARRAY_CPYD, 0, v0_d);
	int *stream = createHostArrayInt(8 * m * n, ARRAY_FILL, 1);
	FLOAT_TYPE *q = createHostArrayFlt(8 * m * n, ARRAY_FILL, 0.5);

	int bcCount = initBoundaryConditions2D(bcNodeIdX, bcNodeIdY, q, bcBoundId,
			nodeType, bcX, bcY, nodeX, nodeY, latticeId, stream, bcType, bcMask,
			bcIdx, mask, delta, m, n, numConns);

	int *bcIdxCollapsed_d = createGpuArrayInt(bcCount, ARRAY_ZERO);
	int *bcMaskCollapsed_d = createGpuArrayInt(bcCount, ARRAY_ZERO);
	FLOAT_TYPE *qCollapsed_d = createGpuArrayFlt(8 * bcCount, ARRAY_ZERO);

	dim3 bpgB((int) (bcCount / THREADS) + 1); // blocks/grid
	int *bcMask_d = createGpuArrayInt(m * n, ARRAY_COPY, 0, bcMask);
	int *bcIdx_d = createGpuArrayInt(m * n, ARRAY_COPY, 0, bcIdx);

	collapseBc2D(bcIdx, bcIdxCollapsed_d, bcMask, bcMaskCollapsed_d, q,
			qCollapsed_d, mask, m, n, bcCount);

	int *stream_d = createGpuArrayInt(8 * m * n, ARRAY_COPY, 0, stream);


//	CHECK(hipMemcpy(u, u_d, SIZEFLT(m*n), hipMemcpyDeviceToHost));
//	CHECK(hipMemcpy(v, v_d, SIZEFLT(m*n), hipMemcpyDeviceToHost));
//	CHECK(hipMemcpy(rho, rho_d, SIZEFLT(m*n), hipMemcpyDeviceToHost));

	CHECK(hipEventRecord(stop, 0));
	CHECK(hipEventSynchronize(stop));
	CHECK(hipEventElapsedTime(&cudatime, start, stop));
	taskTime[T_INIT] += cudatime / 1000;
	fclose(logFile);
	writeNodeNumbers(logFilename, numNodes, numConns, bcCount);
	logFile = fopen(logFilename, "a");


	void *hostArrays[] = { nodeIdX, nodeIdY, nodeX, nodeY, nodeType, bcNodeIdX,
			bcNodeIdY, latticeId, bcType, bcX, bcY, bcBoundId, u, v, rho, mask,
			bcMask, bcIdx, stream, q, norm, dragSum, liftSum, r_rho, b_rho,
			color_gradient,r_f,b_f,r_fColl, b_fColl, st_error, cg_directions};
	void *gpuArrays[] = { coordX_d, coordY_d, fluid_d, bcNodeIdX_d, bcNodeIdY_d,
			latticeId_d, bcType_d, bcX_d, bcY_d, bcBoundId_d, u_d, v_d, rho_d,
			u0_d, v0_d, drag_d, lift_d, f_d, fColl_d, temp9a_d, temp9b_d,
			tempA_d, tempB_d, bcMask_d, bcMaskCollapsed_d, bcIdx_d,
			bcIdxCollapsed_d, stream_d,  qCollapsed_d, r_f_d, r_fColl_d, b_f_d,
			b_fColl_d, cg_dir_d, r_rho_d, b_rho_d, p_in_d, p_out_d, num_in_d, num_out_d};
	fprintf(logFile, "\n:::: Initialization done! ::::\n");

	printf("Initialization took %f seconds\n", taskTime[T_INIT]);

	// Write Initialized data
	switch (args->outputFormat) {
	case CSV:
		sprintf(finalFilename, "%sFinalData.csv", inFn->result);
		break;
	case TECPLOT:
		sprintf(finalFilename, "%sFinalData.dat", inFn->result);
		break;
	case PARAVIEW:
		sprintf(finalFilename, "%sFinalData.vti", inFn->result);
		break;
	}

	tInstant1 = clock(); // Start measuring time
	if(args->multiPhase){
		WriteResultsMultiPhase(finalFilename, nodeType, nodeX, nodeY, nodeZ, u, v, w, rho,r_rho,b_rho, nodeType,
				n, m, 1, args->outputFormat);
	}
	else
		WriteResults3D(finalFilename, nodeType, nodeX, nodeY, nodeZ, u, v, w, rho, nodeType,
				n, m, 1, args->outputFormat);
	tInstant2 = clock();
	taskTime[T_WRIT] += (FLOAT_TYPE) (tInstant2 - tInstant1) / CLOCKS_PER_SEC;
	printf("\nInitialized data was written to %s\n", outputFilename);

	////////////////// ITERATION ///////////////////////

	fprintf(logFile, "\n:::: Start Iterations ::::\n");
	printf("\n:::: Start Iterations ::::\n");

	printf("%d is the number of iterations \n", args->iterations);

	tIterStart = clock(); // Start measuring time of main loop
	int iter = 0;
	while (iter < args->iterations) {


		//		if(args->multiPhase){
		//			CHECK(hipMemcpy(r_rho_d, r_rho, SIZEFLT(m * n), hipMemcpyHostToDevice));
		//			CHECK(hipMemcpy(b_rho_d, b_rho, SIZEFLT(m * n), hipMemcpyHostToDevice));
		//			CHECK(hipMemcpy(r_f_d, r_f, SIZEFLT(m*n *9), hipMemcpyHostToDevice));
		//			CHECK(hipMemcpy(b_f_d, b_f, SIZEFLT(m*n *9), hipMemcpyHostToDevice));
		//			CHECK(hipMemcpy(r_fColl_d, r_fColl, SIZEFLT(m*n *9), hipMemcpyHostToDevice));
		//			CHECK(hipMemcpy(b_fColl_d, b_fColl, SIZEFLT(m*n *9), hipMemcpyHostToDevice));
		//			CHECK(hipMemcpy(u_d, u, SIZEFLT(m * n), hipMemcpyHostToDevice));
		//			CHECK(hipMemcpy(v_d, v, SIZEFLT(m * n), hipMemcpyHostToDevice));
		//			CHECK(hipMemcpy(rho_d, rho, SIZEFLT(m * n), hipMemcpyHostToDevice));
		//		}
		CHECK(hipDeviceSynchronize());
		CHECK(hipEventRecord(start, 0)); // Start measuring time
		switch (args->collisionModel) {

		case BGKW:
			if(args->multiPhase){
				//Collision

#if !CUDA
				mp2DColl(n, m, rho, u, v, r_f, b_f, r_rho, b_rho, r_phi, b_phi, w_pert, color_gradient,
						r_omega, b_omega, args->control_param, args->del, args->beta,
						args->g_limit, args->r_A, args->b_A, r_fColl, b_fColl, weight, cx, cy);
#else
				gpuCollBgkwGC2D<<<bpg1, tpb>>>(fluid_d, rho_d, r_rho_d, b_rho_d, u_d, v_d, r_f_d, b_f_d, r_fColl_d, b_fColl_d, cg_dir_d);
#endif
			}else{
				gpuCollBgkw2D<<<bpg1, tpb>>>(fluid_d, rho_d, u_d, v_d, f_d,
						fColl_d);
			}
			break;
		case TRT:
			gpuCollTrt<<<bpg1, tpb>>>(fluid_d, rho_d, u_d, v_d, f_d, fColl_d);
			break;

		case MRT:
			gpuCollMrt2D<<<bpg1, tpb>>>(fluid_d, rho_d, u_d, v_d, f_d, fColl_d);
			break;
		}

		CHECK(hipEventRecord(stop, 0));
		CHECK(hipEventSynchronize(stop));
		CHECK(hipEventElapsedTime(&cudatime, start, stop));
		taskTime[T_COLL] += cudatime;

		////////////// STREAMING ///////////////
		CHECK(hipDeviceSynchronize());
		CHECK(hipEventRecord(start, 0));
		if(args->multiPhase){
#if !CUDA
			streamMP(n, m, r_f, b_f, r_fColl, b_fColl);
			//			gpuStreaming2D<<<bpg1, tpb>>>(fluid_d, stream_d, r_f_d, r_fColl_d);
			//			gpuStreaming2D<<<bpg1, tpb>>>(fluid_d, stream_d, b_f_d, b_fColl_d);
#else
			gpuStreaming2DCG<<<bpg1, tpb>>>(fluid_d, stream_d, r_f_d, r_fColl_d, b_f_d, b_fColl_d);
#endif
		}
		else{
			gpuStreaming2D<<<bpg1, tpb>>>(fluid_d, stream_d, f_d, fColl_d);
		}

		CHECK(hipEventRecord(stop, 0));
		CHECK(hipEventSynchronize(stop));
		CHECK(hipEventElapsedTime(&cudatime, start, stop));
		taskTime[T_STRM] += cudatime;


		// make the host block until the device is finished with foo
		CHECK(hipDeviceSynchronize());

		// check for error
		hipError_t error = hipGetLastError();
		if (error != hipSuccess) {
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			exit(-1);
		}
		////////////// BOUNDARIES ///////////////
		CHECK(hipDeviceSynchronize());
		CHECK(hipEventRecord(start, 0));

		if(args->multiPhase){
#if !CUDA
			peridicBoundaries(n, m, r_f, b_f);
#else
			gpuBcPeriodic2D<<<bpgB, tpb>>>(bcIdxCollapsed_d, bcMaskCollapsed_d, r_f_d, b_f_d,bcCount, cg_dir_d);
#endif

		} else{
			gpuBcInlet2D<<<bpgB, tpb>>>(bcIdxCollapsed_d, bcMaskCollapsed_d, f_d,
					u0_d, v0_d, bcCount);
			gpuBcWall2D<<<bpgB, tpb>>>(bcIdxCollapsed_d, bcMaskCollapsed_d, f_d,
					fColl_d, qCollapsed_d, bcCount);
			gpuBcOutlet2D<<<bpgB, tpb>>>(bcIdxCollapsed_d, bcMaskCollapsed_d, f_d,
					u0_d, v0_d, bcCount);
		}

		//				if(args->multiPhase){
		//					CHECK(hipMemcpy(r_f, r_f_d, SIZEFLT(m*n *9), hipMemcpyDeviceToHost));
		//					CHECK(hipMemcpy(b_f, b_f_d, SIZEFLT(m*n *9), hipMemcpyDeviceToHost));
		//				}

		CHECK(hipEventRecord(stop, 0));
		CHECK(hipEventSynchronize(stop));
		CHECK(hipEventElapsedTime(&cudatime, start, stop));
		taskTime[T_BNDC] += cudatime;

		// UPDATE VELOCITY AND DENSITY
		CHECK(hipDeviceSynchronize());
		CHECK(hipEventRecord(start, 0));
		if(args->multiPhase){
#if !CUDA
			updateMacroMP(n,m,u,v,r_rho, b_rho, r_f, b_f, rho, args->control_param,args->r_alpha, args->b_alpha,
					args->bubble_radius,st_error, iter,st_predicted);
#else
			gpuUpdateMacro2DCG<<<bpg1, tpb>>>(fluid_d, rho_d, u_d, v_d, r_f_d, b_f_d, r_rho_d, b_rho_d, p_in_d, p_out_d, num_in_d, num_out_d);

			//			CHECK(hipMemcpy(r_rho, r_rho_d, SIZEFLT(m*n), hipMemcpyDeviceToHost));
			//			CHECK(hipMemcpy(b_rho, b_rho_d, SIZEFLT(m*n), hipMemcpyDeviceToHost));
			//			updateSurfaceTension(r_rho,b_rho,args->control_param, st_predicted, st_error, iter,args->r_alpha, args->b_alpha, args->bubble_radius, n ,m);
			//gpu reduction is faster than serial surface tension
			p_in_mean = gpu_sum_h(p_in_d, p_in_d, ms) / gpu_sum_int_h(num_in_d, num_in_d, ms);
			p_out_mean = gpu_sum_h(p_out_d, p_out_d, ms) / gpu_sum_int_h(num_out_d, num_out_d, ms);
			st_error[iter] = calculateSurfaceTension(p_in_mean, p_out_mean,args->r_alpha, args->b_alpha, args->bubble_radius, st_predicted);
#endif
		}
		else gpuUpdateMacro2D<<<bpg1, tpb>>>(fluid_d, rho_d, u_d, v_d, bcMask_d,
				drag_d, lift_d, coordX_d, coordY_d, f_d);

		tInstant2 = clock();
		CHECK(hipEventRecord(stop, 0));
		CHECK(hipEventSynchronize(stop));
		CHECK(hipEventElapsedTime(&cudatime, start, stop));
		taskTime[T_MACR] += cudatime;

		// COMPUTE RESIDUALS
		CHECK(hipDeviceSynchronize());
		CHECK(hipEventRecord(start, 0));
		FLOAT_TYPE r;
		if(args->multiPhase){
#if !CUDA
			CHECK(hipMemcpy(r_f_d, r_f, SIZEFLT(m*n *9), hipMemcpyHostToDevice));
			CHECK(hipMemcpy(b_f_d, b_f, SIZEFLT(m*n *9), hipMemcpyHostToDevice));
			CHECK(hipMemcpy(r_fColl_d, r_fColl, SIZEFLT(m*n *9), hipMemcpyHostToDevice));
			CHECK(hipMemcpy(b_fColl_d, b_fColl, SIZEFLT(m*n *9), hipMemcpyHostToDevice));
#endif
			r = computeResidual2D(r_f_d, r_fColl_d, temp9a_d, temp9b_d, m,n);

		}else
			r = computeResidual2D(f_d, fColl_d, temp9a_d, temp9b_d, m,n);
		if (r != r) {
			fprintf(stderr, "\nDIVERGENCE!\n");

			writeResiduals(residualsFilename, norm, dragSum, liftSum, m * n,
					iter + 1);
			hipEventDestroy(start);
			hipEventDestroy(stop);

			freeAllHost(hostArrays, sizeof(hostArrays) / sizeof(hostArrays[0]));
			freeAllGpu(gpuArrays, sizeof(gpuArrays) / sizeof(gpuArrays[0]));

			return 1; // ERROR!
		}
		norm[iter] = r;
		if (args->boundaryId > 0) {
			dragSum[iter] = computeDragLift2D(bcMask_d, drag_d, tempA_d,
					tempB_d, m, n, args->boundaryId);
			liftSum[iter] = computeDragLift2D(bcMask_d, lift_d, tempA_d,
					tempB_d, m, n, args->boundaryId);
		}

		CHECK(hipEventRecord(stop, 0));
		CHECK(hipEventSynchronize(stop));
		CHECK(hipEventElapsedTime(&cudatime, start, stop));
		taskTime[T_RESI] += cudatime;
		printf("Iterating... %d/%d (%3.1f %%)\r", iter + 1, args->iterations,
				(FLOAT_TYPE) (iter + 1) * 100
				/ (FLOAT_TYPE) (args->iterations));

		iter++; // update loop variable
		////////////// Autosave ///////////////

		if (iter == (args->autosaveEvery * autosaveIt)) {
			autosaveIt++;
			if (iter > args->autosaveAfter) {
				printf("autosave\n\n");
				//////////// COPY VARIABLES TO HOST ////////////////
				CHECK(hipMemcpy(u, u_d, SIZEFLT(m*n), hipMemcpyDeviceToHost));
				CHECK(hipMemcpy(v, v_d, SIZEFLT(m*n), hipMemcpyDeviceToHost));
				CHECK(hipMemcpy(rho, rho_d, SIZEFLT(m*n), hipMemcpyDeviceToHost));
				if(args->multiPhase){
					CHECK(hipMemcpy(r_rho, r_rho_d, SIZEFLT(m*n), hipMemcpyDeviceToHost));
					CHECK(hipMemcpy(b_rho, b_rho_d, SIZEFLT(m*n), hipMemcpyDeviceToHost));
				}

				switch (args->outputFormat) {
				case CSV:
					sprintf(finalFilename, "%sFinalData.csv", inFn->result);
					break;
				case TECPLOT:
					sprintf(finalFilename, "%sFinalData.dat", inFn->result);
					break;
				case PARAVIEW:
					sprintf(finalFilename, "%sFinalData.vti", inFn->result);
					break;
				}

				tInstant1 = clock(); // Start measuring time
				WriteResults3D(finalFilename, nodeType,nodeX, nodeY, nodeZ, u, v, w, rho,
						nodeType, n, m, 1, args->outputFormat);
				tInstant2 = clock();
				taskTime[T_WRIT] += (FLOAT_TYPE) (tInstant2 - tInstant1)
																																																																																																																										/ CLOCKS_PER_SEC;
			}
		}
	}     ////////////// END OF MAIN WHILE CYCLE! ///////////////
	tIterEnd = clock(); // End measuring time of main loop
	taskTime[T_ITER] = (FLOAT_TYPE) (tIterEnd - tIterStart) / CLOCKS_PER_SEC;

	clock_t tEnd = clock();
	taskTime[T_OALL] = (FLOAT_TYPE) (tEnd - tStart) / CLOCKS_PER_SEC; // Calculate elapsed time
	taskTime[T_COLL] /= 1000;
	taskTime[T_STRM] /= 1000;
	taskTime[T_BNDC] /= 1000;
	taskTime[T_MACR] /= 1000;
	taskTime[T_RESI] /= 1000;
	fclose(logFile);
	writeEndLog(logFilename, taskTime);
	writeTimerLog(timeFilename, taskTime);
	writeResiduals(residualsFilename, norm, dragSum, liftSum, m * n,
			args->iterations);

#if CUDA
	//WRITE VARIABLES TO HOST
	CHECK(hipMemcpy(u, u_d, SIZEFLT(m*n), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(v, v_d, SIZEFLT(m*n), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(rho, rho_d, SIZEFLT(m*n), hipMemcpyDeviceToHost));
	if(args->multiPhase){
		CHECK(hipMemcpy(r_rho, r_rho_d, SIZEFLT(m*n), hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(b_rho, b_rho_d, SIZEFLT(m*n), hipMemcpyDeviceToHost));
	}
#endif
	switch (args->outputFormat) {
	case CSV:
		sprintf(finalFilename, "%sFinalData.csv", inFn->result);
		break;
	case TECPLOT:
		sprintf(finalFilename, "%sFinalData.dat", inFn->result);
		break;
	case PARAVIEW:
		sprintf(finalFilename, "%sFinalData.vti", inFn->result);
		break;
	}
	if(args->multiPhase){
		WriteResultsMultiPhase(finalFilename, nodeType, nodeX, nodeY, nodeZ, u, v, w, rho,r_rho,b_rho, nodeType,
				n, m, 1, args->outputFormat);
		printf("Suface tension error: "FLOAT_FORMAT"\n", st_error[iter-1]);
		WriteArray("surface tension",st_error, args->iterations,1);
	}
	else
		WriteResults3D(finalFilename, nodeType, nodeX, nodeY, nodeZ, u, v, w, rho, nodeType,
				n, m, 1, args->outputFormat);

	// Write information for user
	printf("\n\nLog was written to %s\n", logFilename);
	printf("Last autosave result can be found at %s\n", autosaveFilename);
	printf("residuals were written to %s\n", residualsFilename);
	printf("Profiling results were written to %s\n", timeFilename);
	printf("Final results were written to %s\n", finalFilename);

	//	compareTestFiles("./TestValues/CUDA/rpert.txt", "./TestValues/CUDA/rpert_gpu.txt");
	hipEventDestroy(start);
	hipEventDestroy(stop);
	freeAllHost(hostArrays, sizeof(hostArrays) / sizeof(hostArrays[0]));
	freeAllGpu(gpuArrays, sizeof(gpuArrays) / sizeof(gpuArrays[0]));
	return 0;
}
