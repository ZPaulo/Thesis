#include "hip/hip_runtime.h"
/**
 * Memory allocation wrappers for host and gpu arrays
 * @file ArrayUtils.cu
 * @author Adam Koleszar (adam.koleszar@gmail.com)
 */
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <assert.h>    // ensure successfull allocation
#include "ArrayUtils.h"
#include "LogWriter.h"


__constant__ FLOAT_TYPE fill_fd; ///< value to fill into floating point array
__constant__ int        fill_id; ///< value to fill into integer array
__constant__ int        fill_idBool; ///< value to fill into integer array
__constant__ unsigned long long        fill_idLL; ///< value to fill into integer array

FLOAT_TYPE getRandom(unsigned long *seed) {
    *seed = (*seed * 279470273u) % 4294967291u;
    return (FLOAT_TYPE)*seed / 4294967291.;
}

/**
 * Get a uniform random number on GPU
 * @param seed seed for the random number (initialise with time(NULL))
 * @return uniform random number
 */
__device__ FLOAT_TYPE getRandomDev(unsigned long seed) {
    seed = (seed * 279470273u) % 4294967291u;
    return (FLOAT_TYPE)seed / 4294967291.;
}
bool *createGpuArrayBool(int length, ArrayOption op, bool fill, bool *copy)
{
    bool *array_d;
    hipError_t err = hipMalloc((void**)&array_d, length);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error allocating on GPU: %d\n", err);
        return NULL;
    }
    dim3 bpg((int)(sqrt(length-1)/THREADS)+1,(int)(sqrt(length-1)/THREADS)+1);
    dim3 tpb(THREADS, THREADS);
    hipMemcpyToSymbol(HIP_SYMBOL(fill_idBool), &fill, 1);
    switch (op)
    {
        case ARRAY_ZERO:
            hipMemset(array_d, 0, length);
        break;
        case ARRAY_FILL:
            gpuArrayFillBool<<<bpg,tpb>>>(array_d, length);
        break;
        case ARRAY_COPY:
            hipMemcpy(array_d, copy, length, hipMemcpyHostToDevice);
        break;
        case ARRAY_CPYD:
            hipMemcpy(array_d, copy, length, hipMemcpyDeviceToDevice);
        break;
        default:
        	writeAlertLog("ArrayUtils.cu", 60);
        break;
    }
#ifdef DEBUG
    printf("cm - %p (%ldB)\n", array_d, length);
#endif
    return array_d;
}
int *createGpuArrayInt(int length, ArrayOption op, int fill, int *copy)
{
    int *array_d;
    hipError_t err = hipMalloc((void**)&array_d, SIZEINT(length));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error allocating on GPU: %d\n", err);
        return NULL;
    }
    dim3 bpg((int)(sqrt(length-1)/THREADS)+1,(int)(sqrt(length-1)/THREADS)+1);
    dim3 tpb(THREADS, THREADS);
    hipMemcpyToSymbol(HIP_SYMBOL(fill_id), &fill, SIZEINT(1));
    switch (op)
    {
        case ARRAY_ZERO:
            hipMemset(array_d, 0, SIZEINT(length));
        break;
        case ARRAY_FILL:
            gpuArrayFillInt<<<bpg,tpb>>>(array_d, length);
        break;
        case ARRAY_COPY:
            hipMemcpy(array_d, copy, SIZEINT(length), hipMemcpyHostToDevice);
        break;
        case ARRAY_CPYD:
            hipMemcpy(array_d, copy, SIZEINT(length), hipMemcpyDeviceToDevice);
        break;
        default:
        	writeAlertLog("ArrayUtils.h", 42);
        break;
    }
#ifdef DEBUG
    printf("cm - %p (%ldB)\n", array_d, SIZEINT(length));
#endif
    return array_d;
}
unsigned  long long  *createGpuArrayLongLong(int length, ArrayOption op, unsigned long long fill, unsigned long long *copy)
{
    unsigned long long *array_d;
    hipError_t err = hipMalloc((void**)&array_d, SIZELONGLONG(length));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error allocating on GPU: %d\n", err);
        return NULL;
    }
    dim3 bpg((int)(sqrt(length-1)/THREADS)+1,(int)(sqrt(length-1)/THREADS)+1);
    dim3 tpb(THREADS, THREADS);
    hipMemcpyToSymbol(HIP_SYMBOL(fill_idLL), &fill, SIZELONGLONG(1));
    switch (op)
    {
        case ARRAY_ZERO:
            hipMemset(array_d, 0ULL, SIZELONGLONG(length));
        break;
        case ARRAY_FILL:
            gpuArrayFillLongLong<<<bpg,tpb>>>(array_d, length);
        break;
        case ARRAY_COPY:
            hipMemcpy(array_d, copy, SIZELONGLONG(length), hipMemcpyHostToDevice);
        break;
        case ARRAY_CPYD:
            hipMemcpy(array_d, copy, SIZELONGLONG(length), hipMemcpyDeviceToDevice);
        break;
        default:
        	writeAlertLog("ArrayUtils.h", 42);
        break;
    }
#ifdef DEBUG
    printf("cm - %p (%ldB)\n", array_d, SIZEINT(length));
#endif
    return array_d;
}
FLOAT_TYPE *createGpuArrayFlt(int length, ArrayOption op, FLOAT_TYPE fill, FLOAT_TYPE *copy)
{
    FLOAT_TYPE *array_d;
    hipError_t err = hipMalloc((void**)&array_d, SIZEFLT(length));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error allocating on GPU: %d\n", err);
        return NULL;
    }
    unsigned long seed = time(NULL);
    dim3 bpg((int)(sqrt(length-1)/THREADS)+1,(int)(sqrt(length-1)/THREADS)+1);
    dim3 tpb(THREADS, THREADS);
    hipMemcpyToSymbol(HIP_SYMBOL(fill_fd), &fill, SIZEFLT(1));
    switch (op)
    {
        case ARRAY_ZERO:
            hipMemset(array_d, 0, SIZEFLT(length));
        break;
        case ARRAY_FILL:
            gpuArrayFillFlt<<<bpg,tpb>>>(array_d, length);
        break;
        case ARRAY_COPY:
            hipMemcpy(array_d, copy, SIZEFLT(length), hipMemcpyHostToDevice);
        break;
        case ARRAY_CPYD:
            hipMemcpy(array_d, copy, SIZEFLT(length), hipMemcpyDeviceToDevice);
        break;
        case ARRAY_RAND:
            gpuArrayFillRandom<<<bpg,tpb>>>(array_d, seed, length);
        break;
        default:
        	writeAlertLog("ArrayUtils.h", 78);
        break;
    }
#ifdef DEBUG
    printf("cm - %p (%ldB)\n", array_d, SIZEFLT(length));
#endif
    return array_d;
}

bool *createHostArrayBool(int length, ArrayOption op, bool fill, bool *copy)
{
    bool *array_h;
    array_h = (bool*)malloc(length);
    switch (op)
    {
        case ARRAY_ZERO:
            memset(array_h, 0, length);
        break;
        case ARRAY_FILL:
            hostArrayFillBool(array_h, fill, length);
        break;
        case ARRAY_COPY:
            memcpy(array_h, copy, length);
        break;
        default:
        	writeAlertLog("ArrayUtils.cu", 195);
        break;
    }
    return array_h;
}
int *createHostArrayInt(int length, ArrayOption op, int fill, int *copy)
{
    int *array_h;
    array_h = (int*)malloc(SIZEINT(length));
    switch (op)
    {
        case ARRAY_ZERO:
            memset(array_h, 0, SIZEINT(length));
        break;
        case ARRAY_FILL:
            hostArrayFillInt(array_h, fill, length);
        break;
        case ARRAY_COPY:
            memcpy(array_h, copy, SIZEINT(length));
        break;
        default:
        	writeAlertLog("ArrayUtils.h", 106);
        break;
    }
    return array_h;
}
unsigned long long *createHostArrayLongLong(int length, ArrayOption op, unsigned long long fill, unsigned long long *copy)
{
    unsigned long long *array_h;
    array_h = (unsigned long long*)malloc(SIZELONGLONG(length));
    switch (op)
    {
        case ARRAY_ZERO:
            hostArrayFillLongLong(array_h, 0ULL, length);
        break;
        case ARRAY_FILL:
            hostArrayFillLongLong(array_h, fill, length);
        break;
        case ARRAY_COPY:
            memcpy(array_h, copy, SIZELONGLONG(length));
        break;
        default:
        	writeAlertLog("ArrayUtils.h", 106);
        break;
    }
    return array_h;
}

FLOAT_TYPE *createHostArrayFlt(int length, ArrayOption op, FLOAT_TYPE fill, FLOAT_TYPE *copy)
{
    FLOAT_TYPE *array_h;
    array_h = (FLOAT_TYPE*)malloc(SIZEFLT(length));
    switch (op)
    {
        case ARRAY_ZERO:
            memset(array_h, 0, SIZEFLT(length));
        break;
        case ARRAY_FILL:
            hostArrayFillFlt(array_h, fill, length);
            // printf("fill: %f\n", array_h[0]);
        break;
        case ARRAY_COPY:
            memcpy(array_h, copy, SIZEFLT(length));
        break;
        case ARRAY_RAND:
            hostArrayFillRandom(array_h, length, (fill)?fill:1.0);
        break;
        default:
        	writeAlertLog("ArrayUtils.h", 128);
        break;
    }
    return array_h;
}

int **create2DHostArrayInt(int width, int height)
{
    int **MyMatrix;
    int i;
    MyMatrix = (int **)calloc(height,sizeof(int*));
    assert(MyMatrix != NULL);
    for (i = 0; i < height; i++)
        MyMatrix[i] = (int *)calloc(width,sizeof(int));
    assert(MyMatrix != NULL);
    return MyMatrix;
}

FLOAT_TYPE **create2DHostArrayFlt(int width, int height)
{
    FLOAT_TYPE **MyMatrix;
    int i;
    MyMatrix = (FLOAT_TYPE **)calloc(height,sizeof(FLOAT_TYPE*));
    assert(MyMatrix != NULL);
    for (i = 0; i < height; i++)
        MyMatrix[i] = (FLOAT_TYPE *)calloc(width,sizeof(FLOAT_TYPE));
    assert(MyMatrix != NULL);
    return MyMatrix;
}

int ***create3DHostArrayInt(int width, int height, int depth)
{
    int ***MyMatrix;
    int i, j, k;

    MyMatrix = (int ***)calloc(height,sizeof(int**));
    assert(MyMatrix != NULL);
    for (i = 0; i < height; i++)
    {
        MyMatrix[i] = (int **)calloc(width,sizeof(int*));
        assert(MyMatrix != NULL);
        for (j = 0; j < width; j++)
        {
            MyMatrix[i][j] = (int *)calloc(depth,sizeof(int));
            assert(MyMatrix != NULL);
            for (k = 0; k < depth; k++)
                MyMatrix[i][j][k] = 0;
        }
    }
    return MyMatrix;
}


FLOAT_TYPE ***create3DHostArrayFlt(int width, int height, int depth)
{
    FLOAT_TYPE ***MyMatrix;
    int i, j, k;

    MyMatrix = (FLOAT_TYPE ***)calloc(height,sizeof(FLOAT_TYPE**));
    assert(MyMatrix != NULL);
    for (i = 0; i < height; i++)
    {
        MyMatrix[i] = (FLOAT_TYPE **)calloc(width,sizeof(FLOAT_TYPE*));
        assert(MyMatrix != NULL);
        for (j = 0; j < width; j++)
        {
            MyMatrix[i][j] = (FLOAT_TYPE *)calloc(depth,sizeof(FLOAT_TYPE));
            assert(MyMatrix != NULL);
            for (k = 0; k < depth; k++)
                MyMatrix[i][j][k] = 0;
        }
    }
    return MyMatrix;
}

bool ***create3DHostArrayBool(int width, int height, int depth)
{
    bool ***MyMatrix;
    int i, j, k;

    MyMatrix = (bool ***)calloc(height,sizeof(bool**));
    assert(MyMatrix != NULL);
    for (i = 0; i < height; i++)
    {
        MyMatrix[i] = (bool **)calloc(width,sizeof(bool*));
        assert(MyMatrix != NULL);
        for (j = 0; j < width; j++)
        {
            MyMatrix[i][j] = (bool *)calloc(depth,sizeof(bool));
            assert(MyMatrix != NULL);
            for (k = 0; k < depth; k++)
                MyMatrix[i][j][k] = 0;
        }
    }
    return MyMatrix;
}
__global__ void gpuArrayFillBool(bool *array_d, int size)
{
	int blockId = blockIdx.x
	+ blockIdx.y * gridDim.x;
    int i =  blockId * (blockDim.x * blockDim.y)
		+ (threadIdx.y * blockDim.x)
		+ threadIdx.x;
    if (i<size) array_d[i] = fill_idBool;
}
__global__ void gpuArrayFillInt(int *array_d, int size)
{
	int blockId = blockIdx.x
	+ blockIdx.y * gridDim.x;
    int i =  blockId * (blockDim.x * blockDim.y)
		+ (threadIdx.y * blockDim.x)
		+ threadIdx.x;
    if (i<size) array_d[i] = fill_id;
}
__global__ void gpuArrayFillLongLong(unsigned long long *array_d, int size)
{
	int blockId = blockIdx.x
	+ blockIdx.y * gridDim.x;
    int i =  blockId * (blockDim.x * blockDim.y)
		+ (threadIdx.y * blockDim.x)
		+ threadIdx.x;
    if (i<size) array_d[i] = fill_idLL;
}

__global__ void gpuArrayFillFlt(FLOAT_TYPE *array_d, int size)
{
	int blockId = blockIdx.x
	+ blockIdx.y * gridDim.x;
    int i =  blockId * (blockDim.x * blockDim.y)
		+ (threadIdx.y * blockDim.x)
		+ threadIdx.x;
    if (i<size) array_d[i] = fill_fd;
}

__global__ void gpuArrayFillRandom(FLOAT_TYPE *array_d, unsigned long seed, int size)
{
	int blockId = blockIdx.x
	+ blockIdx.y * gridDim.x;
    int i =  blockId * (blockDim.x * blockDim.y)
		+ (threadIdx.y * blockDim.x)
		+ threadIdx.x;
    if (i<size) array_d[i] = getRandomDev(seed+i);
}
void hostArrayFillBool(bool *array_h, bool fill, int size)
{
    int i;
    for (i=0; i<size; ++i) array_h[i] = fill;
}
void hostArrayFillInt(int *array_h, int fill, int size)
{
    int i;
    for (i=0; i<size; ++i) array_h[i] = fill;
}

void hostArrayFillLongLong(unsigned long long *array_h, unsigned long long fill, int size)
{
    int i;
    for (i=0; i<size; ++i) array_h[i] = fill;
}

void hostArrayFillFlt(FLOAT_TYPE *array_h, FLOAT_TYPE fill, int size)
{
    int i;
    // printf("fill_in:%f -> %f\n", fill, array_h[0]);
    for (i=0; i<size; ++i) array_h[i] = fill;
}

void hostArrayFillRandom(FLOAT_TYPE *array_h, int size, FLOAT_TYPE r)
{
    int i;
    unsigned long seed = time(NULL);
    for (i=0; i<size; ++i) array_h[i] = r*getRandom(&seed);
}

void freeAllHost(void **as, int n)
{
    int i;
    for (i=0; i<n; ++i)
    {
        free(as[i]);
        as[i] = NULL;
    }
    as = NULL;
}

void freeAllGpu(void **as, int n)
{
    int i;
    for (i=0; i<n; ++i)
    {
        hipFree(as[i]);
        as[i] = NULL;
    }
    as = NULL;
}
