#include "hip/hip_runtime.h"
/**
 * @author Alfonso Aguilar (a.aguilar-pontes@cranfield.ac.uk) - implementation of the physics
 * @author Maciej Kubat (m.j.kubat@cranfield.ac.uk) - software aspects of the implementation
 */

#include <stdio.h>                      // printf();
#include <math.h>                       // need to compile with -lm
#include <stdlib.h>                     // for calloc();
#include <stdbool.h>                    // Include for bool type variables!
#include <string.h>                     // String operations
#include <time.h>                       // time functions
#include <errno.h>
#include "GpuFunctions.h"       // GPU kernels
#include "ShellFunctions.h"     // For convenience
#include "FilesReading.h"       // For reading files
#include "FilesWriting.h"       // For writing files e.g. tecplot
#include "CellFunctions.h"      // For cell modifications
#include "ComputeResiduals.h"   // residuals
#include "LogWriter.h"
#include "Iterate.h"
#include "ArrayUtils.h"
#include "Check.h"
#include "hip/hip_runtime.h"
#include "GpuSum.h"
#include "Multiphase.h"


#define CUDA 1

int Iterate3D(InputFilenames *inFn, Arguments *args) {
	// Time measurement: declaration, begin
	clock_t tStart = clock();

	FILE* logFile;               // file for log
	char autosaveFilename[768];  // autosave filename
	char outputFilename[768];    // initial data will be written to this file
	char finalFilename[768];     // final data will be written to this file
	char logFilename[768];       // path of the .log file
	char residualsFilename[768]; // path of the residuals file
	char timeFilename[768];      // path of time measurement file
	bool firstIter = true;
	bool *d_divergence;
	int AuxMacroDiff = 1;
	FLOAT_TYPE r = -1.0;
	logFilename[0] = '\0';
	residualsFilename[0] = '\0';
	timeFilename[0] = '\0';

	if (strlen(inFn->result)) {
		strcat(logFilename, inFn->result);
		strcat(residualsFilename, inFn->result);
		strcat(timeFilename, inFn->result);
	}
	strcat(logFilename, "lbmsolver.log");
	strcat(residualsFilename, "residuals.dat");
	strcat(timeFilename, "runtimes.dat");

	int autosaveIt = 1; // autosave i variable, will be incremented after every autosave
	int numNodes, numConns; // This will store the number of lines of the read files
	FLOAT_TYPE delta;          // grid spacing
	int n, m, h;                 // number of nodes in the x, y and z directions
	FLOAT_TYPE maxInletCoordY; // maximum inlet coordinate in y
	FLOAT_TYPE minInletCoordY; // minimum inlet coordinate in y
	FLOAT_TYPE maxInletCoordZ; // maximum inlet coordinate in z
	FLOAT_TYPE minInletCoordZ; // minimum inlet coordinate in z
	int numInletNodes;         // number of inlet nodes
	FLOAT_TYPE uMaxDiff = -1, vMaxDiff = -1, wMaxDiff = -1, rhoMaxDiff = -1, fMaxDiff = -1;
	int *nodeIdX, *nodeIdY, *nodeIdZ, *nodeType, *bcNodeIdX, *bcNodeIdY,
	*bcNodeIdZ, *latticeId, *bcType, *bcBoundId;
	FLOAT_TYPE *nodeX, *nodeY, *nodeZ, *bcX, *bcY, *bcZ;

	FLOAT_TYPE taskTime[9];
	int i;
	for (i = 0; i < 9; ++i) {
		taskTime[i] = 0.0;
	}

	clock_t tInstant1, tInstant2; // Time measurement points, universal
	clock_t tIterStart, tIterEnd; // Time measurement points: main loop

	// cuda time measurement variables
	hipEvent_t start, stop;
	float cudatime;
	CHECK(hipEventCreate(&start));
	CHECK(hipEventCreate(&stop));

	numNodes = readNodeFile(inFn->node, &nodeIdX, &nodeIdY, &nodeIdZ, &nodeX,
			&nodeY, &nodeZ, &nodeType, args->TypeOfProblem);
	if (numNodes == 0) {
		printf("NODES NOT FOUND in file\n");
		return 2;
	}

	int *fluid_d = createGpuArrayInt(numNodes, ARRAY_COPY, 0, nodeType);
	FLOAT_TYPE *coordX_d = createGpuArrayFlt(numNodes, ARRAY_COPY, 0., nodeX);
	FLOAT_TYPE *coordY_d = createGpuArrayFlt(numNodes, ARRAY_COPY, 0., nodeY);
	FLOAT_TYPE *coordZ_d = createGpuArrayFlt(numNodes, ARRAY_COPY, 0., nodeZ);

	numConns = readConnFile(inFn->bc, &bcNodeIdX, &bcNodeIdY, &bcNodeIdZ,
			&latticeId, &bcType, &bcX, &bcY, &bcZ, &bcBoundId,
			args->TypeOfProblem);
	if (numConns == 0) {
		printf("NEIGHBOURING NOT FOUND in file\n");
		return 2;
	}

	int *bcNodeIdX_d = createGpuArrayInt(numConns, ARRAY_COPY, 0, bcNodeIdX);
	int *bcNodeIdY_d = createGpuArrayInt(numConns, ARRAY_COPY, 0, bcNodeIdY);
	int *bcNodeIdZ_d = createGpuArrayInt(numConns, ARRAY_COPY, 0, bcNodeIdZ);
	int *latticeId_d = createGpuArrayInt(numConns, ARRAY_COPY, 0, latticeId);
	int *bcType_d = createGpuArrayInt(numConns, ARRAY_COPY, 0, bcType);
	int *bcBoundId_d = createGpuArrayInt(numConns, ARRAY_COPY, 0, bcBoundId);
	FLOAT_TYPE *bcX_d = createGpuArrayFlt(numConns, ARRAY_COPY, 0., bcX);
	FLOAT_TYPE *bcY_d = createGpuArrayFlt(numConns, ARRAY_COPY, 0., bcY);
	FLOAT_TYPE *bcZ_d = createGpuArrayFlt(numConns, ARRAY_COPY, 0., bcZ);

	m = getLastValue(nodeIdY, numNodes);
	n = getLastValue(nodeIdX, numNodes);
	h = getLastValue(nodeIdZ, numNodes);

	delta = getGridSpacing(nodeIdX, nodeIdY, nodeX, numNodes);
	//  printf("checkComment: delta, %f \n",delta);//checkComment
	numInletNodes = getNumInletNodes(bcType, latticeId, numConns,
			args->TypeOfProblem);
	maxInletCoordY = getMaxInletCoordY(bcType, latticeId, bcY, delta, numConns,
			args->TypeOfProblem);
	minInletCoordY = getMinInletCoordY(bcType, latticeId, bcY, delta, numConns,
			args->TypeOfProblem);
	maxInletCoordZ = getMaxInletCoordZ(bcType, latticeId, bcZ, delta, numConns,
			args->TypeOfProblem);
	minInletCoordZ = getMinInletCoordZ(bcType, latticeId, bcZ, delta, numConns,
			args->TypeOfProblem);


	printf("Nx: n= %d \n", n); //checkComment
	printf("Ny: m= %d \n", m); //checkComment
	printf("Nz: h= %d \n", h); //checkComment

	writeInitLog(logFilename, args, delta, m, n, h, numInletNodes,
			maxInletCoordY, minInletCoordY, maxInletCoordZ, minInletCoordZ);
	logFile = fopen(logFilename, "a");
	// In case of no autosave
	sprintf(autosaveFilename, "NOWHERE!");

	initConstants3D(args, maxInletCoordY, minInletCoordY, maxInletCoordZ,
			minInletCoordZ, delta, m, n, h);

	dim3 tpb(THREADS, THREADS); 					     // THREADS/block
	dim3 bpg1((int) (sqrt(m * n * h) / THREADS) + 1,
			(int) (sqrt(m * n * h) / THREADS) + 1);       // blocks/grid   MxNxH
	dim3 bpg18((int) (sqrt(18 * m * n * h) / THREADS) + 1,
			(int) (sqrt(18 * m * n * h) / THREADS) + 1);  // blocks/grid 18MxNxH
	dim3 bpg19((int) (sqrt(19 * m * n * h) / THREADS) + 1,
			(int) (sqrt(19 * m * n * h) / THREADS) + 1);  // blocks/grid 19MxNxH
	dim3 bpgBC((int) (sqrt(numConns) / THREADS) + 1,
			(int) (sqrt(numConns) / THREADS) + 1); 	 // blocks/grid N_BC

	// residuals
	FLOAT_TYPE *norm = createHostArrayFlt(args->iterations, ARRAY_ZERO);
	FLOAT_TYPE *dragSum = createHostArrayFlt(args->iterations, ARRAY_ZERO);
	FLOAT_TYPE *liftSum = createHostArrayFlt(args->iterations, ARRAY_ZERO);
	FLOAT_TYPE *latFSum = createHostArrayFlt(args->iterations, ARRAY_ZERO);


	fprintf(logFile, "\n:::: Initializing ::::\n");
	printf("\n:::: Initializing ::::\n");
	CHECK(hipEventRecord(start, 0));

	FLOAT_TYPE *u, *v, *w, *rho;

	int InitialCondLoadingErr = -1;
	if (args->UseInitialCondFromFile) {
		InitialCondLoadingErr = readInitConditionsFile(inFn->InitialConditions,
				numNodes, n, m, h, &u, &v, &w, &rho);
	} else {
		u = createHostArrayFlt(m * n * h, ARRAY_ZERO);
		v = createHostArrayFlt(m * n * h, ARRAY_ZERO);
		w = createHostArrayFlt(m * n * h, ARRAY_ZERO);
		rho = createHostArrayFlt(m * n * h, ARRAY_ZERO);
	}
	FLOAT_TYPE *rho_d;
	if (InitialCondLoadingErr)
		rho_d = createGpuArrayFlt(m * n * h, ARRAY_FILL, args->rho);
	else
		rho_d = createGpuArrayFlt(m * n * h, ARRAY_COPY, 0, rho);
	FLOAT_TYPE *u1_d, *v1_d, *w1_d;
	if (args->inletProfile == NO_INLET) {
		if (InitialCondLoadingErr) {
			u1_d = createGpuArrayFlt(m * n * h, ARRAY_FILL, args->u);
			v1_d = createGpuArrayFlt(m * n * h, ARRAY_FILL, args->v);
			w1_d = createGpuArrayFlt(m * n * h, ARRAY_FILL, args->w);
		} else {
			u1_d = createGpuArrayFlt(m * n * h, ARRAY_COPY, 0, u);
			v1_d = createGpuArrayFlt(m * n * h, ARRAY_COPY, 0, v);
			w1_d = createGpuArrayFlt(m * n * h, ARRAY_COPY, 0, w);
			printf("Initial conditions loaded from file\n");
		}
	}

	if (args->inletProfile == INLET) { 	 //m*h means to do in the inlet face
		printf(
				"Inlet profile is not currently available! Please initiate Inlet profile from file!\n");
		return 0;
		//		gpuInitInletProfile3D<<<(int) (m * h / THREADS) + 1, tpb>>>(u1_d, v1_d,
		//				w1_d, coordY_d, coordZ_d, m * h);
	}
	FLOAT_TYPE *u_prev_d, *v_prev_d, *w_prev_d, *rho_prev_d, *f_prev_d;
	if (args->TypeOfResiduals == MacroDiff) {
		if(args->multiPhase)
			f_prev_d = createGpuArrayFlt(m * n * h * 19, ARRAY_ZERO);
		else{
			u_prev_d = createGpuArrayFlt(m * n * h, ARRAY_ZERO);
			v_prev_d = createGpuArrayFlt(m * n * h, ARRAY_ZERO);
			w_prev_d = createGpuArrayFlt(m * n * h, ARRAY_ZERO);
			rho_prev_d = createGpuArrayFlt(m * n * h, ARRAY_ZERO);
		}
	}

	//Multiphase Color Gradient
	FLOAT_TYPE *f, *r_rho, *b_rho, *st_error, *color_gradient, *r_f, *b_f, *r_fColl, *b_fColl;
	if(args->multiPhase){
		r_rho = createHostArrayFlt(m * n * h, ARRAY_ZERO);
		b_rho = createHostArrayFlt(m * n * h, ARRAY_ZERO);
		st_error = createHostArrayFlt(args->iterations, ARRAY_ZERO);
	}

	FLOAT_TYPE aux1 = args->r_density / ((args->r_density + args->b_density) * args->r_viscosity) +
			args->b_density / ((args->r_density + args->b_density) * args->b_viscosity);
	FLOAT_TYPE mean_nu = 1.0/aux1;
	FLOAT_TYPE omega_eff = 1.0/(3.0*mean_nu+0.5);

	FLOAT_TYPE st_predicted = 4.0 * args->A / 9.0 / omega_eff;
#if !CUDA
	f = createHostArrayFlt(m * n * h * 19, ARRAY_ZERO);
	color_gradient = createHostArrayFlt(m * n * h * 3, ARRAY_ZERO);
	r_f = createHostArrayFlt(m * n * h * 19, ARRAY_ZERO);
	b_f = createHostArrayFlt(m * n * h * 19, ARRAY_ZERO);
	r_fColl = createHostArrayFlt(m * n * h * 19, ARRAY_ZERO);
	b_fColl = createHostArrayFlt(m * n * h * 19, ARRAY_ZERO);
	FLOAT_TYPE w_pert[19];
	int cx[19] = { 0, 1, -1, 0, 0, 0, 0, 1, -1, 1, -1, 1, -1, 1, -1, 0, 0, 0, 0 };
	int cy[19] = { 0, 0, 0, 1, -1, 0, 0, 1, 1, -1, -1, 0, 0, 0, 0, 1, -1, 1, -1 };
	int cz[19] = { 0, 0, 0, 0, 0, 1, -1, 0, 0, 0, 0, 1, 1, -1, -1, 1, 1, -1, -1 };
	FLOAT_TYPE phi[19];
	FLOAT_TYPE teta[19];
	FLOAT_TYPE chi[19];
	FLOAT_TYPE psi[19];
	FLOAT_TYPE cg_w[19];
	FLOAT_TYPE weight[19] =
	{ 1. / 3., 1. / 18., 1. / 18., 1. / 18., 1. / 18., 1. / 18., 1.
			/ 18., 1. / 36., 1. / 36., 1. / 36., 1. / 36., 1. / 36., 1.
			/ 36., 1. / 36., 1. / 36., 1. / 36., 1. / 36., 1. / 36., 1.
			/ 36. };
	if(args->multiPhase){
		int i;
		w_pert[0] = -2.0/ 9.0;
		phi[0]=0;
		teta[0] = 1;
		chi[0] = -5.0/2.0;
		psi[0] = 0;
		cg_w[0] = 0.0;
		for(i = 1; i < 7; i++){
			w_pert[i] = 1.0 / 54.0;
			phi[i] = 1.0 / 12.0;
			teta[i] = -1.0 / 12.0;
			chi[i] = -1.0 / 6.0;
			psi[i] = 1.0 / 4.0;
			cg_w[0] = 1.0 / 6.0;
		}
		for(i = 7; i < 19; i++){
			w_pert[i] = 1.0 / 27.0;
			phi[i] = 1.0 / 24.0;
			teta[i] = -1.0 / 24.0;
			chi[i] = 1.0 / 24.0;
			psi[i] = 1.0 / 8.0;
			cg_w[0] = 1.0 / 12.0;
		}

		createBubble3D(nodeX, nodeY,nodeZ, n, m, h,args->bubble_radius, r_f, b_f,r_rho,b_rho, args->r_density, args->b_density, phi, rho, f);
	}
#endif

	int *cg_directions, *cg_dir_d;
	FLOAT_TYPE *r_rho_d, *b_rho_d, *r_f_d, *b_f_d, *r_fColl_d, *b_fColl_d, *p_in_d, *p_out_d;
	int *num_in_d, *num_out_d;
	if(args->multiPhase){
		r_rho_d = createGpuArrayFlt(m * n * h, ARRAY_ZERO);
		b_rho_d = createGpuArrayFlt(m * n * h, ARRAY_ZERO);
		r_f_d = createGpuArrayFlt(m * n * h * 19, ARRAY_ZERO);
		b_f_d = createGpuArrayFlt(m * n * h * 19, ARRAY_ZERO);
		r_fColl_d = createGpuArrayFlt(m * n * h * 19, ARRAY_ZERO);
		b_fColl_d = createGpuArrayFlt(m * n * h * 19, ARRAY_ZERO);
		cg_dir_d = createGpuArrayInt(m * n * h, ARRAY_ZERO);
		cg_directions = createHostArrayInt(n * m * h, ARRAY_ZERO);
		if(args->test_case == 1){
			p_in_d = createGpuArrayFlt(n*m*h, ARRAY_ZERO);
			p_out_d = createGpuArrayFlt(n*m*h, ARRAY_ZERO);
			num_in_d = createGpuArrayInt(n*m*h, ARRAY_ZERO);
			num_out_d = createGpuArrayInt(n*m*h, ARRAY_ZERO);
		}
	}

	FLOAT_TYPE *f_d = createGpuArrayFlt(19 * m * n * h, ARRAY_ZERO);
	FLOAT_TYPE *fColl_d = createGpuArrayFlt(19 * m * n * h, ARRAY_ZERO);
	FLOAT_TYPE *f1_d, *fprev_d;
	if (args->TypeOfResiduals == FdRelDiff) {
		fprev_d = createGpuArrayFlt(19 * m * n * h, ARRAY_ZERO);
	}

#if CUDA
	FLOAT_TYPE p_in_mean;
	FLOAT_TYPE p_out_mean;
	FLOAT_TYPE ms = n * m * h;
	if(args->multiPhase){
		if(args->high_order)
			initHOColorGradient3D(cg_directions, n, m, h);
		else
			initColorGradient3D(cg_directions, n, m, h);
		CHECK(hipMemcpy(cg_dir_d, cg_directions, SIZEINT(m*n*h), hipMemcpyHostToDevice));
		initCGBubble3D<<<bpg1,tpb>>>(coordX_d,coordY_d,coordZ_d,r_rho_d, b_rho_d, rho_d, r_f_d, b_f_d, f_d, args->test_case);
	}
#endif

	if(args->multiPhase){
#if !CUDA
		CHECK(hipMemcpy(f_d,f,SIZEFLT(m*n*h*19),hipMemcpyHostToDevice));
		CHECK(hipMemcpy(r_rho_d,r_rho,SIZEFLT(m*n*h),hipMemcpyHostToDevice));
		CHECK(hipMemcpy(b_rho_d,b_rho,SIZEFLT(m*n*h),hipMemcpyHostToDevice));
		CHECK(hipMemcpy(rho_d,rho,SIZEFLT(m*n*h),hipMemcpyHostToDevice));
#endif
	}

	FLOAT_TYPE *temp19a_d, *temp19b_d;
	if(args->multiPhase){
		temp19a_d = createGpuArrayFlt(19 * m * n * h, ARRAY_ZERO);
		temp19b_d = createGpuArrayFlt(19 * m * n * h, ARRAY_ZERO);
	}
	else if (args->TypeOfResiduals != MacroDiff) {
		temp19a_d = createGpuArrayFlt(19 * m * n * h, ARRAY_ZERO);
		temp19b_d = createGpuArrayFlt(19 * m * n * h, ARRAY_ZERO);
	}
	FLOAT_TYPE *tempA_d = createGpuArrayFlt(m * n * h, ARRAY_ZERO);
	FLOAT_TYPE *tempB_d = createGpuArrayFlt(m * n * h, ARRAY_ZERO);

	int *mask = createHostArrayInt(m * n * h, ARRAY_ZERO);
	unsigned long long *bcMask = createHostArrayLongLong(m * n * h, ARRAY_ZERO);
	int *bcIdx = createHostArrayInt(m * n * h, ARRAY_ZERO);

	FLOAT_TYPE *u_d, *v_d, *w_d;

		u_d = createGpuArrayFlt(m * n * h, ARRAY_CPYD, 0, u1_d);
		v_d = createGpuArrayFlt(m * n * h, ARRAY_CPYD, 0, v1_d);
		w_d = createGpuArrayFlt(m * n * h, ARRAY_CPYD, 0, w1_d);



	bool *stream = createHostArrayBool(18 * m * n * h, ARRAY_FILL, 1);
	FLOAT_TYPE *q = createHostArrayFlt(18 * m * n * h, ARRAY_FILL, 0.5);

	int bcCount = initBoundaryConditions3D(bcNodeIdX, bcNodeIdY, bcNodeIdZ, q,
			bcBoundId, nodeType, bcX, bcY, bcZ, nodeX, nodeY, nodeZ, latticeId,
			stream, bcType, bcMask, bcIdx, mask, delta, m, n, h, numConns,
			args->boundaryType);
	unsigned long long *bcMask_d = createGpuArrayLongLong(m * n * h, ARRAY_COPY,
			0, bcMask);
	int *bcIdxCollapsed_d = createGpuArrayInt(bcCount, ARRAY_ZERO);
	unsigned long long *bcMaskCollapsed_d = createGpuArrayLongLong(bcCount,
			ARRAY_ZERO);

	FLOAT_TYPE *qCollapsed_d;
	if (args->boundaryType == CURVED)
		qCollapsed_d = createGpuArrayFlt(18 * bcCount, ARRAY_ZERO);

	dim3 bpgB((int) (sqrt(bcCount) / THREADS) + 1,
			(int) (sqrt(bcCount) / THREADS) + 1); // blocks/grid

	int *bcIdx_d = createGpuArrayInt(m * n * h, ARRAY_COPY, 0, bcIdx);

	collapseBc3D(bcIdx, bcIdxCollapsed_d, bcMask, bcMaskCollapsed_d, q,
			qCollapsed_d, mask, m, n, h, bcCount, args->boundaryType);

	bool *stream_d = createGpuArrayBool(18 * m * n * h, ARRAY_COPY, 0, stream);

#if CUDA
	CHECK(hipMemcpy(u, u_d, SIZEFLT(m*n*h), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(v, v_d, SIZEFLT(m*n*h), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(w, w_d, SIZEFLT(m*n*h), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(rho, rho_d, SIZEFLT(m*n*h), hipMemcpyDeviceToHost));
	if(args->multiPhase){
		CHECK(hipMemcpy(r_rho, r_rho_d, SIZEFLT(m*n*h), hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(b_rho, b_rho_d, SIZEFLT(m*n*h), hipMemcpyDeviceToHost));
	}
#endif
	CHECK(hipEventRecord(stop, 0));
	CHECK(hipEventSynchronize(stop));
	CHECK(hipEventElapsedTime(&cudatime, start, stop));
	taskTime[T_INIT] += cudatime / 1000;

	fclose(logFile);
	writeNodeNumbers(logFilename, numNodes, numConns, bcCount);
	logFile = fopen(logFilename, "a");

	void *hostArrays[] = { nodeIdX, nodeIdY, nodeIdZ, nodeX, nodeY, nodeZ,
			nodeType, bcNodeIdX, bcNodeIdY, bcNodeIdZ, latticeId, bcType, bcX,
			bcY, bcZ, bcBoundId, u, v, w, rho, mask, bcMask, bcIdx, stream, q,
			norm, dragSum, liftSum, latFSum};

	void *gpuArrays[] =
	{ coordX_d, coordY_d, coordZ_d, fluid_d, bcNodeIdX_d, bcNodeIdY_d,
			bcNodeIdZ_d, latticeId_d, bcType_d, bcX_d, bcY_d, bcZ_d,
			bcBoundId_d, u_d, v_d, w_d, rho_d, u1_d, v1_d, w1_d, f_d, fColl_d, tempA_d, tempB_d,
			bcMaskCollapsed_d, bcIdx_d, bcIdxCollapsed_d,
			stream_d, qCollapsed_d}; //drag_d, lift_d, latF_d,


	void *mpHostArrays[] = {
			r_rho, b_rho, st_error, color_gradient, r_f,b_f, r_fColl, b_fColl, cg_directions, f
	};

	void *mpGpuArrays[] = {
			r_rho_d, b_rho_d, r_f_d, b_f_d, r_fColl_d, b_fColl_d, cg_dir_d
	};

	void *FDdifGpuArrays[] = {
			fprev_d, f1_d
	};

	void *nonMacroDiffGpuArrays[] = {
			temp19a_d, temp19b_d
	};

	fprintf(logFile, "\n:::: Initialization done! ::::\n");

	printf("Initialization took %f seconds\n", taskTime[T_INIT]);

	// Write Initialized data
	switch (args->outputFormat) {
	case CSV:
		sprintf(outputFilename, "%sInitialData.csv", inFn->result);
		break;
	case TECPLOT:
		sprintf(outputFilename, "%sInitialData.dat", inFn->result);
		break;
	case PARAVIEW:
		sprintf(outputFilename, "%sInitialData.vti", inFn->result);
		break;
	}

	tInstant1 = clock(); // Start measuring time
	if(args->multiPhase){
		WriteResultsMultiPhase(outputFilename, nodeType, nodeX, nodeY, nodeZ, u, v, w, rho,r_rho,b_rho, nodeType,
				n, m, h, args->outputFormat);
	}
	else{
		WriteResults3D(outputFilename, nodeType, nodeX, nodeY, nodeZ, u, v, w, rho,
				nodeType, n, m, h, args->outputFormat);
	}
	tInstant2 = clock();
	taskTime[T_WRIT] += (FLOAT_TYPE) (tInstant2 - tInstant1) / CLOCKS_PER_SEC;

	printf("\nInitialized data was written to %s\n", outputFilename);

	////////////////// ITERATION ///////////////////////

	fprintf(logFile, "\n:::: Start Iterations ::::\n");
	printf("\n:::: Start Iterations ::::\n");

	printf("%d is the number of iterations \n", args->iterations);

	tIterStart = clock(); // Start measuring time of main loop
	size_t free, total;

	hipMemGetInfo(&free, &total);
	printf("^^^^ Free : %llu Mbytes \n",
			(unsigned long long) free / 1024 / 1024);

	printf("^^^^ Total: %llu Mbytes \n",
			(unsigned long long) total / 1024 / 1024);

	printf("^^^^ %f%% free, %f%% used\n", 100.0 * free / (double) total,
			100.0 * (total - free) / (double) total);
	int iter = 0;
	while (iter < args->iterations) {
		CHECK(hipDeviceSynchronize());
		CHECK(hipEventRecord(start, 0)); // Start measuring time
		////////////// COLLISION ///////////////
		switch (args->collisionModel) {
		case BGKW:
			if(args->multiPhase){
#if !CUDA
				mp3DColl(n, m, h, rho, u, v, w, r_rho, b_rho, w_pert, color_gradient, args->beta,
						args->g_limit, args->A,r_fColl, b_fColl, weight, cx, cy, cz, f, args->r_viscosity,
						args->b_viscosity, args->r_alpha, args->b_alpha, chi, phi, psi, teta, cg_w);
#else
				gpuCollBgkwGC3D<<<bpg1, tpb>>>(fluid_d, rho_d, r_rho_d, b_rho_d, u_d, v_d, w_d, f_d, r_fColl_d, b_fColl_d, cg_dir_d, args->high_order);
#endif
			}
			else{
				gpuCollBgkw3D<<<bpg1, tpb>>>(fluid_d, rho_d, u_d, v_d, w_d, f_d,
						fColl_d);
			}
			break;

		case TRT:
			printf("TRT not implemented in 3D go for MRT \n");
			//        gpuCollTrt<<<bpg1,tpb>>>(fluid_d, rho_d, u_d, v_d, w_d, f_d, fColl_d);
			break;

		case MRT:
			gpuCollMrt3D<<<bpg1, tpb>>>(fluid_d, rho_d, u_d, v_d, w_d, f_d,
					fColl_d);
			break;
		}

		CHECK(hipEventRecord(stop, 0));
		CHECK(hipEventSynchronize(stop));
		CHECK(hipEventElapsedTime(&cudatime, start, stop));
		taskTime[T_COLL] += cudatime;

		////////////// STREAMING ///////////////
		CHECK(hipDeviceSynchronize());
		CHECK(hipEventRecord(start, 0));

		if(args->multiPhase){
#if !CUDA
			streamMP3D(n, m, h, r_f, b_f, r_fColl, b_fColl, stream);
#else
			gpuStreaming3D<<<bpg1, tpb>>>(fluid_d, stream_d, r_f_d, r_fColl_d);
			gpuStreaming3D<<<bpg1, tpb>>>(fluid_d, stream_d, b_f_d, b_fColl_d);
#endif
		}
		else{
			gpuStreaming3D<<<bpg1, tpb>>>(fluid_d, stream_d, f_d, fColl_d);
		}

		CHECK(hipEventRecord(stop, 0));
		CHECK(hipEventSynchronize(stop));
		CHECK(hipEventElapsedTime(&cudatime, start, stop));
		taskTime[T_STRM] += cudatime;

		// make the host block until the device is finished with foo
		CHECK(hipDeviceSynchronize());

		// check for error
		hipError_t error = hipGetLastError();
		if (error != hipSuccess) {
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			exit(-1);
		}

		////////////// BOUNDARIES ///////////////
		CHECK(hipDeviceSynchronize());
		CHECK(hipEventRecord(start, 0));

		if(args->multiPhase){
#if !CUDA
			peridicBoundaries3D(n, m, h,r_f, b_f, r_rho, b_rho);
#else
			gpuBcInlet3D<<<bpgB, tpb>>>(bcIdxCollapsed_d, bcMaskCollapsed_d, r_f_d,
					u1_d, v1_d, w1_d, bcCount);
			gpuBcInlet3D<<<bpgB, tpb>>>(bcIdxCollapsed_d, bcMaskCollapsed_d, b_f_d,
					u1_d, v1_d, w1_d, bcCount);
			switch (args->bcwallmodel) {
			case SIMPLE:
				gpuBcSimpleWall3D<<<bpgB, tpb>>>(bcIdxCollapsed_d,
						bcMaskCollapsed_d, r_f_d, r_fColl_d, qCollapsed_d, bcCount);
				gpuBcSimpleWall3D<<<bpgB, tpb>>>(bcIdxCollapsed_d,
						bcMaskCollapsed_d, b_f_d, b_fColl_d, qCollapsed_d, bcCount);

				break;
			case COMPLEX:
				gpuBcComplexWall3D<<<bpgB, tpb>>>(bcIdxCollapsed_d,
						bcMaskCollapsed_d, r_f_d, r_fColl_d, qCollapsed_d, bcCount);
				gpuBcComplexWall3D<<<bpgB, tpb>>>(bcIdxCollapsed_d,
						bcMaskCollapsed_d, b_f_d, b_fColl_d, qCollapsed_d, bcCount);

				break;
			}

			gpuBcPeriodic3D<<<bpgB, tpb>>>(bcIdxCollapsed_d, bcMaskCollapsed_d, r_f_d,
					bcCount);
			gpuBcPeriodic3D<<<bpgB, tpb>>>(bcIdxCollapsed_d, bcMaskCollapsed_d, b_f_d,
					bcCount);
#endif
		}
		else{
			gpuBcInlet3D<<<bpgB, tpb>>>(bcIdxCollapsed_d, bcMaskCollapsed_d, f_d,
					u1_d, v1_d, w1_d, bcCount);
			switch (args->bcwallmodel) {
			case SIMPLE:
				gpuBcSimpleWall3D<<<bpgB, tpb>>>(bcIdxCollapsed_d,
						bcMaskCollapsed_d, f_d, fColl_d, qCollapsed_d, bcCount);

				break;
			case COMPLEX:
				gpuBcComplexWall3D<<<bpgB, tpb>>>(bcIdxCollapsed_d,
						bcMaskCollapsed_d, f_d, fColl_d, qCollapsed_d, bcCount);

				break;
			}
			gpuBcOutlet3D<<<bpgB, tpb>>>(bcIdxCollapsed_d, bcMaskCollapsed_d, f_d,
					u_d, v_d, w_d, rho_d, bcCount);
			gpuBcPeriodic3D<<<bpgB, tpb>>>(bcIdxCollapsed_d, bcMaskCollapsed_d, f_d,
					bcCount);
			gpuBcSymm3D<<<bpgB, tpb>>>(bcIdxCollapsed_d, bcMaskCollapsed_d, f_d,
					bcCount);
		}
		CHECK(hipEventRecord(stop, 0));
		CHECK(hipEventSynchronize(stop));
		CHECK(hipEventElapsedTime(&cudatime, start, stop));
		taskTime[T_BNDC] += cudatime;

		// UPDATE VELOCITY AND DENSITY
		CHECK(hipDeviceSynchronize());
		CHECK(hipEventRecord(start, 0));

		if(args->multiPhase){
#if !CUDA
			updateMacroMP3D(n, m, h, u, v, w, r_rho, b_rho, r_f, b_f, rho, args->control_param,args->r_alpha, args->b_alpha,
					args->bubble_radius,st_error, iter, 1, cx, cy, cz, f);
#else
			gpuUpdateMacro3DCG<<<bpg1, tpb>>>(fluid_d, rho_d, u_d, v_d, w_d,
					bcBoundId_d, f_d, args->g,bcMask_d,args->UpdateInltOutl, r_f_d, b_f_d, r_rho_d, b_rho_d, p_in_d, p_out_d, num_in_d, num_out_d, args->test_case);
			switch(args->test_case){
			case 1:
				p_in_mean = gpu_sum_h(p_in_d, p_in_d, ms) / gpu_sum_int_h(num_in_d, num_in_d, ms);
				p_out_mean = gpu_sum_h(p_out_d, p_out_d, ms) / gpu_sum_int_h(num_out_d, num_out_d, ms);
				st_error[iter] = calculateSurfaceTension3D(p_in_mean, p_out_mean,args->r_alpha, args->b_alpha, args->bubble_radius * n, st_predicted);
				break;
			default:
				break;
			}
#endif
		}
		else{
			gpuUpdateMacro3D<<<bpg1, tpb>>>(fluid_d, rho_d, u_d, v_d, w_d,
					bcBoundId_d, coordX_d, coordY_d, coordZ_d, f_d, args->g,bcMask_d,args->UpdateInltOutl);
		}
		tInstant2 = clock();
		CHECK(hipEventRecord(stop, 0));
		CHECK(hipEventSynchronize(stop));
		CHECK(hipEventElapsedTime(&cudatime, start, stop));
		taskTime[T_MACR] += cudatime;

		// COMPUTE RESIDUALS

		if (AuxMacroDiff * args->ShowMacroDiff == iter + 1) {
			CHECK(hipDeviceSynchronize());
			CHECK(hipEventRecord(start, 0));

			if (args->TypeOfResiduals == L2) {
				if(args->multiPhase){
#if !CUDA
					CHECK(hipMemcpy(f_d,r_f,SIZEFLT(m*n*h*19),hipMemcpyHostToDevice));
					CHECK(hipMemcpy(fColl_d,r_fColl,SIZEFLT(m*n*h*19),hipMemcpyHostToDevice));
#endif
				}
				r = computeResidual3D(f_d, fColl_d, temp19a_d, temp19b_d, m, n,
						h);
			}
			else {
				if (args->TypeOfResiduals == FdRelDiff) {
					if(args->multiPhase){
#if !CUDA
						CHECK(hipMemcpy(f_d,r_f,SIZEFLT(m*n*h*19),hipMemcpyHostToDevice));
#endif
					}

					if (firstIter) {
						firstIter = false;
						f1_d = createGpuArrayFlt(19 * n * m * h, ARRAY_CPYD, 0,
								f_d);

					}
					r = computeNewResidual3D(f_d, fprev_d, f1_d, temp19a_d,
							temp19b_d, m, n, h);
					CHECK(hipFree(fprev_d));
					fprev_d = createGpuArrayFlt(19 * n * m * h, ARRAY_CPYD, 0,
							f_d);

				} else {
					bool h_divergence = false;
					CHECK(hipMalloc(&d_divergence,sizeof(bool)));
					CHECK(hipMemcpy(d_divergence,&h_divergence,sizeof(bool),hipMemcpyHostToDevice));
					if(args->multiPhase){
						gpu_abs_sub<<<bpg1, tpb>>>(f_d, f_prev_d, temp19a_d, n * m * h * 19, d_divergence);
						fMaxDiff = gpu_max_h(temp19a_d, temp19b_d, n * m * h * 19);
					}
					else{
						gpu_abs_sub<<<bpg1, tpb>>>(u_d, u_prev_d, tempA_d,
								n * m * h, d_divergence);
						uMaxDiff = gpu_max_h(tempA_d, tempB_d, n * m * h);
						gpu_abs_sub<<<bpg1, tpb>>>(v_d, v_prev_d, tempA_d,
								n * m * h, d_divergence);
						vMaxDiff = gpu_max_h(tempA_d, tempB_d, n * m * h);
						gpu_abs_sub<<<bpg1, tpb>>>(w_d, w_prev_d, tempA_d,
								n * m * h, d_divergence);
						wMaxDiff = gpu_max_h(tempA_d, tempB_d, n * m * h);
						gpu_abs_sub<<<bpg1, tpb>>>(rho_d, rho_prev_d, tempA_d,
								n * m * h, d_divergence);
						rhoMaxDiff = gpu_max_h(tempA_d, tempB_d, n * m * h);
					}
					CHECK(hipMemcpy(&h_divergence,d_divergence,sizeof(bool),hipMemcpyDeviceToHost));
					CHECK(hipFree(d_divergence));
					if (h_divergence) {
						fprintf(stderr, "\nDIVERGENCE!\n");
						break;
					}

					if(args->multiPhase){
						if(abs(fMaxDiff) < args->StopCondition[0]){
							printf("simulation converged!\n");
							break;
						}
					}
					else if (abs(uMaxDiff) < args->StopCondition[0] &&
							abs(vMaxDiff) < args->StopCondition[1] &&
							abs(wMaxDiff) < args->StopCondition[2] &&
							abs(rhoMaxDiff) < args->StopCondition[3]) {
						printf("simulation converged!\n");
						break;
					}

					if(args->multiPhase){
						CHECK(hipFree(f_prev_d));
						f_prev_d = createGpuArrayFlt(n * m * h * 19, ARRAY_CPYD, 0, f_d);
					}else{
						writeMacroDiffs(iter + 1, uMaxDiff, vMaxDiff, wMaxDiff,	rhoMaxDiff);
						CHECK(hipFree(u_prev_d));
						CHECK(hipFree(v_prev_d));
						CHECK(hipFree(w_prev_d));
						CHECK(hipFree(rho_prev_d));
						u_prev_d = createGpuArrayFlt(n * m * h, ARRAY_CPYD, 0, u_d);
						v_prev_d = createGpuArrayFlt(n * m * h, ARRAY_CPYD, 0, v_d);
						w_prev_d = createGpuArrayFlt(n * m * h, ARRAY_CPYD, 0, w_d);
						rho_prev_d = createGpuArrayFlt(n * m * h, ARRAY_CPYD, 0,
								rho_d);
					}
				}
			}

			if (abs(r) < args->StopCondition[0]) {
				printf("simulation converged!\n");
				break;
			}
			if (r != r) {
				fprintf(stderr, "\nDIVERGENCE!\n");
				break;
			}

			CHECK(hipEventRecord(stop, 0));
			CHECK(hipEventSynchronize(stop));
			CHECK(hipEventElapsedTime(&cudatime, start, stop));
			taskTime[T_RESI] += cudatime;

			AuxMacroDiff++;

		}
		if(args->multiPhase){
			CHECK(hipFree(f_prev_d));
			f_prev_d = createGpuArrayFlt(n * m * h * 19, ARRAY_CPYD, 0, f_d);
		}
		norm[iter] = r;
		if(args->multiPhase){
			printf(
					"Iterating... %d/%d (%3.1f %%) Max macro diffs: f= %.10f\r",
					iter + 1, args->iterations,
					(FLOAT_TYPE) (iter + 1) * 100
					/ (FLOAT_TYPE) (args->iterations), fMaxDiff);
		}
		else if (args->TypeOfResiduals == MacroDiff) {
			printf(
					"Iterating... %d/%d (%3.1f %%) Max macro diffs: u= %.10f v= %.10f w= %.10f rho= %.10f \r",
					iter + 1, args->iterations,
					(FLOAT_TYPE) (iter + 1) * 100
					/ (FLOAT_TYPE) (args->iterations), uMaxDiff,
					vMaxDiff, wMaxDiff, rhoMaxDiff);
		} else {
			printf("Iterating... %d/%d (%3.1f %%)  residual="FLOAT_FORMAT" \r",
					iter + 1, args->iterations,
					(FLOAT_TYPE) (iter + 1) * 100
					/ (FLOAT_TYPE) (args->iterations), r);
		}

		iter++; // update loop variable

		////////////// Autosave ///////////////

		if (iter == (args->autosaveEvery * autosaveIt)) {
			autosaveIt++;
			if (iter > args->autosaveAfter) {
				printf("autosave\n\n");
				//////////// COPY VARIABLES TO HOST ////////////////
				CHECK(hipMemcpy(u, u_d, SIZEFLT(m*n*h), hipMemcpyDeviceToHost));
				CHECK(hipMemcpy(v, v_d, SIZEFLT(m*n*h), hipMemcpyDeviceToHost));
				CHECK(hipMemcpy(w, w_d, SIZEFLT(m*n*h), hipMemcpyDeviceToHost));
				CHECK(hipMemcpy(rho, rho_d, SIZEFLT(m*n*h), hipMemcpyDeviceToHost));

				switch (args->outputFormat) {
				case CSV:
					sprintf(autosaveFilename, "%sautosave_iter%05d.csv",
							inFn->result, iter);
					break;
				case TECPLOT:
					sprintf(autosaveFilename, "%sautosave_iter%05d.dat",
							inFn->result, iter);
					break;
				case PARAVIEW:
					sprintf(autosaveFilename, "%sautosave_iter%05d.vti",
							inFn->result, iter);
					break;
				}

				tInstant1 = clock(); // Start measuring time
				WriteResults3D(autosaveFilename, nodeType, nodeX, nodeY, nodeZ,
						u, v, w, rho, nodeType, n, m, h, args->outputFormat);
				tInstant2 = clock();
				taskTime[T_WRIT] += (FLOAT_TYPE) (tInstant2 - tInstant1)
																																																																																												/ CLOCKS_PER_SEC;
			}
		}
	}     ////////////// END OF MAIN WHILE CYCLE! ///////////////

	tIterEnd = clock(); // End measuring time of main loop
	taskTime[T_ITER] = (FLOAT_TYPE) (tIterEnd - tIterStart) / CLOCKS_PER_SEC;

	clock_t tEnd = clock();
	taskTime[T_OALL] = (FLOAT_TYPE) (tEnd - tStart) / CLOCKS_PER_SEC; // Calculate elapsed time
	taskTime[T_COLL] /= 1000;
	taskTime[T_STRM] /= 1000;
	taskTime[T_BNDC] /= 1000;
	taskTime[T_MACR] /= 1000;
	taskTime[T_RESI] /= 1000;

	fclose(logFile);
	writeEndLog(logFilename, taskTime);
	writeTimerLog(timeFilename, taskTime);
	if (args->TypeOfResiduals != MacroDiff) {
		writeResiduals(residualsFilename, norm, dragSum, liftSum, m * n * h,
				args->iterations);
	}
	// Write final data
#if CUDA
	CHECK(hipMemcpy(u, u_d, SIZEFLT(m*n*h), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(v, v_d, SIZEFLT(m*n*h), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(w, w_d, SIZEFLT(m*n*h), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(rho, rho_d, SIZEFLT(m*n*h), hipMemcpyDeviceToHost));
	if(args->multiPhase){
		CHECK(hipMemcpy(r_rho, r_rho_d, SIZEFLT(m*n*h), hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(b_rho, b_rho_d, SIZEFLT(m*n*h), hipMemcpyDeviceToHost));
	}
#endif

	switch (args->outputFormat) {
	case CSV:
		sprintf(finalFilename, "%sFinalData.csv", inFn->result);
		break;
	case TECPLOT:
		sprintf(finalFilename, "%sFinalData.dat", inFn->result);
		break;
	case PARAVIEW:
		sprintf(finalFilename, "%sFinalData.vti", inFn->result);
		break;
	}
	if(args->multiPhase){
		FLOAT_TYPE *analytical = createHostArrayFlt(m, ARRAY_ZERO);
		switch (args->test_case) {
		case 1:
			printf("Suface tension error: "FLOAT_FORMAT"\n", st_error[iter-1]);
			WriteArray("surface tension",st_error, args->iterations,1);
			break;
		case 2:
			deformingBubbleValid(r_rho, b_rho, n, m, h);
			break;
		case 3:
			validateCoalescenceCase(r_rho, b_rho, n, m, args->bubble_radius, h);
			break;
		case 4: //COUETTE
			analyticalCouette(args->kappa, nodeY, m, n, analytical, args->u, h);
			writeCouetteSolution("Profile_Couette", analytical, u, nodeY, m, n, h);
			printf("Couette profile written to Profile_Couette in Results/\n");
			break;
		default:
			break;
		}
		WriteResultsMultiPhase(finalFilename, nodeType, nodeX, nodeY, nodeZ, u, v, w, rho,r_rho,b_rho, nodeType,
				n, m, h, args->outputFormat);
	}
	else{
		WriteResults3D(finalFilename, nodeType, nodeX, nodeY, nodeZ, u, v, w, rho,
				nodeType, n, m, h, args->outputFormat);
	}

	WriteLidDrivenCavityMidLines3D(nodeX, nodeY, nodeZ, u, w, n, m, h, args->u);
	WriteChannelCrossSection3D(nodeX, nodeY, nodeZ, u, v, w, n, m, h, args->u);

	// Write information for user
	printf("\n\nLog was written to %s\n", logFilename);
	printf("Last autosave result can be found at %s\n", autosaveFilename);
	printf("residuals were written to %s\n", residualsFilename);
	printf("Profiling results were written to %s\n", timeFilename);
	//	compareTestFiles("./TestValues/CUDA/cg2d.txt", "./TestValues/CUDA/cg3d.txt");
	printf("Final results were written to %s\n", finalFilename);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	freeAllHost(hostArrays, sizeof(hostArrays) / sizeof(hostArrays[0]));
	freeAllGpu(gpuArrays, sizeof(gpuArrays) / sizeof(gpuArrays[0]));
	if(args->multiPhase){
#if CUDA
		freeAllGpu(mpGpuArrays, sizeof(mpGpuArrays) / sizeof(mpGpuArrays[0]));
#else
		freeAllHost(mpHostArrays, sizeof(mpHostArrays) / sizeof(mpHostArrays[0]));
#endif
	}
	if (args->TypeOfResiduals == FdRelDiff) {
		freeAllGpu(FDdifGpuArrays, sizeof(FDdifGpuArrays) / sizeof(FDdifGpuArrays[0]));
	}
	if (args->TypeOfResiduals != MacroDiff) {
		freeAllGpu(nonMacroDiffGpuArrays, sizeof(nonMacroDiffGpuArrays) / sizeof(nonMacroDiffGpuArrays[0]));
	}

	return 0;
}
