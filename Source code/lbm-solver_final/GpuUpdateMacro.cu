#include "hip/hip_runtime.h"
#include "GpuFunctions.h"
#include "BcMacros.h"
#include "BcMacros3D.h"
#include "GpuConstants.h"

__global__ void gpuUpdateMacro2D(int *fluid_d, FLOAT_TYPE* rho_d,
		FLOAT_TYPE* u_d, FLOAT_TYPE* v_d, int *bcMask_d, FLOAT_TYPE* drag_d,
		FLOAT_TYPE* lift_d,
		FLOAT_TYPE* coordX_d, FLOAT_TYPE* coordY_d, FLOAT_TYPE* f_d) {
	int ind = threadIdx.x + blockIdx.x * blockDim.x;

	int ms = depth_d * length_d;

	FLOAT_TYPE r, u, v;

	if (ind < ms) {
		if (fluid_d[ind] == 1) {
			r = u = v = 0.0;
			r = f_d[ind] + f_d[ind + ms] + f_d[ind + 2 * ms] + f_d[ind + 3 * ms]
			                                                       + f_d[ind + 4 * ms] + f_d[ind + 5 * ms] + f_d[ind + 6 * ms]
			                                                                                                     + f_d[ind + 7 * ms] + f_d[ind + 8 * ms];
			u = f_d[ind + ms] - f_d[ind + 3 * ms] + f_d[ind + 5 * ms]
			                                            - f_d[ind + 6 * ms] - f_d[ind + 7 * ms] + f_d[ind + 8 * ms];
			v = f_d[ind + 2 * ms] - f_d[ind + 4 * ms] + f_d[ind + 5 * ms]
			                                                + f_d[ind + 6 * ms] - f_d[ind + 7 * ms] - f_d[ind + 8 * ms];

			rho_d[ind] = r;
			u_d[ind] = u / r;
			///@todo code: probably should handle outlet on other sides
			v_d[ind] = ((bcMask_d[ind] & BC_OUTL_E) == BC_OUTL_E) ? 0.0 : v / r;

			//   DRAG/LIFT FORCE
			if (dlBoundaryId_d
					!= 0&& (bcMask_d[ind] & BND_ID_ALL) == BOUND_ID(dlBoundaryId_d)) {
				// printf("draglift: %d\n",ind);
				drag_d[ind] = 0.33333333 * r * (20 - coordX_d[ind]) * 0.2;
				lift_d[ind] = 0.33333333 * r * (20 - coordY_d[ind]) * 0.2;
			}
		}
	}
}

__global__ void gpuUpdateMacro2DCG(int *fluid_d, FLOAT_TYPE* rho_d,
		FLOAT_TYPE* u_d, FLOAT_TYPE* v_d, FLOAT_TYPE* r_f_d, FLOAT_TYPE* b_f_d, FLOAT_TYPE* r_rho_d,
		FLOAT_TYPE* b_rho_d, FLOAT_TYPE *p_in_d, FLOAT_TYPE *p_out_d,
		int *num_in_d, int *num_out_d) {
	int ind = threadIdx.x + blockIdx.x * blockDim.x;

	int ms = depth_d * length_d;

	FLOAT_TYPE r_r, b_r, u, v, r, chi;

	if (ind < ms) {
		//necessary because of sum
		p_in_d[ind] = 0;
		p_out_d[ind] = 0;
		num_in_d[ind] = 0;
		num_out_d[ind] = 0;

		if (fluid_d[ind] == 1) {
			r_r = b_r = u = v = 0.0;

			r_r = r_f_d[ind] +
					r_f_d[ind + ms] +
					r_f_d[ind + 2 * ms] +
					r_f_d[ind + 3 * ms] +
					r_f_d[ind + 4 * ms] +
					r_f_d[ind + 5 * ms] +
					r_f_d[ind + 6 * ms]	+
					r_f_d[ind + 7 * ms] +
					r_f_d[ind + 8 * ms];
			b_r = b_f_d[ind] +
					b_f_d[ind + ms] +
					b_f_d[ind + 2 * ms] +
					b_f_d[ind + 3 * ms] +
					b_f_d[ind + 4 * ms] +
					b_f_d[ind + 5 * ms] +
					b_f_d[ind + 6 * ms] +
					b_f_d[ind + 7 * ms] +
					b_f_d[ind + 8 * ms];

			r_rho_d[ind] = r_r;
			b_rho_d[ind] = b_r;
			r = r_r + b_r;
			rho_d[ind] = r;

			u = (r_f_d[ind + ms] + b_f_d[ind + ms]) -
					(r_f_d[ind + 3 * ms] + b_f_d[ind + 3 * ms]) +
					(r_f_d[ind + 5 * ms] + b_f_d[ind + 5 * ms]) -
					(r_f_d[ind + 6 * ms] + b_f_d[ind + 6 * ms]) -
					(r_f_d[ind + 7 * ms] + b_f_d[ind + 7 * ms]) +
					(r_f_d[ind + 8 * ms] + b_f_d[ind + 8 * ms]);

			v = (r_f_d[ind + 2 * ms] + b_f_d[ind + 2 * ms]) -
					(r_f_d[ind + 4 * ms] + b_f_d[ind + 4 * ms]) +
					(r_f_d[ind + 5 * ms] + b_f_d[ind + 5 * ms]) +
					(r_f_d[ind + 6 * ms] + b_f_d[ind + 6 * ms]) -
					(r_f_d[ind + 7 * ms] + b_f_d[ind + 7 * ms]) -
					(r_f_d[ind + 8 * ms] + b_f_d[ind + 8 * ms]);

			u_d[ind] = u / r;

			v_d[ind] = v / r;

			// p_in and p_out for the surface tension
			chi=(r_r-b_r)/r;

			if (chi >= control_param_d){
				num_in_d[ind] = 1;
				p_in_d[ind] = r_r;
			}
			else if (chi <= -control_param_d){
				num_out_d[ind] = 1;
				p_out_d[ind] = b_r;
			}
		}
	}
}

__global__ void gpuUpdateMacro3D(int *fluid_d, FLOAT_TYPE* rho_d,
		FLOAT_TYPE* u_d, FLOAT_TYPE* v_d, FLOAT_TYPE* w_d, int* bcBoundId_d,
		FLOAT_TYPE* coordX_d, FLOAT_TYPE* coordY_d, FLOAT_TYPE* coordZ_d,
		FLOAT_TYPE* f_d, FLOAT_TYPE g, unsigned long long *bcMask_d,int updateInltOutl) //	 FLOAT_TYPE* drag_d, FLOAT_TYPE* lift_d, FLOAT_TYPE* latF_d,
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int ind = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x)
															+ threadIdx.x;
	int ms = depth_d * length_d * height_d;

	FLOAT_TYPE r, rU, rV, rW;

	if (ind < ms) {

		//    printf("bcMask[ind] |= BC3D_MASK((unsigned long long)bcType[bci], dir); %#016lX\n", (bcMask_d[bci] & BC3D_FLUID) );
		if (fluid_d[ind] == 1
				&& (!(((bcMask_d[ind] & BC3D_OUTL_B) ==BC3D_INLT_B))||updateInltOutl)) {
			//((bcMask_d[ind] & BC3D_OUTL_B) == BC3D_OUTL_B)||
			r = rU = rV = rW = 0.0;
			r = f_d[ind] + f_d[ind + ms] + f_d[ind + 2 * ms] + f_d[ind + 3 * ms]
			                                                       + f_d[ind + 4 * ms] + f_d[ind + 5 * ms] + f_d[ind + 6 * ms]
			                                                                                                     + f_d[ind + 7 * ms] + f_d[ind + 8 * ms] + f_d[ind + 9 * ms]
			                                                                                                                                                   + f_d[ind + 10 * ms] + f_d[ind + 11 * ms]
			                                                                                                                                                                              + f_d[ind + 12 * ms] + f_d[ind + 13 * ms]
			                                                                                                                                                                                                         + f_d[ind + 14 * ms] + f_d[ind + 15 * ms]
			                                                                                                                                                                                                                                    + f_d[ind + 16 * ms] + f_d[ind + 17 * ms]
			                                                                                                                                                                                                                                                               + f_d[ind + 18 * ms];
			//                        if(ind==1) printf("i am here BOTTOM WEST_1 r= %.14f f0: %.14f f1: %.14f f2: %.14f f3: %.14f f4: %.14f f5: %.14f f6: %.14f f7: %.14f f8: %.14f "
			//                        		"f9: %.14f f10: %.14f f11: %.14f f12: %.14f f3: %.14f f4: %.14f f15: %.14f f16: %.14f f17: %.14f f18: %.14f\n",
			//                        		r, f_d[ind+ 0*ms],f_d[ind+ 1*ms],f_d[ind+ 2*ms],f_d[ind+ 3*ms],f_d[ind+ 4*ms],f_d[ind+ 5*ms],f_d[ind+ 6*ms],f_d[ind+ 7*ms], f_d[ind+ 8*ms], f_d[ind+ 9*ms],
			//                        		f_d[ind+10*ms],f_d[ind+11*ms],f_d[ind+12*ms],f_d[ind+13*ms],f_d[ind+14*ms],f_d[ind+15*ms], f_d[ind+16*ms], f_d[ind+17*ms], f_d[ind+18*ms]);

			rU = f_d[ind + ms] - f_d[ind + 2 * ms] + f_d[ind + 7 * ms]
			                                             - f_d[ind + 8 * ms] + f_d[ind + 9 * ms] - f_d[ind + 10 * ms]
			                                                                                           + f_d[ind + 11 * ms] - f_d[ind + 12 * ms]
			                                                                                                                      + f_d[ind + 13 * ms] - f_d[ind + 14 * ms];
			//            if(ind==1) printf("i am here TOP lid u= %.14f f1: %.14f f2: %.14f f7: %.14f f8: %.14f "
			//                        		"f9: %.14f f10: %.14f f11: %.14f f12: %.14f f3: %.14f f4: %.14f  \n",
			//                        		rU/r, f_d[ind+ 1*ms],f_d[ind+ 2*ms],f_d[ind+ 7*ms], f_d[ind+ 8*ms], f_d[ind+ 9*ms],
			//                        		f_d[ind+10*ms],f_d[ind+11*ms],f_d[ind+12*ms],f_d[ind+13*ms],f_d[ind+14*ms]);
			rV = f_d[ind + 3 * ms] - f_d[ind + 4 * ms] + f_d[ind + 7 * ms]
			                                                 + f_d[ind + 8 * ms] - f_d[ind + 9 * ms] - f_d[ind + 10 * ms]
			                                                                                               + f_d[ind + 15 * ms] - f_d[ind + 16 * ms]
			                                                                                                                          + f_d[ind + 17 * ms] - f_d[ind + 18 * ms];
			//
			//            			if(ind==30) printf("i am here BOTTOM WEST_1 v= %.14f f3: %.14f f4: %.14f f7: %.14f f8: %.14f f9: %.14f f10: %.14f f15: %.14f f16: %.14f f17: %.14f f18: %.14f\n",rV/r, f_d[ind+ 3*ms],f_d[ind+ 4*ms],f_d[ind+ 7*ms], f_d[ind+ 8*ms], f_d[ind+ 9*ms], f_d[ind+10*ms],
			//            	            	 f_d[ind+15*ms], f_d[ind+16*ms], f_d[ind+17*ms], f_d[ind+18*ms]);
			rW = f_d[ind + 5 * ms] - f_d[ind + 6 * ms] + f_d[ind + 11 * ms]
			                                                 + f_d[ind + 12 * ms] - f_d[ind + 13 * ms]
			                                                                            - f_d[ind + 14 * ms] + f_d[ind + 15 * ms]
			                                                                                                       + f_d[ind + 16 * ms] - f_d[ind + 17 * ms]
			                                                                                                                                  - f_d[ind + 18 * ms];
			rho_d[ind] = r;
			u_d[ind] = rU / r + g / (omega_d);
			;
			v_d[ind] = rV / r;
			w_d[ind] = rW / r;
			///@todo code: probably should handle outlet on other sides
			//			v_d[ind] =
			//					((bcMask_d[ind] & BC3D_OUTL_1) == BC3D_OUTL_1) ?
			//							0.0 : rV / r;
			//			w_d[ind] =
			//					((bcMask_d[ind] & BC3D_OUTL_1) == BC3D_OUTL_1) ?
			//							0.0 : rW / r;
			//            if(ind==1 )printf(" macro %d %f   %f   %f\n",ind, u_d[ind],v_d[ind],w_d[ind]);

			//            if(u_d[ind]>0.0 | u_d[ind]<0.0) {printf("ind: %d, x: %lf y %lf z %lf\n", ind, coordX_d[ind],coordY_d[ind],coordZ_d[ind]); return ;}
			//			if(u_d[ind]>0.0 | u_d[ind]<0.0) {printf("ind: %d, x: %lf y %lf z %.14f\n", ind, coordX_d[ind],coordY_d[ind],coordZ_d[ind]); return ;}
			//			if(u_d[ind]>0.0 | u_d[ind]<0.0) {printf("ind: %d, x: %lf y %lf z %lf\n", ind, coordX_d[ind],coordY_d[ind],coordZ_d[ind]); return ;}

			//   DRAG/LIFT/LATERAL FORCES TODO: find reference and check

			//			if (dlBoundaryId_d != 0 && bcBoundId_d[ind] == dlBoundaryId_d) {
			// printf("draglift: %d\n",ind);
			//                drag_d[ind] = 0.33333333*r*(20-coordX_d[ind])*0.2;
			//                lift_d[ind] = 0.33333333*r*(20-coordZ_d[ind])*0.2;
			//                latF_d[ind] = 0.33333333*r*(20-coordY_d[ind])*0.2;
			//			}
		}
	}

}
//            if(ind==1) printf("i am here BOTTOM WEST_1 r= %.14f f0: %.14f f1: %.14f f2: %.14f f3: %.14f f4: %.14f f5: %.14f f6: %.14f f7: %.14f f8: %.14f "
//            		"f9: %.14f f10: %.14f f11: %.14f f12: %.14f f3: %.14f f4: %.14f f15: %.14f f16: %.14f f17: %.14f f18: %.14f\n",
//            		r, f_d[ind+ 0*ms],f_d[ind+ 1*ms],f_d[ind+ 2*ms],f_d[ind+ 3*ms],f_d[ind+ 4*ms],f_d[ind+ 5*ms],f_d[ind+ 6*ms],f_d[ind+ 7*ms], f_d[ind+ 8*ms], f_d[ind+ 9*ms],
//            		f_d[ind+10*ms],f_d[ind+11*ms],f_d[ind+12*ms],f_d[ind+13*ms],f_d[ind+14*ms],f_d[ind+15*ms], f_d[ind+16*ms], f_d[ind+17*ms], f_d[ind+18*ms]);
//			if(ind==30) printf("i am here BOTTOM WEST_1 v= %.14f f3: %.14f f4: %.14f f7: %.14f f8: %.14f f9: %.14f f10: %.14f f15: %.14f f16: %.14f f17: %.14f f18: %.14f\n",rV/r, f_d[ind+ 3*ms],f_d[ind+ 4*ms],f_d[ind+ 7*ms], f_d[ind+ 8*ms], f_d[ind+ 9*ms], f_d[ind+10*ms],
//	            	 f_d[ind+15*ms], f_d[ind+16*ms], f_d[ind+17*ms], f_d[ind+18*ms]); 26739
//           if(ind==3) printf("i am here BOTTOM South_1 rW= %.14f f5: %.14f f6: %.14f f11: %.14f f12: %.14f f13: %.14f f14: %.14f f15: %.14f "
//            					"f16: %.14f f17: %.14f f18: %.14f\n",rW, f_d[ind+ 5*ms], f_d[ind+ 6*ms], f_d[ind+11*ms], f_d[ind+12*ms], f_d[ind+13*ms], f_d[ind+14*ms],
//            	            	 f_d[ind+15*ms], f_d[ind+16*ms], f_d[ind+17*ms], f_d[ind+18*ms] );
