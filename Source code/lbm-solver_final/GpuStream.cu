#include "hip/hip_runtime.h"
#include <stdio.h>

#include "GpuFunctions.h"
#include "BcMacros.h"
#include "BcMacros3D.h"
#include "GpuConstants.h"

__global__ void gpuStreaming2D(int* fluid_d, int* stream_d, FLOAT_TYPE* f_d, FLOAT_TYPE* fColl_d)
{
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	int ms = depth_d*length_d;
	FLOAT_TYPE *f, *mf;
	int n = length_d;
	if (ind < ms && fluid_d[ind] == 1)
	{
		f_d[ind] = fColl_d[ind];	//Update fNewStep = fColl
		f = f_d + ms;				// f is f_d memory positions but f starts in f_d 1st level==1st lattice direction
		mf = fColl_d + ms;
//		f[ind]      = (stream_d[ind]      == 1) ? mf[ind-1]        : mf[ind];		// stream_d == 1 means that
//		f[ind+ms]   = (stream_d[ind+ms]   == 1) ? mf[ind+ms-n]     : mf[ind+ms]; 	// the streaming is allowed
//		f[ind+2*ms] = (stream_d[ind+2*ms] == 1) ? mf[ind+2*ms+1]   : mf[ind+2*ms];  // "the regular case"
//		f[ind+3*ms] = (stream_d[ind+3*ms] == 1) ? mf[ind+3*ms+n]   : mf[ind+3*ms];  // stream_d != 1 means
//		f[ind+4*ms] = (stream_d[ind+4*ms] == 1) ? mf[ind+4*ms-n-1] : mf[ind+4*ms]; 	// wall or node outside dom.
//		f[ind+5*ms] = (stream_d[ind+5*ms] == 1) ? mf[ind+5*ms-n+1] : mf[ind+5*ms];
//		f[ind+6*ms] = (stream_d[ind+6*ms] == 1) ? mf[ind+6*ms+n+1] : mf[ind+6*ms];
//		f[ind+7*ms] = (stream_d[ind+7*ms] == 1) ? mf[ind+7*ms+n-1] : mf[ind+7*ms];

		//ASK ANTONIO

		f[ind]      = (stream_d[ind]      == 1) ? mf[ind-1]        : f[ind];		// stream_d == 1 means that
		f[ind+ms]   = (stream_d[ind+ms]   == 1) ? mf[ind+ms-n]     : f[ind+ms]; 	// the streaming is allowed
		f[ind+2*ms] = (stream_d[ind+2*ms] == 1) ? mf[ind+2*ms+1]   : f[ind+2*ms];  // "the regular case"
		f[ind+3*ms] = (stream_d[ind+3*ms] == 1) ? mf[ind+3*ms+n]   : f[ind+3*ms];  // stream_d != 1 means
		f[ind+4*ms] = (stream_d[ind+4*ms] == 1) ? mf[ind+4*ms-n-1] : f[ind+4*ms]; 	// wall or node outside dom.
		f[ind+5*ms] = (stream_d[ind+5*ms] == 1) ? mf[ind+5*ms-n+1] : f[ind+5*ms];
		f[ind+6*ms] = (stream_d[ind+6*ms] == 1) ? mf[ind+6*ms+n+1] : f[ind+6*ms];
		f[ind+7*ms] = (stream_d[ind+7*ms] == 1) ? mf[ind+7*ms+n-1] : f[ind+7*ms];

	}
}

__global__ void gpuStreaming3D(int* fluid_d, bool* stream_d, FLOAT_TYPE* f_d, FLOAT_TYPE* fColl_d)
{
	int blockId = blockIdx.x
			+ blockIdx.y * gridDim.x;
	int ind =  blockId * (blockDim.x * blockDim.y)
				+ (threadIdx.y * blockDim.x)
				+ threadIdx.x;

	int ms = depth_d*length_d*height_d;
	FLOAT_TYPE *f, *mf;
	if (ind < ms && fluid_d[ind] == 1)
	{
		f_d[ind] = fColl_d[ind];	//Update fNewStep = fColl
		f = f_d + ms;				// f is f_d memory position but f starts in f_d 1st level==1st lattice direction
		mf = fColl_d + ms;
		f[ind+0  *ms]	=	(stream_d[ind+0	 *ms]	==	1)	?	mf[ind+0  *ms +	c3D_d[1	]]:	mf[ind+0  *ms];
		f[ind+1	 *ms]	=	(stream_d[ind+1	 *ms]	==	1)	?	mf[ind+1  *ms +	c3D_d[2	]]:	mf[ind+1  *ms];
		f[ind+2	 *ms]	=	(stream_d[ind+2	 *ms]	==	1)	?	mf[ind+2  *ms +	c3D_d[3	]]:	mf[ind+2  *ms];
		f[ind+3	 *ms]	=	(stream_d[ind+3	 *ms]	==	1)	?	mf[ind+3  *ms +	c3D_d[4	]]:	mf[ind+3  *ms];
		f[ind+4	 *ms]	=	(stream_d[ind+4	 *ms]	==	1)	?	mf[ind+4  *ms +	c3D_d[5	]]:	mf[ind+4  *ms];
		f[ind+5	 *ms]	=	(stream_d[ind+5	 *ms]	==	1)	?	mf[ind+5  *ms +	c3D_d[6	]]:	mf[ind+5  *ms];
		f[ind+6	 *ms]	=	(stream_d[ind+6	 *ms]	==	1)	?	mf[ind+6  *ms +	c3D_d[7	]]:	mf[ind+6  *ms];
		f[ind+7	 *ms]	=	(stream_d[ind+7	 *ms]	==	1)	?	mf[ind+7  *ms +	c3D_d[8	]]:	mf[ind+7  *ms];
		f[ind+8	 *ms]	=	(stream_d[ind+8	 *ms]	==	1)	?	mf[ind+8  *ms +	c3D_d[9	]]:	mf[ind+8  *ms];
		f[ind+9	 *ms]	=	(stream_d[ind+9	 *ms]	==	1)	?	mf[ind+9  *ms +	c3D_d[10]]:	mf[ind+9  *ms];
		f[ind+10 *ms]	=	(stream_d[ind+10 *ms]	==	1)	?	mf[ind+10 *ms +	c3D_d[11]]:	mf[ind+10 *ms];
		f[ind+11 *ms]	=	(stream_d[ind+11 *ms]	==	1)	?	mf[ind+11 *ms +	c3D_d[12]]:	mf[ind+11 *ms];
		f[ind+12 *ms]	=	(stream_d[ind+12 *ms]	==	1)	?	mf[ind+12 *ms +	c3D_d[13]]:	mf[ind+12 *ms];
		f[ind+13 *ms]	=	(stream_d[ind+13 *ms]	==	1)	?	mf[ind+13 *ms +	c3D_d[14]]:	mf[ind+13 *ms];
		f[ind+14 *ms]	=	(stream_d[ind+14 *ms]	==	1)	?	mf[ind+14 *ms +	c3D_d[15]]:	mf[ind+14 *ms];
		f[ind+15 *ms]	=	(stream_d[ind+15 *ms]	==	1)	?	mf[ind+15 *ms +	c3D_d[16]]:	mf[ind+15 *ms];
		f[ind+16 *ms]	=	(stream_d[ind+16 *ms]	==	1)	?	mf[ind+16 *ms +	c3D_d[17]]:	mf[ind+16 *ms];
		f[ind+17 *ms]	=	(stream_d[ind+17 *ms]	==	1)	?	mf[ind+17 *ms +	c3D_d[18]]:	mf[ind+17 *ms];
	}
}
