#include "hip/hip_runtime.h"
#include "Multiphase.h"
#include <math.h>
#include <stdio.h>
void mp2DColl(int *fluid, FLOAT_TYPE *rho, FLOAT_TYPE *u,
		FLOAT_TYPE *v, FLOAT_TYPE *f, FLOAT_TYPE *fColl){

}

void createBubble(float *x, float *y,int n, int m, FLOAT_TYPE radius, FLOAT_TYPE *r_f, FLOAT_TYPE *b_f, FLOAT_TYPE *r_rho, FLOAT_TYPE *b_rho,
		FLOAT_TYPE r_density, FLOAT_TYPE b_density, FLOAT_TYPE *r_phi, FLOAT_TYPE *b_phi, FLOAT_TYPE *rho) {
	int i, j, k;
	int index, index2;
	for(i = 0; i < n; i++){
		for (j=0; j < m; j++){
			index = i*m + j;
			//printf("pow1: %f  --  pow2: %f  --  sqrt: %f\n",pow((x[index]-0.5), 2),pow((y[index]-0.5),2), sqrt( pow((x[index]-0.5), 2) + pow((y[index]-0.5),2)));
			if( sqrt( pow((x[index]-0.5), 2) + pow((y[index]-0.5),2)) <= radius ){
				for (k=0; k < 9; k++){
					// initialise distribution function with small, non-zero values
					r_rho[index] = r_density;
					//x + WIDTH * (y + DEPTH * z)
					index2 = k + index * 9;
					r_f[index2] = r_rho[index] * r_phi[k];
				}
				//printf("JSAHBDJSABDSHBCJHABSHDBSAHDBASHBDSHABDHSABHDBSAHDBASBDSHABDA\n");
			}
			else {
				for (k=0; k < 9; k++){
					// initialise distribution function with small, non-zero values
					b_rho[index]=b_density;
					index2 = k + index * 9;
					b_f[index2]   = b_rho[index]*b_phi[k];
				}
			}
			// initialise density
			rho[index] = r_rho[index]+b_rho[index];

			printf("%f\n", r_rho[i*m + j]);
		}
	}
}
