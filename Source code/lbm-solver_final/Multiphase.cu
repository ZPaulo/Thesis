#include "hip/hip_runtime.h"
#include "Multiphase.h"
#include <math.h>
#include <stdio.h>
#include "ArrayUtils.h"

void mp2DColl(int n, int m, FLOAT_TYPE *rho, FLOAT_TYPE *u,
		FLOAT_TYPE *v, FLOAT_TYPE *r_f, FLOAT_TYPE *b_f, FLOAT_TYPE *r_rho, FLOAT_TYPE *b_rho,
		FLOAT_TYPE *r_phi, FLOAT_TYPE *b_phi, FLOAT_TYPE *w_pert, FLOAT_TYPE *color_gradient,
		FLOAT_TYPE r_omega, FLOAT_TYPE b_omega, FLOAT_TYPE control_param, FLOAT_TYPE del,
		FLOAT_TYPE beta, FLOAT_TYPE g_limit,  FLOAT_TYPE r_A,  FLOAT_TYPE b_A, FLOAT_TYPE *r_fPert, FLOAT_TYPE *b_fPert,
		FLOAT_TYPE *weight, int *cx, int *cy){

	FLOAT_TYPE cu1, cu2, r_CollPert, b_CollPert;
	FLOAT_TYPE cosin;
	FLOAT_TYPE chi;
	FLOAT_TYPE r_omega_temp, b_omega_temp;
	FLOAT_TYPE a1      =   2.0 * r_omega * b_omega/(r_omega+b_omega);
	FLOAT_TYPE a2      =   2.0 * (r_omega - a1) / del;
	FLOAT_TYPE a3      =   -a2 / (2.0 * del);
	FLOAT_TYPE a4      =   2.0 * (a1 - b_omega) / del;
	FLOAT_TYPE a5      =   a4 / (2.0 * del);
	FLOAT_TYPE color_gradient_norm;
	FLOAT_TYPE k_r, k_b, k_k;
	FLOAT_TYPE norm_c;
	FLOAT_TYPE prod_c_g;
	FLOAT_TYPE r_pert, b_pert;
	FLOAT_TYPE r_feq, b_feq;
	FLOAT_TYPE fn05;
	FLOAT_TYPE cg_w[9] = {0.0, 4.0/12.0, 4.0/12.0, 4.0/12.0, 4.0/12.0, 1.0/12.0, 1.0/12.0, 1.0/12.0, 1.0/12.0};
	int index, index9, temp_index;
	for (int j=0; j < m; j++){
		for (int i=0;i < n; i++){
			index = j*n + i;


			color_gradient[index * 2] = 0;
			color_gradient[index * 2 + 1] = 0;
			for (int k=0; k<9; k++){

				// calculate color gradient - 4th order

				if (i!=0 && j!=0 && i!=(n-1) && j!=(m-1)){ // Interior points - In the boundary it is calculated by "mirroring" the density
					temp_index = (j + cy[k]) * n + i + cx[k];
					color_gradient[index * 2] += (r_rho[temp_index] - b_rho[temp_index]) * cx[k] * cg_w[k];
					color_gradient[index * 2 + 1] += (r_rho[temp_index] - b_rho[temp_index]) * cy[k] * cg_w[k];
				}
				else if (j==(m-1) && i!=0 && i!=(n-1)) {// north boundary
					temp_index = (j - abs(cy[k])) * n + i + cx[k];
					color_gradient[index * 2] += (r_rho[temp_index] - b_rho[temp_index]) * cx[k] * cg_w[k];
					color_gradient[index * 2 + 1] = 0;
				}
				else if (j==0 && i!=0 && i!=(n-1)){  // south boundary
					temp_index = (j + abs(cy[k])) * n + i + cx[k];
					color_gradient[index * 2] += (r_rho[temp_index] - b_rho[temp_index]) * cx[k] * cg_w[k];
					color_gradient[index * 2 + 1] = 0;
				}
				else if (i==(n-1) && j!=0 && j!=(m-1)){  // east boundary
					temp_index = (j + cy[k]) * n + i - abs(cx[k]);
					color_gradient[index * 2] = 0;
					color_gradient[index * 2 + 1] +=  (r_rho[temp_index] - b_rho[temp_index]) * cy[k] * cg_w[k];
				}
				else if (i==0 && j!=0 && j!=(m-1)){ //  west boundary
					temp_index = (j + cy[k]) * n + i + abs(cx[k]);
					color_gradient[index * 2] = 0;
					color_gradient[index * 2 + 1] +=  (r_rho[temp_index] - b_rho[temp_index]) * cy[k] * cg_w[k];
				}
			}

			// relaxation parameter to choose a proper omega at the interface
			if (r_omega != b_omega){
				chi=(r_rho[index] - b_rho[index])/rho[index];
				if(chi >= -control_param && chi <= control_param){
					if (chi > del)
						r_omega_temp=r_omega;
					else if (chi <= del && chi > 0)
						r_omega_temp=a1 + a2 * chi + a3 * chi * chi;
					else if (chi <= 0 && chi >= -del)
						r_omega_temp=a1 + a4 * chi + a5 * chi * chi;
					else if (chi < -del)
						r_omega_temp=b_omega;
				}
			}
			else
				r_omega_temp=r_omega;

			b_omega_temp=r_omega_temp;

			cu1 = u[index]*u[index] + v[index]*v[index];

			// invariable quantities
			color_gradient_norm = sqrt(pow(color_gradient[index * 2],2) + pow(color_gradient[index * 2 + 1],2));
			k_r=r_rho[index]/rho[index];
			k_b=b_rho[index]/rho[index];
			k_k= beta * r_rho[index] * b_rho[index]/(pow(rho[index],2));
			for (int k=0;k<9;k++){
				if (color_gradient_norm > g_limit){
					prod_c_g=cx[k]*color_gradient[index * 2]+cy[k]*color_gradient[index * 2 + 1];
					if (k!=0){
						norm_c= sqrt(pow(cx[k],2)+pow(cy[k],2));
						cosin= prod_c_g / (color_gradient_norm*norm_c);
					}
					else
						cosin=0.0;
					// calculate perturbation terms

					r_pert=0.5*r_A*color_gradient_norm*(weight[k]*pow((prod_c_g/color_gradient_norm),2)-w_pert[k]);
					b_pert=0.5*b_A*color_gradient_norm*(weight[k]*pow((prod_c_g/color_gradient_norm),2)-w_pert[k]);

				}
				else{
					// ther perturbation terms are null
					r_pert=0.0;
					b_pert=0.0;
				}


				cu2 = u[index]*cx[k] + v[index]*cy[k];
				// calculate equilibrium distribution function
				r_feq = r_rho[index] * (r_phi[k] + weight[k] * (3 * cu2 + 4.5 * cu2 * cu2 - 1.5 * cu1));
				b_feq = b_rho[index] * (b_phi[k] + weight[k] * (3 * cu2 + 4.5 * cu2 * cu2 - 1.5 * cu1));

				index9 = i + j * n + k * m * n;
				// calculate updated distribution function
				r_CollPert = r_omega_temp*r_feq + (1-r_omega_temp)*r_f[index9]+r_pert;
				b_CollPert = b_omega_temp*b_feq + (1-b_omega_temp)*b_f[index9]+b_pert;

				fn05 = r_CollPert + b_CollPert;
				//				// perform recolor step
				r_fPert[index9]=k_r*fn05+k_k*cosin*(r_rho[index]*r_phi[k]+b_rho[index]*b_phi[k]);
				b_fPert[index9]=k_b*fn05-k_k*cosin*(r_rho[index]*r_phi[k]+b_rho[index]*b_phi[k]);
			}
		}
	}
}

void mp3DColl(int n, int m, int h, FLOAT_TYPE *rho, FLOAT_TYPE *u,
		FLOAT_TYPE *v, FLOAT_TYPE *w, FLOAT_TYPE *r_rho, FLOAT_TYPE *b_rho,
		FLOAT_TYPE *w_pert, FLOAT_TYPE *color_gradient,
		FLOAT_TYPE beta, FLOAT_TYPE g_limit,  FLOAT_TYPE A, FLOAT_TYPE *r_fColl, FLOAT_TYPE *b_fColl,
		FLOAT_TYPE *weight, int *cx, int *cy, int *cz, FLOAT_TYPE *f, FLOAT_TYPE r_nu, FLOAT_TYPE b_nu, FLOAT_TYPE r_alpha,
		FLOAT_TYPE b_alpha, FLOAT_TYPE *chi, FLOAT_TYPE *phi, FLOAT_TYPE *psi, FLOAT_TYPE *teta, FLOAT_TYPE *cg_w){

	FLOAT_TYPE cu1, cu2, f_CollPert;
	FLOAT_TYPE cosin;
	FLOAT_TYPE color_gradient_norm;
	FLOAT_TYPE k_r, k_b, k_k;
	FLOAT_TYPE norm_c;
	FLOAT_TYPE prod_c_g;
	FLOAT_TYPE pert;
	FLOAT_TYPE f_eq;
	int index, index9, temp_index;
	FLOAT_TYPE grad_rho_x, grad_rho_y, grad_rho_z;
	FLOAT_TYPE G[9];
	FLOAT_TYPE H[9];
	FLOAT_TYPE prod_u_grad_rho, aux1, mean_nu, omega_eff, mean_alpha, TC;

	for(int k = 0; k < h; k++){
		for (int j=0; j < m; j++){
			for (int i=0;i < n; i++){
				index = k*n*m + j*n + i;

				color_gradient[index * 3] = 0;
				color_gradient[index * 3 + 1] = 0;
				color_gradient[index * 3 + 2] = 0;
				grad_rho_x = 0.0;
				grad_rho_y = 0.0;
				grad_rho_z = 0.0;
				for (int dir=0; dir < 19; dir++){

					// calculate color gradient - 4th order

					if (i!=0 && j!=0 && k != 0 && i!=(n-1) && j!=(m-1) && k != (h-1)){ // Interior points - In the boundary it is calculated by "mirroring" the density
						temp_index = (k + cz[dir]) * n * m + (j + cy[dir]) * n + i + cx[dir];
						grad_rho_x += rho[temp_index] * cx[dir] * cg_w[dir];
						grad_rho_y += rho[temp_index] * cy[dir] * cg_w[dir];
						grad_rho_z += rho[temp_index] * cz[dir] * cg_w[dir];
						color_gradient[index * 3] += ((r_rho[temp_index] - b_rho[temp_index]) / rho[temp_index]) * cx[dir] * cg_w[dir];
						color_gradient[index * 3 + 1] += ((r_rho[temp_index] - b_rho[temp_index]) / rho[temp_index]) * cy[dir] * cg_w[dir];
						color_gradient[index * 3 + 2] += ((r_rho[temp_index] - b_rho[temp_index]) / rho[temp_index]) * cz[dir] * cg_w[dir];
					}
					else if (j==(m-1) && i!=0 && i!=(n-1)) {// north boundary
						temp_index = (k + cz[dir]) * n * m + (j - abs(cy[dir])) * n + i + cx[dir];
						grad_rho_x += rho[temp_index] * cx[dir] * cg_w[dir];
						grad_rho_y = 0;
						grad_rho_z += rho[temp_index] * cz[dir] * cg_w[dir];
						color_gradient[index * 3] += ((r_rho[temp_index] - b_rho[temp_index]) / rho[temp_index]) * cx[dir] * cg_w[dir];
						color_gradient[index * 3 + 1] = 0;
						color_gradient[index * 3 + 2] += ((r_rho[temp_index] - b_rho[temp_index]) / rho[temp_index]) * cz[dir] * cg_w[dir];
					}
					else if (j==0 && i!=0 && i!=(n-1)){  // south boundary
						temp_index = (k + cz[dir]) * n * m + (j + abs(cy[dir])) * n + i + cx[dir];
						grad_rho_x += rho[temp_index] * cx[dir] * cg_w[dir];
						grad_rho_y = 0;
						grad_rho_z += rho[temp_index] * cz[dir] * cg_w[dir];
						color_gradient[index * 3] += ((r_rho[temp_index] - b_rho[temp_index]) / rho[temp_index]) * cx[dir] * cg_w[dir];
						color_gradient[index * 3 + 1] = 0;
						color_gradient[index * 3 + 2] += ((r_rho[temp_index] - b_rho[temp_index]) / rho[temp_index]) * cz[dir] * cg_w[dir];
					}
					else if (i==(n-1) && j!=0 && j!=(m-1)){  // east boundary
						temp_index = (k + cz[dir]) * n * m + (j + cy[dir]) * n + i - abs(cx[dir]);
						grad_rho_x = 0;
						grad_rho_y += rho[temp_index] * cy[dir] * cg_w[dir];
						grad_rho_z += rho[temp_index] * cz[dir] * cg_w[dir];
						color_gradient[index * 3] = 0;
						color_gradient[index * 3 + 1] +=  ((r_rho[temp_index] - b_rho[temp_index]) / rho[temp_index]) * cy[dir] * cg_w[dir];
						color_gradient[index * 3 + 2] += ((r_rho[temp_index] - b_rho[temp_index]) / rho[temp_index]) * cz[dir] * cg_w[dir];
					}
					else if (i==0 && j!=0 && j!=(m-1)){ //  west boundary
						temp_index = (k + cz[dir]) * n * m + (j + cy[dir]) * n + i + abs(cx[dir]);
						grad_rho_x = 0;
						grad_rho_y += rho[temp_index] * cy[dir] * cg_w[dir];
						grad_rho_z += rho[temp_index] * cz[dir] * cg_w[dir];
						color_gradient[index * 3] = 0;
						color_gradient[index * 3 + 1] +=  ((r_rho[temp_index] - b_rho[temp_index]) / rho[temp_index]) * cy[dir] * cg_w[dir];
						color_gradient[index * 3 + 2] += ((r_rho[temp_index] - b_rho[temp_index]) / rho[temp_index]) * cz[dir] * cg_w[dir];
					}
					else if (k==(h-1) && j!=0 && j!=(m-1)){ //  front boundary
						temp_index = (k - abs(cz[dir])) * n * m + (j + cy[dir]) * n + i + abs(cx[dir]);
						grad_rho_x += rho[temp_index] * cx[dir] * cg_w[dir];
						grad_rho_y += rho[temp_index] * cy[dir] * cg_w[dir];
						grad_rho_z = 0;
						color_gradient[index * 3] += ((r_rho[temp_index] - b_rho[temp_index]) / rho[temp_index]) * cx[dir] * cg_w[dir];
						color_gradient[index * 3 + 1] +=  ((r_rho[temp_index] - b_rho[temp_index]) / rho[temp_index]) * cy[dir] * cg_w[dir];
						color_gradient[index * 3 + 2] = 0;
					}
					else if (k==0 && j!=0 && j!=(m-1)){ //  back boundary
						temp_index = (k + abs(cz[dir])) * n * m + (j + cy[dir]) * n + i + abs(cx[dir]);
						grad_rho_x += rho[temp_index] * cx[dir] * cg_w[dir];
						grad_rho_y += rho[temp_index] * cy[dir] * cg_w[dir];
						grad_rho_z = 0;
						color_gradient[index * 3] += ((r_rho[temp_index] - b_rho[temp_index]) / rho[temp_index]) * cx[dir] * cg_w[dir];
						color_gradient[index * 3 + 1] +=  ((r_rho[temp_index] - b_rho[temp_index]) / rho[temp_index]) * cy[dir] * cg_w[dir];
						color_gradient[index * 3 + 2] = 0;
					}
				}



				G[0] = 2.0 * u[index] * grad_rho_x;
				G[1] = u[index]*grad_rho_y + v[index]*grad_rho_x;
				G[2] = u[index]*grad_rho_z + w[index]*grad_rho_x;
				G[3] = G[1];
				G[4] = 2.0*v[index]*grad_rho_y;
				G[5] = v[index]*grad_rho_z + w[index]*grad_rho_y;
				G[6] = G[2];
				G[7] = G[5];
				G[8] = 2.0*w[index]*grad_rho_z;

				prod_u_grad_rho = u[index]*grad_rho_x + v[index]*grad_rho_y + w[index]*grad_rho_z;

				cu1 = u[index]*u[index] + v[index]*v[index] + w[index] * w[index];

				// invariable quantities
				color_gradient_norm = sqrt(pow(color_gradient[index * 3],2) + pow(color_gradient[index * 3 + 1],2) + pow(color_gradient[index * 3 + 2],2));
				k_r=r_rho[index]/rho[index];
				k_b=b_rho[index]/rho[index];
				k_k= beta * r_rho[index] * b_rho[index]/rho[index];

				aux1 = r_rho[index]/(rho[index]*r_nu) + b_rho[index]/(rho[index]*b_nu);
				mean_nu = 1.0/aux1;

				omega_eff = 1.0/(3.0*mean_nu+0.5);

				mean_alpha = r_alpha*r_rho[index]/rho[index] + b_alpha*b_rho[index]/rho[index];

				for (int dir=0;dir<19;dir++){

					if (color_gradient_norm > g_limit){
						prod_c_g=cx[dir]*color_gradient[index * 3]+cy[dir]*color_gradient[index * 3 + 1] + cz[dir] * color_gradient[index * 3 + 2];
						if (dir!=0){
							norm_c= sqrt(pow(cx[dir],2)+pow(cy[dir],2) + pow(cz[dir],2));
							cosin= prod_c_g / (color_gradient_norm*norm_c);
						}
						else
							cosin=0.0;
						// calculate perturbation terms

						pert=0.5*A*color_gradient_norm*(weight[dir]*pow((prod_c_g/color_gradient_norm),2)-w_pert[dir]);

					}
					else{
						// ther perturbation terms are null
						pert = 0.0;
					}
					// Auxiliar tensor: diadic product of the speed velcity:
					//[cx,cy,cx]*[cx cy cz]
					H[0] = cx[dir]*cx[dir];
					H[1] = cx[dir]*cy[dir];
					H[2] = cx[dir]*cz[dir];
					H[3] = H[1];
					H[4] = cy[dir]*cy[dir];
					H[5] = cy[dir]*cz[dir];
					H[6] = H[2];
					H[7] = H[5];
					H[8] = cz[dir]*cz[dir];

					//Tensor contraction
					TC = 0;
					for(int l = 0; l < 9; l++){
						TC += G[l] * H[l];
					}



					cu2 = u[index]*cx[dir] + v[index]*cy[dir] + w[index]*cz[dir];
					// calculate equilibrium distribution function
					f_eq = mean_alpha*(chi[dir]*prod_u_grad_rho + psi[dir]*TC) + rho[index] * ( phi[dir] + teta[dir]*mean_alpha + weight[dir] * (3*cu2+4.5*cu2*cu2-1.5*cu1));

					index9 = index + dir * n * m * h;
					// calculate updated distribution function
					f_CollPert = omega_eff*f_eq + (1-omega_eff)*f[index9] + pert;

					r_fColl[index9] = k_r*f_CollPert + k_k*cosin*(phi[dir]+teta[dir]*mean_alpha);
					b_fColl[index9] = k_b*f_CollPert + k_k*cosin*(phi[dir]+teta[dir]*mean_alpha);
				}
			}
		}
	}
}

void createBubble(FLOAT_TYPE *x, FLOAT_TYPE *y,int n, int m, FLOAT_TYPE radius, FLOAT_TYPE *r_f, FLOAT_TYPE *b_f, FLOAT_TYPE *r_rho, FLOAT_TYPE *b_rho,
		FLOAT_TYPE r_density, FLOAT_TYPE b_density, FLOAT_TYPE *r_phi, FLOAT_TYPE *b_phi, FLOAT_TYPE *rho) {
	int i, j, k;
	int index, index2;
	for (j=0; j < m; j++){
		for(i = 0; i < n; i++){
			index = j * n + i;

			if( sqrt( pow((x[index]-0.5), 2) + pow((y[index]-0.5),2)) <= radius ){
				r_rho[index] = r_density;
				for (k=0; k < 9; k++){
					// initialise distribution function with small, non-zero values
					index2 = i + j * n + k * m * n;
					r_f[index2] = r_rho[index] * r_phi[k];
				}
			}
			else {
				b_rho[index]=b_density;
				for (k=0; k < 9; k++){
					// initialise distribution function with small, non-zero values
					index2 = i + j * n + k * m * n;
					b_f[index2]   = b_rho[index]*b_phi[k];
				}
			}
			// initialise density
			rho[index] = r_rho[index]+b_rho[index];
		}
	}
}

void createBubble3D(FLOAT_TYPE *x, FLOAT_TYPE *y, FLOAT_TYPE *z, int n, int m, int h, FLOAT_TYPE radius, FLOAT_TYPE *r_f, FLOAT_TYPE *b_f, FLOAT_TYPE *r_rho, FLOAT_TYPE *b_rho,
		FLOAT_TYPE r_density, FLOAT_TYPE b_density, FLOAT_TYPE *phi, FLOAT_TYPE *rho, FLOAT_TYPE *f) {
	int i, j, k, dir;
	int index, index2;
	for(k = 0; k < h; k++){
		for (j=0; j < m; j++){
			for(i = 0; i < n; i++){
				index = k * m * n + j * n + i;

				if( sqrt( pow((x[index]-0.5), 2) + pow((y[index]-0.5),2) + pow((z[index]-0.5),2)) <= radius ){
					r_rho[index] = r_density;
					for (dir=0; dir < 19; dir++){
						// initialise distribution function with small, non-zero values
						index2 = index + dir * h*m*n;
						r_f[index2] = r_rho[index] * phi[dir];
					}
				}
				else {
					b_rho[index]=b_density;
					for (dir=0; dir < 19; dir++){
						// initialise distribution function with small, non-zero values
						index2 = index + dir * h*m*n;
						b_f[index2]   = b_rho[index]*phi[dir];
					}
				}
				for(dir = 0; dir < 19; dir++){
					index2 = index + dir * h*m*n;
					f[index2] = b_f[index2] + r_f[index2];
				}
				// initialise density
				rho[index] = r_rho[index]+b_rho[index];
			}
		}
	}
}

void updateMacroMP(int n, int m, FLOAT_TYPE *u, FLOAT_TYPE *v,FLOAT_TYPE *r_rho, FLOAT_TYPE *b_rho, FLOAT_TYPE *r_f, FLOAT_TYPE *b_f, FLOAT_TYPE *rho, FLOAT_TYPE control_param,
		FLOAT_TYPE r_alpha, FLOAT_TYPE b_alpha, FLOAT_TYPE bubble_radius, FLOAT_TYPE *st_error, int iteration, FLOAT_TYPE st_predicted){

	int index_aux1=0;
	int index_aux2=0;
	FLOAT_TYPE p_in=0.0;
	FLOAT_TYPE p_out=0.0;
	FLOAT_TYPE u_cum, v_cum;
	FLOAT_TYPE r_sum, b_sum;
	FLOAT_TYPE chi;
	int index, index9;
	int cx[9] = {0, 1, 0, -1, 0, 1, -1, -1, 1};
	int cy[9] = {0, 0, 1, 0, -1, 1, 1, -1, -1};
	FLOAT_TYPE st_laplace;
	// Density and Velocity
	for (int j=1; j < m - 1;j++){
		for (int i=1; i < n - 1; i++){
			// auxiliar variables
			u_cum=0.0;
			v_cum=0.0;

			// densities
			index = j * n + i;
			r_sum = 0.0;
			b_sum = 0.0;
			for(int k = 0; k < 9; k++){
				index9 = i + j * n + k * m * n;
				r_sum += r_f[index9];
				b_sum += b_f[index9];
			}
			r_rho[index] = r_sum;
			b_rho[index]= b_sum;
			rho[index] = r_rho[index]+b_rho[index];

			// p_in and p_out for the surface tension
			chi=(r_rho[index]-b_rho[index])/rho[index];
			if (chi >= control_param){
				index_aux1++;
				p_in += r_rho[index];
			}
			else if (chi <= -control_param){
				index_aux2++;
				p_out+=b_rho[index];
			}

			// velocities
			for (int k=0; k < 9; k++){
				index9 = i + j * n + k * m * n;
				u_cum += (r_f[index9]+b_f[index9])*cx[k];
				v_cum += (r_f[index9]+b_f[index9])*cy[k];
			}
			u[index]   = u_cum/rho[index];
			v[index]  = v_cum/rho[index];

		}
	}

	// Calculate surface tension
	p_in=(3.0/5.0)*(1.0-r_alpha)*p_in/index_aux1;      // pressure average inside the bubble
	p_out=(3.0/5.0)*(1.0-b_alpha)*p_out/index_aux2;   // pressure average outside the bubble
	st_laplace=bubble_radius*(p_in-p_out);

	st_error[iteration]=abs(st_predicted-st_laplace)/(st_predicted)*100.0;
}

void updateMacroMP3D(int n, int m, int h, FLOAT_TYPE *u, FLOAT_TYPE *v, FLOAT_TYPE *w,FLOAT_TYPE *r_rho, FLOAT_TYPE *b_rho, FLOAT_TYPE *r_f, FLOAT_TYPE *b_f, FLOAT_TYPE *rho, FLOAT_TYPE control_param,
		FLOAT_TYPE r_alpha, FLOAT_TYPE b_alpha, FLOAT_TYPE bubble_radius, FLOAT_TYPE *st_error, int iteration, FLOAT_TYPE st_predicted, int *cx, int *cy, int *cz, FLOAT_TYPE *f){

	int index_aux1=0;
	int index_aux2=0;
	FLOAT_TYPE p_in=0.0;
	FLOAT_TYPE p_out=0.0;
	FLOAT_TYPE u_cum, v_cum, w_cum;
	FLOAT_TYPE r_sum, b_sum;
	FLOAT_TYPE chi;
	int index, index9;
	FLOAT_TYPE st_laplace;
	// Density and Velocity
	for(int k = 1; k < h - 1; k++){
		for (int j=1; j < m - 1;j++){
			for (int i=1; i < n - 1; i++){
				// auxiliar variables
				u_cum=0.0;
				v_cum=0.0;
				w_cum=0.0;
				// densities
				index = k * m * n + j * n + i;
				r_sum = 0.0;
				b_sum = 0.0;
				for(int dir = 0; dir < 19; dir++){
					r_sum += r_f[index + dir * m * n * h];
					b_sum += b_f[index + dir * m * n * h];
					f[index + dir * m * n * h] = r_f[index + dir * m * n * h] + b_f[index + dir * m * n * h];
				}
				r_rho[index] = r_sum;
				b_rho[index]= b_sum;
				rho[index] = r_rho[index]+b_rho[index];

				// p_in and p_out for the surface tension
				chi=(r_rho[index]-b_rho[index])/rho[index];
				if (chi >= control_param){
					index_aux1++;
					p_in += r_rho[index];
				}
				else if (chi <= -control_param){
					index_aux2++;
					p_out+=b_rho[index];
				}

				// velocities
				for (int dir=0; dir < 19; dir++){
					index9 = index + dir * m * n * h;
					u_cum += (r_f[index9]+b_f[index9])*cx[dir];
					v_cum += (r_f[index9]+b_f[index9])*cy[dir];
					w_cum += (r_f[index9]+b_f[index9])*cz[dir];
				}
				u[index]   = u_cum/rho[index];
				v[index]  = v_cum/rho[index];
				w[index]  = w_cum/rho[index];

			}
		}
	}

	// Calculate surface tension
	p_in=(3.0/5.0)*(1.0-r_alpha)*p_in/index_aux1;      // pressure average inside the bubble
	p_out=(3.0/5.0)*(1.0-b_alpha)*p_out/index_aux2;   // pressure average outside the bubble
	st_laplace=bubble_radius*(p_in-p_out);

	st_error[iteration]=abs(st_predicted-st_laplace)/(st_predicted)*100.0;
}

void peridicBoundaries(int n, int m, FLOAT_TYPE *r_f, FLOAT_TYPE *b_f){

	int index_end, index_start;
	int jn = m-1;
	int js = 0;
	int ie = n-1;
	int iw = 0;

	for (int i=1; i < n-1; i++){
		// north boundary
		index_end = jn * n + i;
		index_start = js * n + i;

		r_f[index_end + 4 * m * n] = r_f[index_start + 4 * m * n];
		r_f[index_end + 7 * m * n] = r_f[index_start + 7 * m * n];
		r_f[index_end + 8 * m * n] = r_f[index_start + 8 * m * n];

		b_f[index_end + 4 * m * n] = b_f[index_start + 4 * m * n];
		b_f[index_end + 7 * m * n] = b_f[index_start + 7 * m * n];
		b_f[index_end + 8 * m * n] = b_f[index_start + 8 * m * n];

		//south boundary
		r_f[index_start + 2 * m * n] = r_f[index_end + 2 * m * n];
		r_f[index_start + 5 * m * n] = r_f[index_end + 5 * m * n];
		r_f[index_start + 6 * m * n] = r_f[index_end + 6 * m * n];

		b_f[index_start + 2 * m * n] = b_f[index_end + 2 * m * n];
		b_f[index_start + 5 * m * n] = b_f[index_end + 5 * m * n];
		b_f[index_start + 6 * m * n] = b_f[index_end + 6 * m * n];
	}



	for (int j=1; j < m-1; j++){
		// east boundary
		index_end = j*n + ie;
		index_start = j*n + iw;

		r_f[index_end + 3 * m * n] = r_f[index_start + 3 * m * n];
		r_f[index_end + 7 * m * n] = r_f[index_start + 7 * m * n];
		r_f[index_end + 6 * m * n] = r_f[index_start + 6 * m * n];

		b_f[index_end + 3 * m * n] = b_f[index_start + 3 * m * n];
		b_f[index_end + 7 * m * n] = b_f[index_start + 7 * m * n];
		b_f[index_end + 6 * m * n] = b_f[index_start + 6 * m * n];

		// west boundary
		r_f[index_start + 1 * m * n] = r_f[index_end + 1 * m * n];
		r_f[index_start + 5 * m * n] = r_f[index_end + 5 * m * n];
		r_f[index_start + 8 * m * n] = r_f[index_end + 8 * m * n];

		b_f[index_start + 1 * m * n] = b_f[index_end + 1 * m * n];
		b_f[index_start + 5 * m * n] = b_f[index_end + 5 * m * n];
		b_f[index_start + 8 * m * n] = b_f[index_end + 8 * m * n];

	}

	// north-east corner
	r_f[(jn*n+ie) + 3 * m * n] = r_f[(jn*n+iw) + 3 * m * n];
	r_f[(jn*n+ie) + 4 * m * n] = r_f[(js*n+ie) + 4 * m * n];
	r_f[(jn*n+ie) + 7 * m * n] = r_f[(js*n+iw) + 7 * m * n];

	b_f[(jn*n+ie) + 3 * m * n] = b_f[(jn*n+iw) + 3 * m * n];
	b_f[(jn*n+ie) + 4 * m * n] = b_f[(js*n+ie) + 4 * m * n];
	b_f[(jn*n+ie) + 7 * m * n] = b_f[(js*n+iw) + 7 * m * n];

	// north-west corner
	r_f[(jn*n+iw) + 1 * m * n] = r_f[(jn*n+ie) + 1 * m * n];
	r_f[(jn*n+iw) + 4 * m * n] = r_f[(js*n+iw) + 4 * m * n];
	r_f[(jn*n+iw) + 8 * m * n] = r_f[(js*n+ie) + 8 * m * n];

	b_f[(jn*n+iw) + 1 * m * n] = b_f[(jn*n+ie) + 1 * m * n];
	b_f[(jn*n+iw) + 4 * m * n] = b_f[(js*n+iw) + 4 * m * n];
	b_f[(jn*n+iw) + 8 * m * n] = b_f[(js*n+ie) + 8 * m * n];

	// south-east corner
	r_f[(js*n+ie) + 2 * m * n] = r_f[(jn*n+ie) + 2 * m * n];
	r_f[(js*n+ie) + 3 * m * n] = r_f[(js*n+iw) + 3 * m * n];
	r_f[(js*n+ie) + 6 * m * n] = r_f[(jn*n+iw) + 6 * m * n];

	b_f[(js*n+ie) + 2 * m * n] = b_f[(jn*n+ie) + 2 * m * n];
	b_f[(js*n+ie) + 3 * m * n] = b_f[(js*n+iw) + 3 * m * n];
	b_f[(js*n+ie) + 6 * m * n] = b_f[(jn*n+iw) + 6 * m * n];


	// south-west corner
	r_f[(js*n+iw) + 2 * m * n] = r_f[(jn*n+iw) + 2 * m * n];
	r_f[(js*n+iw) + 1 * m * n] = r_f[(js*n+ie) + 1 * m * n];
	r_f[(js*n+iw) + 5 * m * n] = r_f[(jn*n+ie) + 5 * m * n];

	b_f[(js*n+iw) + 2 * m * n] = b_f[(jn*n+iw) + 2 * m * n];
	b_f[(js*n+iw) + 1 * m * n] = b_f[(js*n+ie) + 1 * m * n];
	b_f[(js*n+iw) + 5 * m * n] = b_f[(jn*n+ie) + 5 * m * n];
}

void peridicBoundaries3D(int n, int m, int h, FLOAT_TYPE *r_f, FLOAT_TYPE *b_f, FLOAT_TYPE *r_rho, FLOAT_TYPE *b_rho){

	int index_end, index_start;
	int jn = m-1;
	int js = 0;
	int ie = n-1;
	int iw = 0;
	int kf = h-1;
	int kb = 0;
	int ms = m * n * h;
	for(int k = 0; k < h - 1; k++){
		for (int i=0; i < n; i++){
			// north boundary
			index_end = k * n * m + jn * n + i;
			index_start = k * n * m + js * n + i;

			r_f[index_end + 4 * ms] = r_f[index_start + 4 * ms];
			r_f[index_end + 9 * ms] = r_f[index_start + 9 * ms];
			r_f[index_end + 10 * ms] = r_f[index_start + 10 * ms];
			r_f[index_end + 16 * ms] = r_f[index_start + 16 * ms];
			r_f[index_end + 18 * ms] = r_f[index_start + 18 * ms];

			b_f[index_end + 4 * ms] = b_f[index_start + 4 * ms];
			b_f[index_end + 9 * ms] = b_f[index_start + 9 * ms];
			b_f[index_end + 10 * ms] = b_f[index_start + 10 * ms];
			b_f[index_end + 16 * ms] = b_f[index_start + 16 * ms];
			r_f[index_end + 18 * ms] = r_f[index_start + 18 * ms];


			//south boundary
			r_f[index_start + 3 * ms] = r_f[index_end + 3 * ms];
			r_f[index_start + 7 * ms] = r_f[index_end + 7 * ms];
			r_f[index_start + 8 * ms] = r_f[index_end + 8 * ms];
			r_f[index_start + 15 * ms] = r_f[index_end + 15 * ms];
			r_f[index_start + 17 * ms] = r_f[index_end + 17 * ms];

			b_f[index_start + 3 * ms] = b_f[index_end + 3 * ms];
			b_f[index_start + 7 * ms] = b_f[index_end + 7 * ms];
			b_f[index_start + 8 * ms] = b_f[index_end + 8 * ms];
			b_f[index_start + 15 * ms] = b_f[index_end + 15 * ms];
			b_f[index_start + 17 * ms] = b_f[index_end + 17 * ms];
		}

		for (int j=1; j < m-1; j++){
			// east boundary
			index_end = k * m * n + j*n + ie;
			index_start = k * m * n + j*n + iw;

			r_f[index_end + 2 * ms] = r_f[index_start + 2 * ms];
			r_f[index_end + 8 * ms] = r_f[index_start + 8 * ms];
			r_f[index_end + 10 * ms] = r_f[index_start + 10 * ms];
			r_f[index_end + 12 * ms] = r_f[index_start + 12 * ms];
			r_f[index_end + 14 * ms] = r_f[index_start + 14 * ms];

			b_f[index_end + 2 * ms] = b_f[index_start + 2 * ms];
			b_f[index_end + 8 * ms] = b_f[index_start + 8 * ms];
			b_f[index_end + 10 * ms] = b_f[index_start + 10 * ms];
			b_f[index_end + 12 * ms] = b_f[index_start + 12 * ms];
			b_f[index_end + 14 * ms] = b_f[index_start + 14 * ms];

			// west boundary
			r_f[index_start + 1 * ms] = r_f[index_end + 1 * ms];
			r_f[index_start + 7 * ms] = r_f[index_end + 7 * ms];
			r_f[index_start + 9 * ms] = r_f[index_end + 9 * ms];
			r_f[index_start + 11 * ms] = r_f[index_end + 11 * ms];
			r_f[index_start + 13 * ms] = r_f[index_end + 13 * ms];

			b_f[index_start + 1 * ms] = b_f[index_end + 1 * ms];
			b_f[index_start + 7 * ms] = b_f[index_end + 7 * ms];
			b_f[index_start + 9 * ms] = b_f[index_end + 9 * ms];
			r_f[index_start + 11 * ms] = r_f[index_end + 11 * ms];
			r_f[index_start + 13 * ms] = r_f[index_end + 13 * ms];

		}
	}

	for(int j = 0; j < m; j++){
		for(int i = 0; i < n; i++){
			index_end = kf * m * n + j*n + i;
			index_start = kb * m * n + j*n + i;

			//Front boundary
			r_f[index_end + 6 * ms] = r_f[index_start + 6 * ms];
			r_f[index_end + 13 * ms] = r_f[index_start + 13 * ms];
			r_f[index_end + 14 * ms] = r_f[index_start + 14 * ms];
			r_f[index_end + 17 * ms] = r_f[index_start + 17 * ms];
			r_f[index_end + 18 * ms] = r_f[index_start + 18 * ms];

			b_f[index_end + 6 * ms] = b_f[index_start + 6 * ms];
			b_f[index_end + 13 * ms] = b_f[index_start + 13 * ms];
			b_f[index_end + 14 * ms] = b_f[index_start + 14 * ms];
			b_f[index_end + 17 * ms] = b_f[index_start + 17 * ms];
			b_f[index_end + 18 * ms] = b_f[index_start + 18 * ms];

			// back boundary
			r_f[index_start + 5 * ms] = r_f[index_end + 5 * ms];
			r_f[index_start + 11 * ms] = r_f[index_end + 11 * ms];
			r_f[index_start + 12 * ms] = r_f[index_end + 12 * ms];
			r_f[index_start + 15 * ms] = r_f[index_end + 15 * ms];
			r_f[index_start + 16 * ms] = r_f[index_end + 16 * ms];

			b_f[index_start + 5 * ms] = b_f[index_end + 5 * ms];
			b_f[index_start + 11 * ms] = b_f[index_end + 11 * ms];
			b_f[index_start + 12 * ms] = b_f[index_end + 12 * ms];
			r_f[index_start + 15 * ms] = r_f[index_end + 15 * ms];
			r_f[index_start + 16 * ms] = r_f[index_end + 16 * ms];
		}
	}
}

void streamMP(int n, int m, FLOAT_TYPE *r_f, FLOAT_TYPE *b_f, FLOAT_TYPE *r_fColl, FLOAT_TYPE *b_fColl){
	// stream on interior first
	int index,i,j;
	for (j=1;j < m-1;j++){
		for (i=1; i < n-1; i++){
			index = j*n+i;
			r_f[index] = r_fColl[index];
			r_f[index + 1 * m * n] = r_fColl[(j*n+i-1) + 1 * m * n];
			r_f[index + 2 * m * n] = r_fColl[((j-1) * n + i) + 2 * m * n];
			r_f[index + 3 * m * n] = r_fColl[(j*n + i + 1) + 3 * m * n];
			r_f[index + 4 * m * n] = r_fColl[((j+1) * n + i) + 4 * m * n];
			r_f[index + 5 * m * n] = r_fColl[((j-1) * n + i - 1) + 5 * m * n];
			r_f[index + 6 * m * n] = r_fColl[((j-1) * n + i + 1) + 6 * m * n];
			r_f[index + 7 * m * n] = r_fColl[((j+1) * n + i + 1) + 7 * m * n];
			r_f[index + 8 * m * n] = r_fColl[((j+1) * n + i - 1) + 8 * m * n];

			b_f[index] = b_fColl[index];
			b_f[index + 1 * m * n] = b_fColl[(j*n+i-1) + 1 * m * n];
			b_f[index + 2 * m * n] = b_fColl[((j-1) * n + i) + 2 * m * n];
			b_f[index + 3 * m * n] = b_fColl[(j*n + i + 1) + 3 * m * n];
			b_f[index + 4 * m * n] = b_fColl[((j+1) * n + i) + 4 * m * n];
			b_f[index + 5 * m * n] = b_fColl[((j-1) * n + i - 1) + 5 * m * n];
			b_f[index + 6 * m * n] = b_fColl[((j-1) * n + i + 1) + 6 * m * n];
			b_f[index + 7 * m * n] = b_fColl[((j+1) * n + i + 1) + 7 * m * n];
			b_f[index + 8 * m * n] = b_fColl[((j+1) * n + i - 1) + 8 * m * n];
		}
	}
	for (i=1; i < n-1; i++){
		//north boundary
		j = m-1;
		index = j*n+i;

		r_f[index] = r_fColl[index];
		r_f[index + 1 * m * n] = r_fColl[(j*n+i-1) + 1 * m * n];
		r_f[index + 2 * m * n] = r_fColl[((j-1) * n + i) + 2 * m * n];
		r_f[index + 3 * m * n] = r_fColl[(j*n + i + 1) + 3 * m * n];
		r_f[index + 5 * m * n] = r_fColl[((j-1) * n + i - 1) + 5 * m * n];
		r_f[index + 6 * m * n] = r_fColl[((j-1) * n + i + 1) + 6 * m * n];

		b_f[index] = b_fColl[index];
		b_f[index + 1 * m * n] = b_fColl[(j*n+i-1) + 1 * m * n];
		b_f[index + 2 * m * n] = b_fColl[((j-1) * n + i) + 2 * m * n];
		b_f[index + 3 * m * n] = b_fColl[(j*n + i + 1) + 3 * m * n];
		b_f[index + 5 * m * n] = b_fColl[((j-1) * n + i - 1) + 5 * m * n];
		b_f[index + 6 * m * n] = b_fColl[((j-1) * n + i + 1) + 6 * m * n];

		//South boundary
		j = 0;
		index = j*n+i;

		r_f[index] = r_fColl[index];
		r_f[index + 1 * m * n] = r_fColl[(j*n+i-1) + 1 * m * n];
		r_f[index + 3 * m * n] = r_fColl[(j*n + i + 1) + 3 * m * n];
		r_f[index + 4 * m * n] = r_fColl[((j+1) * n + i) + 4 * m * n];
		r_f[index + 7 * m * n] = r_fColl[((j+1) * n + i + 1) + 7 * m * n];
		r_f[index + 8 * m * n] = r_fColl[((j+1) * n + i - 1) + 8 * m * n];

		b_f[index] = b_fColl[index];
		b_f[index + 1 * m * n] = b_fColl[(j*n+i-1) + 1 * m * n];
		b_f[index + 3 * m * n] = b_fColl[(j*n + i + 1) + 3 * m * n];
		b_f[index + 4 * m * n] = b_fColl[((j+1) * n + i) + 4 * m * n];
		b_f[index + 7 * m * n] = b_fColl[((j+1) * n + i + 1) + 7 * m * n];
		b_f[index + 8 * m * n] = b_fColl[((j+1) * n + i - 1) + 8 * m * n];
	}

	for (j=1;j < m-1;j++){
		//east
		i = n-1;
		index = j*n+i;

		r_f[index] = r_fColl[index];
		r_f[index + 1 * m * n] = r_fColl[(j*n+i-1) + 1 * m * n];
		r_f[index + 2 * m * n] = r_fColl[((j-1) * n + i) + 2 * m * n];
		r_f[index + 4 * m * n] = r_fColl[((j+1) * n + i) + 4 * m * n];
		r_f[index + 5 * m * n] = r_fColl[((j-1) * n + i - 1) + 5 * m * n];
		r_f[index + 8 * m * n] = r_fColl[((j+1) * n + i - 1) + 8 * m * n];

		b_f[index] = b_fColl[index];
		b_f[index + 1 * m * n] = b_fColl[(j*n+i-1) + 1 * m * n];
		b_f[index + 2 * m * n] = b_fColl[((j-1) * n + i) + 2 * m * n];
		b_f[index + 4 * m * n] = b_fColl[((j+1) * n + i) + 4 * m * n];
		b_f[index + 5 * m * n] = b_fColl[((j-1) * n + i - 1) + 5 * m * n];
		b_f[index + 8 * m * n] = b_fColl[((j+1) * n + i - 1) + 8 * m * n];

		//west
		i = 0;
		index = j*n+i;

		r_f[index] = r_fColl[index];
		r_f[index + 2 * m * n] = r_fColl[((j-1) * n + i) + 2 * m * n];
		r_f[index + 3 * m * n] = r_fColl[(j*n + i + 1) + 3 * m * n];
		r_f[index + 4 * m * n] = r_fColl[((j+1) * n + i) + 4 * m * n];
		r_f[index + 6 * m * n] = r_fColl[((j-1) * n + i + 1) + 6 * m * n];
		r_f[index + 7 * m * n] = r_fColl[((j+1) * n + i + 1) + 7 * m * n];

		b_f[index] = b_fColl[index];
		b_f[index + 2 * m * n] = b_fColl[((j-1) * n + i) + 2 * m * n];
		b_f[index + 3 * m * n] = b_fColl[(j*n + i + 1) + 3 * m * n];
		b_f[index + 4 * m * n] = b_fColl[((j+1) * n + i) + 4 * m * n];
		b_f[index + 6 * m * n] = b_fColl[((j-1) * n + i + 1) + 6 * m * n];
		b_f[index + 7 * m * n] = b_fColl[((j+1) * n + i + 1) + 7 * m * n];
	}

	// north-east corner
	i=n-1; j=m-1;
	index = j*n+i;

	r_f[index] = r_fColl[index];
	r_f[index + 1 * m * n] = r_fColl[(j*n+i-1) + 1 * m * n];
	r_f[index + 2 * m * n] = r_fColl[((j-1) * n + i) + 2 * m * n];
	r_f[index + 5 * m * n] = r_fColl[((j-1) * n + i - 1) + 5 * m * n];

	b_f[index] = b_fColl[index];
	b_f[index + 1 * m * n] = b_fColl[(j*n+i-1) + 1 * m * n];
	b_f[index + 2 * m * n] = b_fColl[((j-1) * n + i) + 2 * m * n];
	b_f[index + 5 * m * n] = b_fColl[((j-1) * n + i - 1) + 5 * m * n];

	//north-west corner
	i=0; j=m-1;
	index = j*n+i;

	r_f[index] = r_fColl[index];
	r_f[index + 2 * m * n] = r_fColl[((j-1) * n + i) + 2 * m * n];
	r_f[index + 3 * m * n] = r_fColl[(j*n + i + 1) + 3 * m * n];
	r_f[index + 6 * m * n] = r_fColl[((j-1) * n + i + 1) + 6 * m * n];

	b_f[index] = b_fColl[index];
	b_f[index + 2 * m * n] = b_fColl[((j-1) * n + i) + 2 * m * n];
	b_f[index + 3 * m * n] = b_fColl[(j*n + i + 1) + 3 * m * n];
	b_f[index + 6 * m * n] = b_fColl[((j-1) * n + i + 1) + 6 * m * n];

	// south-east corner
	i=n-1; j=0;
	index = j*n+i;

	r_f[index] = r_fColl[index];
	r_f[index + 1 * m * n] = r_fColl[(j*n+i-1) + 1 * m * n];
	r_f[index + 4 * m * n] = r_fColl[((j+1) * n + i) + 4 * m * n];
	r_f[index + 8 * m * n] = r_fColl[((j+1) * n + i - 1) + 8 * m * n];

	b_f[index] = b_fColl[index];
	b_f[index + 1 * m * n] = b_fColl[(j*n+i-1) + 1 * m * n];
	b_f[index + 4 * m * n] = b_fColl[((j+1) * n + i) + 4 * m * n];
	b_f[index + 8 * m * n] = b_fColl[((j+1) * n + i - 1) + 8 * m * n];

	// south-west corner
	i=0; j=0;
	index = j*n+i;

	r_f[index] = r_fColl[index];
	r_f[index + 3 * m * n] = r_fColl[(j*n + i + 1) + 3 * m * n];
	r_f[index + 4 * m * n] = r_fColl[((j+1) * n + i) + 4 * m * n];
	r_f[index + 7 * m * n] = r_fColl[((j+1) * n + i + 1) + 7 * m * n];

	b_f[index] = b_fColl[index];
	b_f[index + 3 * m * n] = b_fColl[(j*n + i + 1) + 3 * m * n];
	b_f[index + 4 * m * n] = b_fColl[((j+1) * n + i) + 4 * m * n];
	b_f[index + 7 * m * n] = b_fColl[((j+1) * n + i + 1) + 7 * m * n];

}

void streamMP3D(int n, int m, int h, FLOAT_TYPE *r_f, FLOAT_TYPE *b_f, FLOAT_TYPE *r_fColl, FLOAT_TYPE *b_fColl, bool *stream){
	// stream on interior first
	int index,i,j,k;
	int ms = m*n*h;
	int c3D[19] = { 0, -1, 1, -1 * n, n, -m * n, +m * n, -1 * n - 1, -1 * n + 1,
			n - 1, n + 1, -m * n - 1, -m * n + 1, +m * n - 1,
			m * n + 1, -m * n - n, -m * n + n, m * n - n, m * n + n };

	for(k = 1; k < h - 1; k++){
		for (j=1;j < m-1;j++){
			for (i=1; i < n-1; i++){
				index = k * m * n + j*n+i;
				r_f[index] = r_fColl[index];
				b_f[index] = b_fColl[index];
				for(int dir = 1; dir < 19; dir++){
					r_f[index + dir * ms] = (stream[index+	(dir-1) * ms]	==	1)	?	r_fColl[index + dir * ms + c3D[dir]]:	r_f[index + dir * ms];
					b_f[index + dir * ms] = (stream[index+	(dir-1) * ms]	==	1)	?	b_fColl[index + dir * ms + c3D[dir]]:	b_f[index + dir * ms];
				}
			}
		}
	}
}

void resetArrays(FLOAT_TYPE *color_gradient, int n, int m){
	for(int i = 0; i < m * n *2; i++){
		color_gradient[i] = 0.0;
	}
}

FLOAT_TYPE* convertArray(int n, int m, FLOAT_TYPE *arr){
	FLOAT_TYPE *result = createHostArrayFlt(n*m, ARRAY_NONE);

	for(int i = 0; i < n; i++){
		for(int j = 0; j < m; j++){
			result[j*n+i] = arr[i*m+j];
		}
	}

	return result;
}

void updateSurfaceTension(FLOAT_TYPE *r_rho, FLOAT_TYPE *b_rho, FLOAT_TYPE control_param,
		FLOAT_TYPE st_predicted, FLOAT_TYPE *st_error, int iteration, FLOAT_TYPE r_alpha, FLOAT_TYPE b_alpha, FLOAT_TYPE bubble_radius, int n, int m){

	FLOAT_TYPE chi, p_in = 0.0, p_out = 0.0, st_laplace, r;
	int index, index_aux1 = 0, index_aux2 = 0;
	for(int j = 1; j < m-1; j++){
		for(int i = 1; i < n-1; i++){
			index = j * n + i;
			r = r_rho[index] + b_rho[index];
			// p_in and p_out for the surface tension
			chi=(r_rho[index]-b_rho[index])/r;
			if (chi >= control_param){
				index_aux1++;
				p_in += r_rho[index];
			}
			else if (chi <= -control_param){
				index_aux2++;
				p_out+=b_rho[index];
			}
		}
	}

	p_in=(3.0/5.0)*(1.0-r_alpha)*p_in/index_aux1;      // pressure average inside the bubble
	p_out=(3.0/5.0)*(1.0-b_alpha)*p_out/index_aux2;   // pressure average outside the bubble
	st_laplace=bubble_radius*(p_in-p_out);

	st_error[iteration]=abs(st_predicted-st_laplace)/(st_predicted)*100.0;
}

FLOAT_TYPE calculateSurfaceTension(FLOAT_TYPE p_in_mean, FLOAT_TYPE p_out_mean, FLOAT_TYPE r_alpha, FLOAT_TYPE b_alpha, FLOAT_TYPE bubble_radius, FLOAT_TYPE st_predicted){

	FLOAT_TYPE st_laplace;
	p_in_mean=(3.0/5.0)*(1.0-r_alpha)*p_in_mean;      // pressure average inside the bubble
	p_out_mean=(3.0/5.0)*(1.0-b_alpha)*p_out_mean;   // pressure average outside the bubble
	st_laplace=20*(p_in_mean-p_out_mean);

	return abs(st_predicted-st_laplace)/(st_predicted)*100.0;
}

void createCoalescenceBubble(FLOAT_TYPE *x, FLOAT_TYPE *y,int n, int m, FLOAT_TYPE radius, FLOAT_TYPE *r_f, FLOAT_TYPE *b_f, FLOAT_TYPE *r_rho, FLOAT_TYPE *b_rho,
		FLOAT_TYPE r_density, FLOAT_TYPE b_density, FLOAT_TYPE *r_phi, FLOAT_TYPE *b_phi, FLOAT_TYPE *rho){

	int i, j, k;
	int index, index2;
	for (j=0; j < m; j++){
		for(i = 0; i < n; i++){
			index = j * n + i;

			if( sqrt( pow((x[index]-0.5), 2) + pow((y[index]-0.5 + radius),2)) <= radius || sqrt( pow((x[index]-0.5), 2) + pow((y[index]-0.5 - radius),2)) <= radius){
				r_rho[index] = r_density;
				for (k=0; k < 9; k++){
					// initialise distribution function with small, non-zero values
					index2 = i + j * n + k * m * n;
					r_f[index2] = r_rho[index] * r_phi[k];
				}
			}
			else {
				b_rho[index]=b_density;
				for (k=0; k < 9; k++){
					// initialise distribution function with small, non-zero values
					index2 = i + j * n + k * m * n;
					b_f[index2]   = b_rho[index]*b_phi[k];
				}
			}
			// initialise density
			rho[index] = r_rho[index]+b_rho[index];
		}
	}
}

FLOAT_TYPE validateCoalescenceCase(FLOAT_TYPE *r_rho, FLOAT_TYPE *b_rho, int n, int m, FLOAT_TYPE radius){
	int j;
	if(m % 2 == 0)
		j = m/2;
	else
		j = (m+1) / 2;

	FLOAT_TYPE rho;
	int aux = 0;
	for(int i = 0; i < n; i++){
		rho = r_rho[j * n + i] + b_rho[j * n + i];
		if((r_rho[j * n + i] - b_rho[j * n + i]) / rho > 0.9){
			aux++;
		}
	}

	printf("counter %d\n", aux);
	return (abs(radius * sqrt(2.0) - ((FLOAT_TYPE)aux) / ( n * 2.0)) / (radius * sqrt(2.0))) * 100.0;


}
