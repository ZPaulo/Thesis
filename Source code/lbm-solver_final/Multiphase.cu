#include "hip/hip_runtime.h"
#include "Multiphase.h"
#include <math.h>
#include <stdio.h>
void mp2DColl(int *fluid,int n, int m, FLOAT_TYPE *rho, FLOAT_TYPE *u,
		FLOAT_TYPE *v, FLOAT_TYPE *r_f, FLOAT_TYPE *b_f, FLOAT_TYPE *r_rho, FLOAT_TYPE *b_rho,
		FLOAT_TYPE *r_phi, FLOAT_TYPE *b_phi, FLOAT_TYPE *w_pert, FLOAT_TYPE *color_gradient,
		FLOAT_TYPE r_omega, FLOAT_TYPE b_omega, FLOAT_TYPE control_param, FLOAT_TYPE del,
		FLOAT_TYPE beta, FLOAT_TYPE g_limit,  FLOAT_TYPE r_A,  FLOAT_TYPE b_A, FLOAT_TYPE *r_fPert, FLOAT_TYPE *b_fPert){

	FLOAT_TYPE cu1, cu2;

	int cx[9] = {0, 1, 0, -1, 0, 1, -1, -1, 1};
	int cy[9] = {0, 0, 1, 0, -1, 1, 1, -1, -1};
	FLOAT_TYPE weight[9] = {4.0/9.0, 1.0/9.0, 1.0/9.0, 1.0/9.0, 1.0/9.0, 1.0/36.0, 1.0/36.0, 1.0/36.0, 1.0/36.0};
	FLOAT_TYPE cosin[9] = {0.0};
	FLOAT_TYPE chi;
	FLOAT_TYPE r_omega_temp, b_omega_temp;
	FLOAT_TYPE a1      =   2.0 * r_omega * b_omega/(r_omega+b_omega);
	FLOAT_TYPE a2      =   2.0 * (r_omega - a1) / del;
	FLOAT_TYPE a3      =   -a2 / (2.0 * del);
	FLOAT_TYPE a4      =   2.0 * (a1 - b_omega) / del;
	FLOAT_TYPE a5      =   a4 / (2.0 * del);
	FLOAT_TYPE color_gradient_norm;
	FLOAT_TYPE k_r, k_b, k_k;
	FLOAT_TYPE norm_c;
	FLOAT_TYPE prod_c_g;
	FLOAT_TYPE r_pert, b_pert;
	FLOAT_TYPE r_feq, b_feq;
	FLOAT_TYPE fn05;
	int index, index9, temp_index;
	for (int i=0;i < n; i++){
		for (int j=0; j < m; j++){
			// temporary variable 1
			index = i*m + j;
			cu1 = u[index]*u[index] + v[index]*v[index];

			for (int k=0; k<9; k++){
				// temporary variable 2
				cu2 = u[index]*cx[k] + v[index]*cy[k];

				index9 = k + index * 9;
				// calculate equilibrium distribution function


				// calculate color gradient - 4th order
				if (k!=0){ // the rest node (k=0) does not contribute to the color gradient

					if (i!=0 && j!=0 && i!=(n-1) && j!=(m-1)){ // Interior points - In the boundary it is calculated by "mirroring" the density
						temp_index = (i + cx[k]) * m + j + cy[k];
						color_gradient[index * 2] += (r_rho[temp_index] - b_rho[temp_index]) * cx[k];
						color_gradient[index * 2 + 1] += (r_rho[temp_index] - b_rho[temp_index]) * cy[k];
					}
					else if (j==(m-1) && i!=0 && i!=(n-1)) {// north boundary
						temp_index = (i + cx[k]) * m + j - abs(cy[k]);
						color_gradient[index * 2] += (r_rho[temp_index] - b_rho[temp_index]) * cx[k];
						color_gradient[index * 2 + 1] = 0;
					}
					else if (j==0 && i!=0 && i!=(n-1)){  // south boundary
						temp_index = (i + cx[k]) * m + j + abs(cy[k]);
						color_gradient[index * 2] += (r_rho[temp_index] - b_rho[temp_index]) * cx[k];
						color_gradient[index * 2 + 1] = 0;
					}
					else if (i==(n-1) && j!=0 && j!=(m-1)){  // east boundary
						temp_index = (i - abs(cx[k])) * m + j + cy[k];
						color_gradient[index * 2] = 0;
						color_gradient[index * 2 + 1] +=  (r_rho[temp_index] - b_rho[temp_index]) * cy[k];
					}
					else if (i==0 && j!=0 && j!=(m-1)){ //  west boundary
						temp_index = (i + abs(cx[k])) * m + j + cy[k];
						color_gradient[index * 2] = 0;
						color_gradient[index * 2 + 1] +=  (r_rho[temp_index] - b_rho[temp_index]) * cy[k];
					}
				}
			}

			// relaxation parameter to choose a proper omega at the interface
			if (r_omega != b_omega){
				chi=(r_rho[index] - b_rho[index])/rho[index];
				if(chi >= -control_param && chi <= control_param){
					if (chi > del)
						r_omega_temp=r_omega;
					else if (chi <= del && chi > 0)
						r_omega_temp=a1 + a2 * chi + a3 * chi * chi;
					else if (chi <= 0 && chi >= -del)
						r_omega_temp=a1 + a4 * chi + a5 * chi * chi;
					else if (chi < -del)
						r_omega_temp=b_omega;
				}
			}
			else
				r_omega_temp=r_omega;

			b_omega_temp=r_omega_temp;

			// invariable quantities
			color_gradient_norm = sqrt(pow(color_gradient[index * 2],2) + pow(color_gradient[index * 2 + 1],2));
			k_r=r_rho[index]/rho[index];
			k_b=b_rho[index]/rho[index];
			k_k= beta * r_rho[index] * b_rho[index]/(pow(rho[index],2));

			for (int k=0;k<9;k++){
				if (color_gradient_norm > g_limit){
					if (k!=0){
						norm_c= sqrt(pow(cx[k],2)+pow(cy[k],2));
						cosin[k]=(cx[k]*color_gradient[index * 2]+cy[k]*color_gradient[index * 2 + 1]) / (color_gradient_norm*norm_c);
					}
					else
						cosin[k]=0.0;
					// calculate perturbation terms
					prod_c_g=cx[k]*color_gradient[index * 2]+cy[k]*color_gradient[index * 2 + 1];
					r_pert=0.5*r_A*color_gradient_norm*(weight[k]*pow((prod_c_g/color_gradient_norm),2)-w_pert[k]);
					b_pert=0.5*b_A*color_gradient_norm*(weight[k]*pow((prod_c_g/color_gradient_norm),2)-w_pert[k]);

				}
				else{
					// ther perturbation terms are null
					r_pert=0;
					b_pert=0;
				}
				// calculate updated distribution function
				index9 = k + index * 9;
				r_feq = r_rho[index] * (r_phi[k] + weight[k] * (3 * cu2 + 4.5 * cu2 * cu2 - 1.5 * cu1 * cu1));
				b_feq = b_rho[index] * (b_phi[k] + weight[k] * (3 * cu2 + 4.5 * cu2 * cu2 - 1.5 * cu1 * cu1));

				r_fPert[index9] = r_omega_temp*r_feq + (1-r_omega_temp)*r_f[index9]+r_pert;
				b_fPert[index9] = b_omega_temp*b_feq + (1-b_omega_temp)*b_f[index9]+b_pert;
				fn05 = r_fPert[index9] + b_fPert[index9];
				// perform recolor step
				r_fPert[index9]=k_r*fn05+k_k*cosin[k]*(r_rho[index]*r_phi[k]+b_rho[index]*b_phi[k]);
				b_fPert[index9]=k_b*fn05-k_k*cosin[k]*(r_rho[index]*r_phi[k]+b_rho[index]*b_phi[k]);
			}
		}
	}



}

void createBubble(float *x, float *y,int n, int m, FLOAT_TYPE radius, FLOAT_TYPE *r_f, FLOAT_TYPE *b_f, FLOAT_TYPE *r_rho, FLOAT_TYPE *b_rho,
		FLOAT_TYPE r_density, FLOAT_TYPE b_density, FLOAT_TYPE *r_phi, FLOAT_TYPE *b_phi, FLOAT_TYPE *rho) {
	int i, j, k;
	int index, index2;
	for(i = 0; i < n; i++){
		for (j=0; j < m; j++){
			index = i*m + j;
			if( sqrt( pow((x[index]-0.5), 2) + pow((y[index]-0.5),2)) <= radius ){
				for (k=0; k < 9; k++){
					// initialise distribution function with small, non-zero values
					r_rho[index] = r_density;
					//x + WIDTH * (y + DEPTH * z)
					index2 = k + index * 9;
					r_f[index2] = r_rho[index] * r_phi[k];
				}
			}
			else {
				for (k=0; k < 9; k++){
					// initialise distribution function with small, non-zero values
					b_rho[index]=b_density;
					index2 = k + index * 9;
					b_f[index2]   = b_rho[index]*b_phi[k];
				}
			}
			// initialise density
			rho[index] = r_rho[index]+b_rho[index];
		}
	}
}
