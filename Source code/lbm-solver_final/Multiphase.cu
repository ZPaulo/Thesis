#include "hip/hip_runtime.h"
#include "Multiphase.h"
#include <math.h>
#include <stdio.h>
#include "ArrayUtils.h"

void mp2DColl(int *fluid,int n, int m, FLOAT_TYPE *rho, FLOAT_TYPE *u,
		FLOAT_TYPE *v, FLOAT_TYPE *r_f, FLOAT_TYPE *b_f, FLOAT_TYPE *r_rho, FLOAT_TYPE *b_rho,
		FLOAT_TYPE *r_phi, FLOAT_TYPE *b_phi, FLOAT_TYPE *w_pert, FLOAT_TYPE *color_gradient,
		FLOAT_TYPE r_omega, FLOAT_TYPE b_omega, FLOAT_TYPE control_param, FLOAT_TYPE del,
		FLOAT_TYPE beta, FLOAT_TYPE g_limit,  FLOAT_TYPE r_A,  FLOAT_TYPE b_A, FLOAT_TYPE *r_fPert, FLOAT_TYPE *b_fPert){

	FLOAT_TYPE cu1, cu2, r_CollPert, b_CollPert;

	int cx[9] = {0, 1, 0, -1, 0, 1, -1, -1, 1};
	int cy[9] = {0, 0, 1, 0, -1, 1, 1, -1, -1};
	FLOAT_TYPE weight[9] = {4.0/9.0, 1.0/9.0, 1.0/9.0, 1.0/9.0, 1.0/9.0, 1.0/36.0, 1.0/36.0, 1.0/36.0, 1.0/36.0};
	FLOAT_TYPE cosin[9] = {0.0};
	FLOAT_TYPE chi;
	FLOAT_TYPE r_omega_temp, b_omega_temp;
	FLOAT_TYPE a1      =   2.0 * r_omega * b_omega/(r_omega+b_omega);
	FLOAT_TYPE a2      =   2.0 * (r_omega - a1) / del;
	FLOAT_TYPE a3      =   -a2 / (2.0 * del);
	FLOAT_TYPE a4      =   2.0 * (a1 - b_omega) / del;
	FLOAT_TYPE a5      =   a4 / (2.0 * del);
	FLOAT_TYPE color_gradient_norm;
	FLOAT_TYPE k_r, k_b, k_k;
	FLOAT_TYPE norm_c;
	FLOAT_TYPE prod_c_g;
	FLOAT_TYPE r_pert, b_pert;
	FLOAT_TYPE r_feq, b_feq;
	FLOAT_TYPE fn05;
	int index, index9, temp_index;
	for (int i=0;i < n; i++){
		for (int j=0; j < m; j++){
			// temporary variable 1
			index = i*m + i;
			cu1 = u[index]*u[index] + v[index]*v[index];

			for (int k=0; k<9; k++){
				// temporary variable 2
				cu2 = u[index]*cx[k] + v[index]*cy[k];

				// calculate color gradient - 4th order
				//	if (k!=0){ // the rest node (k=0) does not contribute to the color gradient

				if (i!=0 && j!=0 && i!=(n-1) && j!=(m-1)){ // Interior points - In the boundary it is calculated by "mirroring" the density
					temp_index = (i + cx[k]) * m + j + cy[k];
					color_gradient[index * 2] += (r_rho[temp_index] - b_rho[temp_index]) * cx[k];
					color_gradient[index * 2 + 1] += (r_rho[temp_index] - b_rho[temp_index]) * cy[k];
					if(index * 2 + 1 == 32459)
						printf("interior\n");
				}
				else if (j==(m-1) && i!=0 && i!=(n-1)) {// north boundary
					temp_index = (i + cx[k]) * m + j - abs(cy[k]);
					color_gradient[index * 2] += (r_rho[temp_index] - b_rho[temp_index]) * cx[k];
					color_gradient[index * 2 + 1] = 0;
					//						if(abs(color_gradient[index * 2]) > 0)
					//						printf("north %d\n", index * 2);
					if(index * 2 + 1 == 13215)
						printf("north\n");
				}
				else if (j==0 && i!=0 && i!=(n-1)){  // south boundary
					temp_index = (i + cx[k]) * m + j + abs(cy[k]);
					color_gradient[index * 2] += (r_rho[temp_index] - b_rho[temp_index]) * cx[k];
					color_gradient[index * 2 + 1] = 0;
					if(index * 2 + 1 == 13215)
						printf("south\n");
				}
				else if (i==(n-1) && j!=0 && j!=(m-1)){  // east boundary
					temp_index = (i - abs(cx[k])) * m + j + cy[k];
					color_gradient[index * 2] = 0;
					color_gradient[index * 2 + 1] +=  (r_rho[temp_index] - b_rho[temp_index]) * cy[k];
					if(index * 2 + 1 == 13215)
						printf("east\n");
				}
				else if (i==0 && j!=0 && j!=(m-1)){ //  west boundary
					temp_index = (i + abs(cx[k])) * m + j + cy[k];
					color_gradient[index * 2] = 0;
					color_gradient[index * 2 + 1] +=  (r_rho[temp_index] - b_rho[temp_index]) * cy[k];
					if(index * 2 + 1 == 13215)
						printf("west\n");
				}
				//}
			}

			// relaxation parameter to choose a proper omega at the interface
			if (r_omega != b_omega){
				chi=(r_rho[index] - b_rho[index])/rho[index];
				if(chi >= -control_param && chi <= control_param){
					if (chi > del)
						r_omega_temp=r_omega;
					else if (chi <= del && chi > 0)
						r_omega_temp=a1 + a2 * chi + a3 * chi * chi;
					else if (chi <= 0 && chi >= -del)
						r_omega_temp=a1 + a4 * chi + a5 * chi * chi;
					else if (chi < -del)
						r_omega_temp=b_omega;
				}
			}
			else
				r_omega_temp=r_omega;

			b_omega_temp=r_omega_temp;

			// invariable quantities
			color_gradient_norm = sqrt(pow(color_gradient[index * 2],2) + pow(color_gradient[index * 2 + 1],2));
			k_r=r_rho[index]/rho[index];
			k_b=b_rho[index]/rho[index];
			k_k= beta * r_rho[index] * b_rho[index]/(pow(rho[index],2));

			for (int k=0;k<9;k++){
				if (color_gradient_norm > g_limit){
					prod_c_g=cx[k]*color_gradient[index * 2]+cy[k]*color_gradient[index * 2 + 1];
					if (k!=0){
						norm_c= sqrt(pow(cx[k],2)+pow(cy[k],2));
						cosin[k]= prod_c_g / (color_gradient_norm*norm_c);
					}
					else
						cosin[k]=0.0;
					// calculate perturbation terms

					r_pert=0.5*r_A*color_gradient_norm*(weight[k]*pow((prod_c_g/color_gradient_norm),2)-w_pert[k]);
					b_pert=0.5*b_A*color_gradient_norm*(weight[k]*pow((prod_c_g/color_gradient_norm),2)-w_pert[k]);

				}
				else{
					// ther perturbation terms are null
					r_pert=0;
					b_pert=0;
				}

				index9 = k + index * 9;
				index9 = k + index * 9;
				// calculate equilibrium distribution function
				r_feq = r_rho[index] * (r_phi[k] + weight[k] * (3 * cu2 + 4.5 * cu2 * cu2 - 1.5 * cu1));
				b_feq = b_rho[index] * (b_phi[k] + weight[k] * (3 * cu2 + 4.5 * cu2 * cu2 - 1.5 * cu1));

				// calculate updated distribution function
				r_CollPert = r_omega_temp*r_feq + (1-r_omega_temp)*r_f[index9]+r_pert;
				b_CollPert = b_omega_temp*b_feq + (1-b_omega_temp)*b_f[index9]+b_pert;

				fn05 = r_CollPert + b_CollPert;
				// perform recolor step
				r_fPert[index9]=k_r*fn05+k_k*cosin[k]*(r_rho[index]*r_phi[k]+b_rho[index]*b_phi[k]);
				b_fPert[index9]=k_b*fn05-k_k*cosin[k]*(r_rho[index]*r_phi[k]+b_rho[index]*b_phi[k]);
			}
		}
	}



}

void createBubble(float *x, float *y,int n, int m, FLOAT_TYPE radius, FLOAT_TYPE *r_f, FLOAT_TYPE *b_f, FLOAT_TYPE *r_rho, FLOAT_TYPE *b_rho,
		FLOAT_TYPE r_density, FLOAT_TYPE b_density, FLOAT_TYPE *r_phi, FLOAT_TYPE *b_phi, FLOAT_TYPE *rho) {
	int i, j, k;
	int index, index2;
	for(i = 0; i < n; i++){
		for (j=0; j < m; j++){
			index = i * m + j;
			if(r_rho[index] > 0){
				b_rho[index]=0.0;
				for (k=0; k < 9; k++){
					// initialise distribution function with small, non-zero values
					index2 = index * 9 + k;
					r_f[index2] = r_rho[index] * r_phi[k];
				}
			}
			else{
				b_rho[index]=b_density;
				for (k=0; k < 9; k++){
					// initialise distribution function with small, non-zero values
					index2 = k + index * 9;
					b_f[index2]   = b_rho[index]*b_phi[k];
				}
			}

			//			if( sqrt( pow((x[index]-0.5), 2) + pow((y[index]-0.5),2)) <= radius ){
			//				r_rho[index] = r_density;
			//				for (k=0; k < 9; k++){
			//					// initialise distribution function with small, non-zero values
			//					index2 = k + index * 9;
			//					r_f[index2] = r_rho[index] * r_phi[k];
			//				}
			//			}
			//			else {
			//				b_rho[index]=b_density;
			//				for (k=0; k < 9; k++){
			//					// initialise distribution function with small, non-zero values
			//					index2 = k + index * 9;
			//					b_f[index2]   = b_rho[index]*b_phi[k];
			//				}
			//			}
			// initialise density
			rho[index] = r_rho[index]+b_rho[index];
		}
	}
}

void updateMacroMP(int n, int m, FLOAT_TYPE *u, FLOAT_TYPE *v,FLOAT_TYPE *r_rho, FLOAT_TYPE *b_rho, FLOAT_TYPE *r_f, FLOAT_TYPE *b_f, FLOAT_TYPE *rho, FLOAT_TYPE control_param,
		FLOAT_TYPE r_alpha, FLOAT_TYPE b_alpha, FLOAT_TYPE bubble_radius, FLOAT_TYPE *st_error, int iteration, FLOAT_TYPE st_predicted){

	int index_aux1=0;
	int index_aux2=0;
	FLOAT_TYPE p_in=0.0;
	FLOAT_TYPE p_out=0.0;
	FLOAT_TYPE u_cum, v_cum;
	FLOAT_TYPE r_sum, b_sum;
	FLOAT_TYPE chi;
	int index, index9;
	int cx[9] = {0, 1, 0, -1, 0, 1, -1, -1, 1};
	int cy[9] = {0, 0, 1, 0, -1, 1, 1, -1, -1};
	FLOAT_TYPE st_laplace;
	// Density and Velocity
	for (int i=1; i < n - 1; i++){
		for (int j=1; j < m - 1;j++){
			// auxiliar variables
			u_cum=0.0;
			v_cum=0.0;

			// densities
			index = i * m + j;
			r_sum = 0.0;
			b_sum = 0.0;
			for(int k = 0; k < 9; k++){
				index9 = index* 9 + k;
				r_sum += r_f[index9];
				b_sum += b_f[index9];
			}
			r_rho[index] = r_sum;
			b_rho[index]= b_sum;
			rho[index] = r_rho[index]+b_rho[index];

			// p_in and p_out for the surface tension
			chi=(r_rho[index]-b_rho[index])/rho[index];
			if (chi>=control_param){
				index_aux1++;
				p_in += r_rho[index];
			}
			else if (chi<control_param){
				index_aux2++;
				p_out+=b_rho[index];
			}

			// velocities
			for (int k=0; k < 9; k++){
				index9 = index * 9 + k;
				u_cum += (r_f[index9]+b_f[index9])*cx[k];
				v_cum += (r_f[index9]+b_f[index9])*cy[k];
			}
			u[index]   = u_cum/rho[index];
			v[index]  = v_cum/rho[index];
		}
	}

	// Calculate surface tension
	p_in=(3.0/5.0)*(1.0-r_alpha)*p_in/index_aux1;      // pressure average inside the bubble
	p_out=(3.0/5.0)*(1.0-b_alpha)*p_out/index_aux2;   // pressure average outside the bubble
	st_laplace=bubble_radius*(p_in-p_out);
	st_error[iteration]=abs(st_predicted-st_laplace)/(st_predicted)*100.0;
}

void peridicBoundaries(int n, int m, FLOAT_TYPE *r_f, FLOAT_TYPE *b_f, FLOAT_TYPE *r_rho, FLOAT_TYPE *b_rho,
		FLOAT_TYPE b_density, FLOAT_TYPE *u, FLOAT_TYPE *v){

	int index_end, index_start;
	int jn = m-1;
	int js = 0;
	int ie = n-1;
	int iw = 0;

	for (int i=1; i < n-1; i++){
		// north boundary
		index_end = jn * n + i;
		index_start = js * n + i;

		r_f[index_end * 9 + 4] = r_f[index_start * 9 + 4];
		r_f[index_end * 9 + 7] = r_f[index_start * 9 + 7];
		r_f[index_end * 9 + 8] = r_f[index_start * 9 + 8];

		b_f[index_end * 9 + 4] = b_f[index_start * 9 + 4];
		b_f[index_end * 9 + 7] = b_f[index_start * 9 + 7];
		b_f[index_end * 9 + 8] = b_f[index_start * 9 + 8];

		// macroscopic boundary conditions
		r_rho[index_end] = 0;
		b_rho[index_end] = b_density;
		u[index_end]   = 0;
		v[index_end]   = 0;

		//south boundary
		r_f[index_start * 9 + 2] = r_f[index_end * 9 + 2];
		r_f[index_start * 9 + 5] = r_f[index_end * 9 + 5];
		r_f[index_start * 9 + 6] = r_f[index_end * 9 + 6];

		b_f[index_start * 9 + 2] = b_f[index_end * 9 + 2];
		b_f[index_start * 9 + 5] = b_f[index_end * 9 + 5];
		b_f[index_start * 9 + 6] = b_f[index_end * 9 + 6];

		r_rho[index_start] = 0;
		b_rho[index_start] = b_density;
		u[index_start]   = 0;
		v[index_start]   = 0;
	}



	for (int j=1; j < m-1; j++){
		// east boundary
		index_end = j*n + ie;
		index_start = j*n + iw;

		r_f[index_end * 9 + 3] = r_f[index_start * 9 + 3];
		r_f[index_end * 9 + 7] = r_f[index_start * 9 + 7];
		r_f[index_end * 9 + 6] = r_f[index_start * 9 + 6];

		b_f[index_end * 9 + 3] = b_f[index_start * 9 + 3];
		b_f[index_end * 9 + 7] = b_f[index_start * 9 + 7];
		b_f[index_end * 9 + 6] = b_f[index_start * 9 + 6];

		//macroscopic boundary conditions
		r_rho[index_end] = 0;
		b_rho[index_end] = b_density;
		u[index_end]   = 0;
		v[index_end]   = 0;

		// west boundary
		r_f[index_start * 9 + 1] = r_f[index_end * 9 + 1];
		r_f[index_start * 9 + 5] = r_f[index_end * 9 + 5];
		r_f[index_start * 9 + 8] = r_f[index_end * 9 + 8];

		b_f[index_start * 9 + 1] = b_f[index_end * 9 + 1];
		b_f[index_start * 9 + 5] = b_f[index_end * 9 + 5];
		b_f[index_start * 9 + 8] = b_f[index_end * 9 + 8];

		r_rho[index_start] = 0;
		b_rho[index_start] = b_density;
		u[index_start]   = 0;
		v[index_start]   = 0;

	}

	// north-east corner
	r_f[(jn*n+ie) * 9 + 3] = r_f[(jn*n+iw) * 9 + 3];
	r_f[(jn*n+ie) * 9 + 4] = r_f[(js*n+ie) * 9 + 4];
	r_f[(jn*n+ie) * 9 + 7] = r_f[(js*n+iw) * 9 + 7];

	b_f[(jn*n+ie) * 9 + 3] = b_f[(jn*n+iw) * 9 + 3];
	b_f[(jn*n+ie) * 9 + 4] = b_f[(js*n+ie) * 9 + 4];
	b_f[(jn*n+ie) * 9 + 7] = b_f[(js*n+iw) * 9 + 7];

	//	FLOAT_TYPE sum_r = 0.0;
	//	FLOAT_TYPE sum_b = 0.0;
	//	for(int i = 0; i < 9; i++){
	//		sum_r += r_f[(jn*n+ie)*9 + i];
	//		sum_b += b_f[(jn*n+ie)*9 + i];
	//	}
	//
	//	r_rho[jn*n+ie] = sum_r;
	//	b_rho[jn*n+ie] = sum_b;

	r_rho[jn*n+ie] = 0;
	b_rho[jn*n+ie] = b_density;

	u[jn*n+ie]   = 0;
	v[jn*n+ie]   = 0;

	// north-west corner
	r_f[(jn*n+iw) * 9 + 1] = r_f[(jn*n+ie) * 9 + 1];
	r_f[(jn*n+iw) * 9 + 4] = r_f[(js*n+iw) * 9 + 4];
	r_f[(jn*n+iw) * 9 + 8] = r_f[(js*n+ie) * 9 + 8];

	b_f[(jn*n+iw) * 9 + 1] = b_f[(jn*n+ie) * 9 + 1];
	b_f[(jn*n+iw) * 9 + 4] = b_f[(js*n+iw) * 9 + 4];
	b_f[(jn*n+iw) * 9 + 8] = b_f[(js*n+ie) * 9 + 8];

	//	sum_r = 0.0;
	//	sum_b = 0.0;
	//	for(int i = 0; i < 9; i++){
	//		sum_r += r_f[(jn*n+iw)*9 + i];
	//		sum_b += b_f[(jn*n+iw)*9 + i];
	//	}
	//
	//	r_rho[jn*n+iw] = sum_r;
	//	b_rho[jn*n+iw] = sum_b;

	r_rho[jn*n+iw] = 0;
	b_rho[jn*n+iw] = b_density;

	u[jn*n+iw]   = 0;
	v[jn*n+iw]   = 0;

	// south-east corner
	r_f[(js*n+ie) * 9 + 2] = r_f[(jn*n+ie) * 9 + 2];
	r_f[(js*n+ie) * 9 + 3] = r_f[(js*n+iw) * 9 + 3];
	r_f[(js*n+ie) * 9 + 6] = r_f[(jn*n+iw) * 9 + 6];

	b_f[(js*n+ie) * 9 + 2] = b_f[(jn*n+ie) * 9 + 2];
	b_f[(js*n+ie) * 9 + 3] = b_f[(js*n+iw) * 9 + 3];
	b_f[(js*n+ie) * 9 + 6] = b_f[(jn*n+iw) * 9 + 6];

	//	sum_r = 0.0;
	//	sum_b = 0.0;
	//	for(int i = 0; i < 9; i++){
	//		sum_r += r_f[(js*n+ie)*9 + i];
	//		sum_b += b_f[(js*n+ie)*9 + i];
	//	}
	//
	//	r_rho[js*n+ie] = sum_r;
	//	b_rho[js*n+ie] = sum_b;

	r_rho[js*n+ie] = 0;
	b_rho[js*n+ie] = b_density;

	u[js*n+ie]   = 0;
	v[js*n+ie]   = 0;


	// south-west corner
	r_f[(js*n+iw) * 9 + 2] = r_f[(jn*n+iw) * 9 + 2];
	r_f[(js*n+iw) * 9 + 1] = r_f[(js*n+ie) * 9 + 1];
	r_f[(js*n+iw) * 9 + 5] = r_f[(jn*n+ie) * 9 + 5];

	b_f[(js*n+iw) * 9 + 2] = b_f[(jn*n+iw) * 9 + 2];
	b_f[(js*n+iw) * 9 + 1] = b_f[(js*n+ie) * 9 + 1];
	b_f[(js*n+iw) * 9 + 5] = b_f[(jn*n+ie) * 9 + 5];

	//	sum_r = 0.0;
	//	sum_b = 0.0;
	//	for(int i = 0; i < 9; i++){
	//		sum_r += r_f[(js*n+iw)*9 + i];
	//		sum_b += b_f[(js*n+iw)*9 + i];
	//	}
	//
	//	r_rho[js*n+iw] = sum_r;
	//	b_rho[js*n+iw] = sum_b;

	r_rho[js*n+iw] = 0;
	b_rho[js*n+iw] = b_density;

	u[js*n+iw]   = 0;
	v[js*n+iw]  = 0;
}

void streamMP(int n, int m, FLOAT_TYPE *r_f, FLOAT_TYPE *b_f, FLOAT_TYPE *r_fColl, FLOAT_TYPE *b_fColl){
	// stream on interior first
	int index,i,j;
	for (i=1; i < n-1; i++){
		for (j=1;j < m-1;j++){
			index = i*m+j;
			r_f[index * 9] = r_fColl[index * 9];
			r_f[index * 9 + 1] = r_fColl[((i-1)*m + j) * 9 + 1];
			r_f[index * 9 + 2] = r_fColl[(i*m+j-1) * 9 + 2];
			r_f[index * 9 + 3] = r_fColl[((i+1)*m + j) * 9 + 3];
			r_f[index * 9 + 4] = r_fColl[(i*m+j+1) * 9 + 4];
			r_f[index * 9 + 5] = r_fColl[((i-1)*m + j-1) * 9 + 5];
			r_f[index * 9 + 6] = r_fColl[((i+1)*m + j-1) * 9 + 6];
			r_f[index * 9 + 7] = r_fColl[((i+1)*m + j+1) * 9 + 7];
			r_f[index * 9 + 8] = r_fColl[((i-1)*m + j+1) * 9 + 8];

			b_f[index * 9] = b_fColl[index * 9];
			b_f[index * 9 + 1] = b_fColl[((i-1)*m + j) * 9 + 1];
			b_f[index * 9 + 2] = b_fColl[(i*m+j-1) * 9 + 2];
			b_f[index * 9 + 3] = b_fColl[((i+1)*m + j) * 9 + 3];
			b_f[index * 9 + 4] = b_fColl[(i*m+j+1) * 9 + 4];
			b_f[index * 9 + 5] = b_fColl[((i-1)*m + j-1) * 9 + 5];
			b_f[index * 9 + 6] = b_fColl[((i+1)*m + j-1) * 9 + 6];
			b_f[index * 9 + 7] = b_fColl[((i+1)*m + j+1) * 9 + 7];
			b_f[index * 9 + 8] = b_fColl[((i-1)*m + j+1) * 9 + 8];
		}
	}
	for (i=1; i < n-1; i++){
		//north boundary
		j = m-1;
		index = i*m+j;

		r_f[index * 9] = r_fColl[index * 9];
		r_f[index * 9 + 1] = r_fColl[((i-1)*m + j) * 9 + 1];
		r_f[index * 9 + 2] = r_fColl[(i*m+j-1) * 9 + 2];
		r_f[index * 9 + 3] = r_fColl[((i+1)*m + j) * 9 + 3];
		r_f[index * 9 + 5] = r_fColl[((i-1)*m + j-1) * 9 + 5];
		r_f[index * 9 + 6] = r_fColl[((i+1)*m + j-1) * 9 + 6];

		b_f[index * 9] = b_fColl[index * 9];
		b_f[index * 9 + 1] = b_fColl[((i-1)*m + j) * 9 + 1];
		b_f[index * 9 + 2] = b_fColl[(i*m+j-1) * 9 + 2];
		b_f[index * 9 + 3] = b_fColl[((i+1)*m + j) * 9 + 3];
		b_f[index * 9 + 5] = b_fColl[((i-1)*m + j-1) * 9 + 5];
		b_f[index * 9 + 6] = b_fColl[((i+1)*m + j-1) * 9 + 6];

		//South boundary
		j = 0;
		index = i*m+j;

		r_f[index * 9] = r_fColl[index * 9];
		r_f[index * 9 + 1] = r_fColl[((i-1)*m + j) * 9 + 1];
		r_f[index * 9 + 3] = r_fColl[((i+1)*m + j) * 9 + 3];
		r_f[index * 9 + 4] = r_fColl[(i*m+j+1) * 9 + 4];
		r_f[index * 9 + 7] = r_fColl[((i+1)*m + j+1) * 9 + 7];
		r_f[index * 9 + 8] = r_fColl[((i-1)*m + j+1) * 9 + 8];

		b_f[index * 9] = b_fColl[index * 9];
		b_f[index * 9 + 1] = b_fColl[((i-1)*m + j) * 9 + 1];
		b_f[index * 9 + 3] = b_fColl[((i+1)*m + j) * 9 + 3];
		b_f[index * 9 + 4] = b_fColl[(i*m+j+1) * 9 + 4];
		b_f[index * 9 + 7] = b_fColl[((i+1)*m + j+1) * 9 + 7];
		b_f[index * 9 + 8] = b_fColl[((i-1)*m + j+1) * 9 + 8];
	}

	for (j=1;j < m-1;j++){
		//east
		i = n-1;
		index = i*m+j;

		r_f[index * 9] = r_fColl[index * 9];
		r_f[index * 9 + 1] = r_fColl[((i-1)*m + j) * 9 + 1];
		r_f[index * 9 + 2] = r_fColl[(i*m+j-1) * 9 + 2];
		r_f[index * 9 + 4] = r_fColl[(i*m+j+1) * 9 + 4];
		r_f[index * 9 + 5] = r_fColl[((i-1)*m + j-1) * 9 + 5];
		r_f[index * 9 + 8] = r_fColl[((i-1)*m + j+1) * 9 + 8];

		b_f[index * 9] = b_fColl[index * 9];
		b_f[index * 9 + 1] = b_fColl[((i-1)*m + j) * 9 + 1];
		b_f[index * 9 + 2] = b_fColl[(i*m+j-1) * 9 + 2];
		b_f[index * 9 + 4] = b_fColl[(i*m+j+1) * 9 + 4];
		b_f[index * 9 + 5] = b_fColl[((i-1)*m + j-1) * 9 + 5];
		b_f[index * 9 + 8] = b_fColl[((i-1)*m + j+1) * 9 + 8];

		//west
		i = 0;
		index = i*m+j;

		r_f[index * 9] = r_fColl[index * 9];
		r_f[index * 9 + 2] = r_fColl[(i*m+j-1) * 9 + 2];
		r_f[index * 9 + 3] = r_fColl[((i+1)*m + j) * 9 + 3];
		r_f[index * 9 + 4] = r_fColl[(i*m+j+1) * 9 + 4];
		r_f[index * 9 + 6] = r_fColl[((i+1)*m + j-1) * 9 + 6];
		r_f[index * 9 + 7] = r_fColl[((i+1)*m + j+1) * 9 + 7];

		b_f[index * 9] = b_fColl[index * 9];
		b_f[index * 9 + 2] = b_fColl[(i*m+j-1) * 9 + 2];
		b_f[index * 9 + 3] = b_fColl[((i+1)*m + j) * 9 + 3];
		b_f[index * 9 + 4] = b_fColl[(i*m+j+1) * 9 + 4];
		b_f[index * 9 + 6] = b_fColl[((i+1)*m + j-1) * 9 + 6];
		b_f[index * 9 + 7] = b_fColl[((i+1)*m + j+1) * 9 + 7];
	}

	// north-east corner
	i=n-1; j=m-1;
	index = i*m+j;

	r_f[index * 9] = r_fColl[index * 9];
	r_f[index * 9 + 1] = r_fColl[((i-1)*m + j) * 9 + 1];
	r_f[index * 9 + 2] = r_fColl[(i*m+j-1) * 9 + 2];
	r_f[index * 9 + 5] = r_fColl[((i-1)*m + j-1) * 9 + 5];

	b_f[index * 9] = b_fColl[index * 9];
	b_f[index * 9 + 1] = b_fColl[((i-1)*m + j) * 9 + 1];
	b_f[index * 9 + 2] = b_fColl[(i*m+j-1) * 9 + 2];
	b_f[index * 9 + 5] = b_fColl[((i-1)*m + j-1) * 9 + 5];

	//north-west corner
	i=0; j=m-1;
	index = i*m+j;

	r_f[index * 9] = r_fColl[index * 9];
	r_f[index * 9 + 2] = r_fColl[(i*m+j-1) * 9 + 2];
	r_f[index * 9 + 3] = r_fColl[((i+1)*m + j) * 9 + 3];
	r_f[index * 9 + 6] = r_fColl[((i+1)*m + j-1) * 9 + 6];

	b_f[index * 9] = b_fColl[index * 9];
	b_f[index * 9 + 2] = b_fColl[(i*m+j-1) * 9 + 2];
	b_f[index * 9 + 3] = b_fColl[((i+1)*m + j) * 9 + 3];
	b_f[index * 9 + 6] = b_fColl[((i+1)*m + j-1) * 9 + 6];

	// south-east corner
	i=n-1; j=0;
	index = i*m+j;

	r_f[index * 9] = r_fColl[index * 9];
	r_f[index * 9 + 1] = r_fColl[((i-1)*m + j) * 9 + 1];
	r_f[index * 9 + 4] = r_fColl[(i*m+j+1) * 9 + 4];
	r_f[index * 9 + 8] = r_fColl[((i-1)*m + j+1) * 9 + 8];

	b_f[index * 9] = b_fColl[index * 9];
	b_f[index * 9 + 1] = b_fColl[((i-1)*m + j) * 9 + 1];
	b_f[index * 9 + 4] = b_fColl[(i*m+j+1) * 9 + 4];
	b_f[index * 9 + 8] = b_fColl[((i-1)*m + j+1) * 9 + 8];

	// south-west corner
	i=0; j=0;
	index = i*m+j;

	r_f[index * 9] = r_fColl[index * 9];
	r_f[index * 9 + 3] = r_fColl[((i+1)*m + j) * 9 + 3];
	r_f[index * 9 + 4] = r_fColl[(i*m+j+1) * 9 + 4];
	r_f[index * 9 + 7] = r_fColl[((i+1)*m + j+1) * 9 + 7];

	b_f[index * 9] = b_fColl[index * 9];
	b_f[index * 9 + 3] = b_fColl[((i+1)*m + j) * 9 + 3];
	b_f[index * 9 + 4] = b_fColl[(i*m+j+1) * 9 + 4];
	b_f[index * 9 + 7] = b_fColl[((i+1)*m + j+1) * 9 + 7];

}

void resetArrays(FLOAT_TYPE *color_gradient, int n, int m){
	int index;
	for(int i = 0; i < n; i++){
		for(int j = 0; j < m; j++){
			index = i*m+j;
			color_gradient[index*2] = 0.0;
			color_gradient[index*2 + 1] = 0.0;
		}
	}
}

FLOAT_TYPE* convertArray(int n, int m, FLOAT_TYPE *arr){
	FLOAT_TYPE *result = createHostArrayFlt(n*m, ARRAY_NONE);

	for(int i = 0; i < n; i++){
		for(int j = 0; j < m; j++){
			result[j*n+i] = arr[i*m+j];
		}
	}

	return result;
}
