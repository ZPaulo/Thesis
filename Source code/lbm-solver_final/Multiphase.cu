#include "hip/hip_runtime.h"
#include "Multiphase.h"
#include <math.h>
#include <stdio.h>
#include "ArrayUtils.h"

void mp2DColl(int n, int m, FLOAT_TYPE *rho, FLOAT_TYPE *u,
		FLOAT_TYPE *v, FLOAT_TYPE *r_f, FLOAT_TYPE *b_f, FLOAT_TYPE *r_rho, FLOAT_TYPE *b_rho,
		FLOAT_TYPE *r_phi, FLOAT_TYPE *b_phi, FLOAT_TYPE *w_pert, FLOAT_TYPE *color_gradient,
		FLOAT_TYPE r_omega, FLOAT_TYPE b_omega, FLOAT_TYPE control_param, FLOAT_TYPE del,
		FLOAT_TYPE beta, FLOAT_TYPE g_limit,  FLOAT_TYPE r_A,  FLOAT_TYPE b_A, FLOAT_TYPE *r_fPert, FLOAT_TYPE *b_fPert,
		FLOAT_TYPE *weight, int *cx, int *cy){

	FLOAT_TYPE cu1, cu2, r_CollPert, b_CollPert;
	FLOAT_TYPE cosin;
	FLOAT_TYPE chi;
	FLOAT_TYPE r_omega_temp, b_omega_temp;
	FLOAT_TYPE a1      =   2.0 * r_omega * b_omega/(r_omega+b_omega);
	FLOAT_TYPE a2      =   2.0 * (r_omega - a1) / del;
	FLOAT_TYPE a3      =   -a2 / (2.0 * del);
	FLOAT_TYPE a4      =   2.0 * (a1 - b_omega) / del;
	FLOAT_TYPE a5      =   a4 / (2.0 * del);
	FLOAT_TYPE color_gradient_norm;
	FLOAT_TYPE k_r, k_b, k_k;
	FLOAT_TYPE norm_c;
	FLOAT_TYPE prod_c_g;
	FLOAT_TYPE r_pert, b_pert;
	FLOAT_TYPE r_feq, b_feq;
	FLOAT_TYPE fn05;
	int index, index9, temp_index;
	for (int j=0; j < m; j++){
		for (int i=0;i < n; i++){
			index = j*n + i;


			color_gradient[index * 2] = 0;
			color_gradient[index * 2 + 1] = 0;
			for (int k=0; k<9; k++){

				// calculate color gradient - 4th order

				if (i!=0 && j!=0 && i!=(n-1) && j!=(m-1)){ // Interior points - In the boundary it is calculated by "mirroring" the density
					temp_index = (j + cy[k]) * n + i + cx[k];
					color_gradient[index * 2] += (r_rho[temp_index] - b_rho[temp_index]) * cx[k];
					color_gradient[index * 2 + 1] += (r_rho[temp_index] - b_rho[temp_index]) * cy[k];
				}
				else if (j==(m-1) && i!=0 && i!=(n-1)) {// north boundary
					temp_index = (j - abs(cy[k])) * n + i + cx[k];
					color_gradient[index * 2] += (r_rho[temp_index] - b_rho[temp_index]) * cx[k];
					color_gradient[index * 2 + 1] = 0;
				}
				else if (j==0 && i!=0 && i!=(n-1)){  // south boundary
					temp_index = (j + abs(cy[k])) * n + i + cx[k];
					color_gradient[index * 2] += (r_rho[temp_index] - b_rho[temp_index]) * cx[k];
					color_gradient[index * 2 + 1] = 0;
				}
				else if (i==(n-1) && j!=0 && j!=(m-1)){  // east boundary
					temp_index = (j + cy[k]) * n + i - abs(cx[k]);
					color_gradient[index * 2] = 0;
					color_gradient[index * 2 + 1] +=  (r_rho[temp_index] - b_rho[temp_index]) * cy[k];
				}
				else if (i==0 && j!=0 && j!=(m-1)){ //  west boundary
					temp_index = (j + cy[k]) * n + i + abs(cx[k]);
					color_gradient[index * 2] = 0;
					color_gradient[index * 2 + 1] +=  (r_rho[temp_index] - b_rho[temp_index]) * cy[k];
				}
			}

			// relaxation parameter to choose a proper omega at the interface
			if (r_omega != b_omega){
				chi=(r_rho[index] - b_rho[index])/rho[index];
				if(chi >= -control_param && chi <= control_param){
					if (chi > del)
						r_omega_temp=r_omega;
					else if (chi <= del && chi > 0)
						r_omega_temp=a1 + a2 * chi + a3 * chi * chi;
					else if (chi <= 0 && chi >= -del)
						r_omega_temp=a1 + a4 * chi + a5 * chi * chi;
					else if (chi < -del)
						r_omega_temp=b_omega;
				}
			}
			else
				r_omega_temp=r_omega;

			b_omega_temp=r_omega_temp;

			cu1 = u[index]*u[index] + v[index]*v[index];

			// invariable quantities
			color_gradient_norm = sqrt(pow(color_gradient[index * 2],2) + pow(color_gradient[index * 2 + 1],2));
			k_r=r_rho[index]/rho[index];
			k_b=b_rho[index]/rho[index];
			k_k= beta * r_rho[index] * b_rho[index]/(pow(rho[index],2));
			for (int k=0;k<9;k++){
				if (color_gradient_norm > g_limit){
					prod_c_g=cx[k]*color_gradient[index * 2]+cy[k]*color_gradient[index * 2 + 1];
					if (k!=0){
						norm_c= sqrt(pow(cx[k],2)+pow(cy[k],2));
						cosin= prod_c_g / (color_gradient_norm*norm_c);
					}
					else
						cosin=0.0;
					// calculate perturbation terms

					r_pert=0.5*r_A*color_gradient_norm*(weight[k]*pow((prod_c_g/color_gradient_norm),2)-w_pert[k]);
					b_pert=0.5*b_A*color_gradient_norm*(weight[k]*pow((prod_c_g/color_gradient_norm),2)-w_pert[k]);

				}
				else{
					// ther perturbation terms are null
					r_pert=0.0;
					b_pert=0.0;
				}


				cu2 = u[index]*cx[k] + v[index]*cy[k];
				// calculate equilibrium distribution function
				r_feq = r_rho[index] * (r_phi[k] + weight[k] * (3 * cu2 + 4.5 * cu2 * cu2 - 1.5 * cu1));
				b_feq = b_rho[index] * (b_phi[k] + weight[k] * (3 * cu2 + 4.5 * cu2 * cu2 - 1.5 * cu1));

				index9 = i + j * n + k * m * n;
				// calculate updated distribution function
				r_CollPert = r_omega_temp*r_feq + (1-r_omega_temp)*r_f[index9]+r_pert;
				b_CollPert = b_omega_temp*b_feq + (1-b_omega_temp)*b_f[index9]+b_pert;

				fn05 = r_CollPert + b_CollPert;
				//				// perform recolor step
				r_fPert[index9]=k_r*fn05+k_k*cosin*(r_rho[index]*r_phi[k]+b_rho[index]*b_phi[k]);
				b_fPert[index9]=k_b*fn05-k_k*cosin*(r_rho[index]*r_phi[k]+b_rho[index]*b_phi[k]);
			}
		}
	}



}

void createBubble(FLOAT_TYPE *x, FLOAT_TYPE *y,int n, int m, FLOAT_TYPE radius, FLOAT_TYPE *r_f, FLOAT_TYPE *b_f, FLOAT_TYPE *r_rho, FLOAT_TYPE *b_rho,
		FLOAT_TYPE r_density, FLOAT_TYPE b_density, FLOAT_TYPE *r_phi, FLOAT_TYPE *b_phi, FLOAT_TYPE *rho) {
	int i, j, k;
	int index, index2;
	for (j=0; j < m; j++){
		for(i = 0; i < n; i++){
			index = j * n + i;

			if( sqrt( pow((x[index]-0.5), 2) + pow((y[index]-0.5),2)) <= radius ){
				r_rho[index] = r_density;
				for (k=0; k < 9; k++){
					// initialise distribution function with small, non-zero values
					index2 = i + j * n + k * m * n;
					r_f[index2] = r_rho[index] * r_phi[k];
				}
			}
			else {
				b_rho[index]=b_density;
				for (k=0; k < 9; k++){
					// initialise distribution function with small, non-zero values
					index2 = i + j * n + k * m * n;
					b_f[index2]   = b_rho[index]*b_phi[k];
				}
			}
			// initialise density
			rho[index] = r_rho[index]+b_rho[index];
		}
	}
}

void updateMacroMP(int n, int m, FLOAT_TYPE *u, FLOAT_TYPE *v,FLOAT_TYPE *r_rho, FLOAT_TYPE *b_rho, FLOAT_TYPE *r_f, FLOAT_TYPE *b_f, FLOAT_TYPE *rho, FLOAT_TYPE control_param,
		FLOAT_TYPE r_alpha, FLOAT_TYPE b_alpha, FLOAT_TYPE bubble_radius, FLOAT_TYPE *st_error, int iteration, FLOAT_TYPE st_predicted){

	int index_aux1=0;
	int index_aux2=0;
	FLOAT_TYPE p_in=0.0;
	FLOAT_TYPE p_out=0.0;
	FLOAT_TYPE u_cum, v_cum;
	FLOAT_TYPE r_sum, b_sum;
	FLOAT_TYPE chi;
	int index, index9;
	int cx[9] = {0, 1, 0, -1, 0, 1, -1, -1, 1};
	int cy[9] = {0, 0, 1, 0, -1, 1, 1, -1, -1};
	FLOAT_TYPE st_laplace;
	// Density and Velocity
	for (int j=1; j < m - 1;j++){
		for (int i=1; i < n - 1; i++){
			// auxiliar variables
			u_cum=0.0;
			v_cum=0.0;

			// densities
			index = j * n + i;
			r_sum = 0.0;
			b_sum = 0.0;
			for(int k = 0; k < 9; k++){
				index9 = i + j * n + k * m * n;
				r_sum += r_f[index9];
				b_sum += b_f[index9];
			}
			r_rho[index] = r_sum;
			b_rho[index]= b_sum;
			rho[index] = r_rho[index]+b_rho[index];

			// p_in and p_out for the surface tension
			chi=(r_rho[index]-b_rho[index])/rho[index];
//			printf("chi "FLOAT_FORMAT" ",chi);
//			printf("control "FLOAT_FORMAT" \n", control_param);
			if (chi >= control_param){
				index_aux1++;
				p_in += r_rho[index];
			}
			else if (chi < control_param){
				index_aux2++;
				p_out+=b_rho[index];
			}

			// velocities
			for (int k=0; k < 9; k++){
				index9 = i + j * n + k * m * n;
				u_cum += (r_f[index9]+b_f[index9])*cx[k];
				v_cum += (r_f[index9]+b_f[index9])*cy[k];
			}
			u[index]   = u_cum/rho[index];
			v[index]  = v_cum/rho[index];

		}
	}

	// Calculate surface tension
	//printf("%f vs %f\n", p_in, p_out);
	p_in=(3.0/5.0)*(1.0-r_alpha)*p_in/index_aux1;      // pressure average inside the bubble
	p_out=(3.0/5.0)*(1.0-b_alpha)*p_out/index_aux2;   // pressure average outside the bubble
	st_laplace=bubble_radius*(p_in-p_out);

	st_error[iteration]=abs(st_predicted-st_laplace)/(st_predicted)*100.0;
}

void peridicBoundaries(int n, int m, FLOAT_TYPE *r_f, FLOAT_TYPE *b_f, FLOAT_TYPE *r_rho, FLOAT_TYPE *b_rho,
		FLOAT_TYPE b_density, FLOAT_TYPE *u, FLOAT_TYPE *v){

	int index_end, index_start;
	int jn = m-1;
	int js = 0;
	int ie = n-1;
	int iw = 0;

	for (int i=1; i < n-1; i++){
		// north boundary
		index_end = jn * n + i;
		index_start = js * n + i;

		r_f[index_end + 4 * m * n] = r_f[index_start + 4 * m * n];
		r_f[index_end + 7 * m * n] = r_f[index_start + 7 * m * n];
		r_f[index_end + 8 * m * n] = r_f[index_start + 8 * m * n];

		b_f[index_end + 4 * m * n] = b_f[index_start + 4 * m * n];
		b_f[index_end + 7 * m * n] = b_f[index_start + 7 * m * n];
		b_f[index_end + 8 * m * n] = b_f[index_start + 8 * m * n];

		// macroscopic boundary conditions
//		r_rho[index_end] = 0;
//		b_rho[index_end] = b_density;
//		u[index_end]   = 0;
//		v[index_end]   = 0;

		//south boundary
		r_f[index_start + 2 * m * n] = r_f[index_end + 2 * m * n];
		r_f[index_start + 5 * m * n] = r_f[index_end + 5 * m * n];
		r_f[index_start + 6 * m * n] = r_f[index_end + 6 * m * n];

		b_f[index_start + 2 * m * n] = b_f[index_end + 2 * m * n];
		b_f[index_start + 5 * m * n] = b_f[index_end + 5 * m * n];
		b_f[index_start + 6 * m * n] = b_f[index_end + 6 * m * n];

//		r_rho[index_start] = 0;
//		b_rho[index_start] = b_density;
//		u[index_start]   = 0;
//		v[index_start]   = 0;
	}



	for (int j=1; j < m-1; j++){
		// east boundary
		index_end = j*n + ie;
		index_start = j*n + iw;

		r_f[index_end + 3 * m * n] = r_f[index_start + 3 * m * n];
		r_f[index_end + 7 * m * n] = r_f[index_start + 7 * m * n];
		r_f[index_end + 6 * m * n] = r_f[index_start + 6 * m * n];

		b_f[index_end + 3 * m * n] = b_f[index_start + 3 * m * n];
		b_f[index_end + 7 * m * n] = b_f[index_start + 7 * m * n];
		b_f[index_end + 6 * m * n] = b_f[index_start + 6 * m * n];

		//macroscopic boundary conditions
//		r_rho[index_end] = 0;
//		b_rho[index_end] = b_density;
//		u[index_end]   = 0;
//		v[index_end]   = 0;

		// west boundary
		r_f[index_start + 1 * m * n] = r_f[index_end + 1 * m * n];
		r_f[index_start + 5 * m * n] = r_f[index_end + 5 * m * n];
		r_f[index_start + 8 * m * n] = r_f[index_end + 8 * m * n];

		b_f[index_start + 1 * m * n] = b_f[index_end + 1 * m * n];
		b_f[index_start + 5 * m * n] = b_f[index_end + 5 * m * n];
		b_f[index_start + 8 * m * n] = b_f[index_end + 8 * m * n];

//		r_rho[index_start] = 0;
//		b_rho[index_start] = b_density;
//		u[index_start]   = 0;
//		v[index_start]   = 0;

	}

	// north-east corner
	r_f[(jn*n+ie) + 3 * m * n] = r_f[(jn*n+iw) + 3 * m * n];
	r_f[(jn*n+ie) + 4 * m * n] = r_f[(js*n+ie) + 4 * m * n];
	r_f[(jn*n+ie) + 7 * m * n] = r_f[(js*n+iw) + 7 * m * n];

	b_f[(jn*n+ie) + 3 * m * n] = b_f[(jn*n+iw) + 3 * m * n];
	b_f[(jn*n+ie) + 4 * m * n] = b_f[(js*n+ie) + 4 * m * n];
	b_f[(jn*n+ie) + 7 * m * n] = b_f[(js*n+iw) + 7 * m * n];

	//	FLOAT_TYPE sum_r = 0.0;
	//	FLOAT_TYPE sum_b = 0.0;
	//	for(int i = 0; i < 9; i++){
	//		sum_r += r_f[(jn*n+ie)*9 + i];
	//		sum_b += b_f[(jn*n+ie)*9 + i];
	//	}
	//
	//	r_rho[jn*n+ie] = sum_r;
	//	b_rho[jn*n+ie] = sum_b;

//	r_rho[jn*n+ie] = 0;
//	b_rho[jn*n+ie] = b_density;
//
//	u[jn*n+ie]   = 0;
//	v[jn*n+ie]   = 0;

	// north-west corner
	r_f[(jn*n+iw) + 1 * m * n] = r_f[(jn*n+ie) + 1 * m * n];
	r_f[(jn*n+iw) + 4 * m * n] = r_f[(js*n+iw) + 4 * m * n];
	r_f[(jn*n+iw) + 8 * m * n] = r_f[(js*n+ie) + 8 * m * n];

	b_f[(jn*n+iw) + 1 * m * n] = b_f[(jn*n+ie) + 1 * m * n];
	b_f[(jn*n+iw) + 4 * m * n] = b_f[(js*n+iw) + 4 * m * n];
	b_f[(jn*n+iw) + 8 * m * n] = b_f[(js*n+ie) + 8 * m * n];

	//	sum_r = 0.0;
	//	sum_b = 0.0;
	//	for(int i = 0; i < 9; i++){
	//		sum_r += r_f[(jn*n+iw)*9 + i];
	//		sum_b += b_f[(jn*n+iw)*9 + i];
	//	}
	//
	//	r_rho[jn*n+iw] = sum_r;
	//	b_rho[jn*n+iw] = sum_b;

//	r_rho[jn*n+iw] = 0;
//	b_rho[jn*n+iw] = b_density;
//
//	u[jn*n+iw]   = 0;
//	v[jn*n+iw]   = 0;

	// south-east corner
	r_f[(js*n+ie) + 2 * m * n] = r_f[(jn*n+ie) + 2 * m * n];
	r_f[(js*n+ie) + 3 * m * n] = r_f[(js*n+iw) + 3 * m * n];
	r_f[(js*n+ie) + 6 * m * n] = r_f[(jn*n+iw) + 6 * m * n];

	b_f[(js*n+ie) + 2 * m * n] = b_f[(jn*n+ie) + 2 * m * n];
	b_f[(js*n+ie) + 3 * m * n] = b_f[(js*n+iw) + 3 * m * n];
	b_f[(js*n+ie) + 6 * m * n] = b_f[(jn*n+iw) + 6 * m * n];

	//	sum_r = 0.0;
	//	sum_b = 0.0;
	//	for(int i = 0; i < 9; i++){
	//		sum_r += r_f[(js*n+ie)*9 + i];
	//		sum_b += b_f[(js*n+ie)*9 + i];
	//	}
	//
	//	r_rho[js*n+ie] = sum_r;
	//	b_rho[js*n+ie] = sum_b;

//	r_rho[js*n+ie] = 0;
//	b_rho[js*n+ie] = b_density;
//
//	u[js*n+ie]   = 0;
//	v[js*n+ie]   = 0;


	// south-west corner
	r_f[(js*n+iw) + 2 * m * n] = r_f[(jn*n+iw) + 2 * m * n];
	r_f[(js*n+iw) + 1 * m * n] = r_f[(js*n+ie) + 1 * m * n];
	r_f[(js*n+iw) + 5 * m * n] = r_f[(jn*n+ie) + 5 * m * n];

	b_f[(js*n+iw) + 2 * m * n] = b_f[(jn*n+iw) + 2 * m * n];
	b_f[(js*n+iw) + 1 * m * n] = b_f[(js*n+ie) + 1 * m * n];
	b_f[(js*n+iw) + 5 * m * n] = b_f[(jn*n+ie) + 5 * m * n];

	//	sum_r = 0.0;
	//	sum_b = 0.0;
	//	for(int i = 0; i < 9; i++){
	//		sum_r += r_f[(js*n+iw)*9 + i];
	//		sum_b += b_f[(js*n+iw)*9 + i];
	//	}
	//
	//	r_rho[js*n+iw] = sum_r;
	//	b_rho[js*n+iw] = sum_b;

//	r_rho[js*n+iw] = 0;
//	b_rho[js*n+iw] = b_density;
//
//	u[js*n+iw]   = 0;
//	v[js*n+iw]  = 0;
}

void streamMP(int n, int m, FLOAT_TYPE *r_f, FLOAT_TYPE *b_f, FLOAT_TYPE *r_fColl, FLOAT_TYPE *b_fColl){
	// stream on interior first
	int index,i,j;
	for (j=1;j < m-1;j++){
		for (i=1; i < n-1; i++){
			index = j*n+i;
			r_f[index] = r_fColl[index];
			r_f[index + 1 * m * n] = r_fColl[(j*n+i-1) + 1 * m * n];
			r_f[index + 2 * m * n] = r_fColl[((j-1) * n + i) + 2 * m * n];
			r_f[index + 3 * m * n] = r_fColl[(j*n + i + 1) + 3 * m * n];
			r_f[index + 4 * m * n] = r_fColl[((j+1) * n + i) + 4 * m * n];
			r_f[index + 5 * m * n] = r_fColl[((j-1) * n + i - 1) + 5 * m * n];
			r_f[index + 6 * m * n] = r_fColl[((j-1) * n + i + 1) + 6 * m * n];
			r_f[index + 7 * m * n] = r_fColl[((j+1) * n + i + 1) + 7 * m * n];
			r_f[index + 8 * m * n] = r_fColl[((j+1) * n + i - 1) + 8 * m * n];

			b_f[index] = b_fColl[index];
			b_f[index + 1 * m * n] = b_fColl[(j*n+i-1) + 1 * m * n];
			b_f[index + 2 * m * n] = b_fColl[((j-1) * n + i) + 2 * m * n];
			b_f[index + 3 * m * n] = b_fColl[(j*n + i + 1) + 3 * m * n];
			b_f[index + 4 * m * n] = b_fColl[((j+1) * n + i) + 4 * m * n];
			b_f[index + 5 * m * n] = b_fColl[((j-1) * n + i - 1) + 5 * m * n];
			b_f[index + 6 * m * n] = b_fColl[((j-1) * n + i + 1) + 6 * m * n];
			b_f[index + 7 * m * n] = b_fColl[((j+1) * n + i + 1) + 7 * m * n];
			b_f[index + 8 * m * n] = b_fColl[((j+1) * n + i - 1) + 8 * m * n];
		}
	}
	for (i=1; i < n-1; i++){
		//north boundary
		j = m-1;
		index = j*n+i;

		r_f[index] = r_fColl[index];
		r_f[index + 1 * m * n] = r_fColl[(j*n+i-1) + 1 * m * n];
		r_f[index + 2 * m * n] = r_fColl[((j-1) * n + i) + 2 * m * n];
		r_f[index + 3 * m * n] = r_fColl[(j*n + i + 1) + 3 * m * n];
		r_f[index + 5 * m * n] = r_fColl[((j-1) * n + i - 1) + 5 * m * n];
		r_f[index + 6 * m * n] = r_fColl[((j-1) * n + i + 1) + 6 * m * n];

		b_f[index] = b_fColl[index];
		b_f[index + 1 * m * n] = b_fColl[(j*n+i-1) + 1 * m * n];
		b_f[index + 2 * m * n] = b_fColl[((j-1) * n + i) + 2 * m * n];
		b_f[index + 3 * m * n] = b_fColl[(j*n + i + 1) + 3 * m * n];
		b_f[index + 5 * m * n] = b_fColl[((j-1) * n + i - 1) + 5 * m * n];
		b_f[index + 6 * m * n] = b_fColl[((j-1) * n + i + 1) + 6 * m * n];

		//South boundary
		j = 0;
		index = j*n+i;

		r_f[index] = r_fColl[index];
		r_f[index + 1 * m * n] = r_fColl[(j*n+i-1) + 1 * m * n];
		r_f[index + 3 * m * n] = r_fColl[(j*n + i + 1) + 3 * m * n];
		r_f[index + 4 * m * n] = r_fColl[((j+1) * n + i) + 4 * m * n];
		r_f[index + 7 * m * n] = r_fColl[((j+1) * n + i + 1) + 7 * m * n];
		r_f[index + 8 * m * n] = r_fColl[((j+1) * n + i - 1) + 8 * m * n];

		b_f[index] = b_fColl[index];
		b_f[index + 1 * m * n] = b_fColl[(j*n+i-1) + 1 * m * n];
		b_f[index + 3 * m * n] = b_fColl[(j*n + i + 1) + 3 * m * n];
		b_f[index + 4 * m * n] = b_fColl[((j+1) * n + i) + 4 * m * n];
		b_f[index + 7 * m * n] = b_fColl[((j+1) * n + i + 1) + 7 * m * n];
		b_f[index + 8 * m * n] = b_fColl[((j+1) * n + i - 1) + 8 * m * n];
	}

	for (j=1;j < m-1;j++){
		//east
		i = n-1;
		index = j*n+i;

		r_f[index] = r_fColl[index];
		r_f[index + 1 * m * n] = r_fColl[(j*n+i-1) + 1 * m * n];
		r_f[index + 2 * m * n] = r_fColl[((j-1) * n + i) + 2 * m * n];
		r_f[index + 4 * m * n] = r_fColl[((j+1) * n + i) + 4 * m * n];
		r_f[index + 5 * m * n] = r_fColl[((j-1) * n + i - 1) + 5 * m * n];
		r_f[index + 8 * m * n] = r_fColl[((j+1) * n + i - 1) + 8 * m * n];

		b_f[index] = b_fColl[index];
		b_f[index + 1 * m * n] = b_fColl[(j*n+i-1) + 1 * m * n];
		b_f[index + 2 * m * n] = b_fColl[((j-1) * n + i) + 2 * m * n];
		b_f[index + 4 * m * n] = b_fColl[((j+1) * n + i) + 4 * m * n];
		b_f[index + 5 * m * n] = b_fColl[((j-1) * n + i - 1) + 5 * m * n];
		b_f[index + 8 * m * n] = b_fColl[((j+1) * n + i - 1) + 8 * m * n];

		//west
		i = 0;
		index = j*n+i;

		r_f[index] = r_fColl[index];
		r_f[index + 2 * m * n] = r_fColl[((j-1) * n + i) + 2 * m * n];
		r_f[index + 3 * m * n] = r_fColl[(j*n + i + 1) + 3 * m * n];
		r_f[index + 4 * m * n] = r_fColl[((j+1) * n + i) + 4 * m * n];
		r_f[index + 6 * m * n] = r_fColl[((j-1) * n + i + 1) + 6 * m * n];
		r_f[index + 7 * m * n] = r_fColl[((j+1) * n + i + 1) + 7 * m * n];

		b_f[index] = b_fColl[index];
		b_f[index + 2 * m * n] = b_fColl[((j-1) * n + i) + 2 * m * n];
		b_f[index + 3 * m * n] = b_fColl[(j*n + i + 1) + 3 * m * n];
		b_f[index + 4 * m * n] = b_fColl[((j+1) * n + i) + 4 * m * n];
		b_f[index + 6 * m * n] = b_fColl[((j-1) * n + i + 1) + 6 * m * n];
		b_f[index + 7 * m * n] = b_fColl[((j+1) * n + i + 1) + 7 * m * n];
	}

	// north-east corner
	i=n-1; j=m-1;
	index = j*n+i;

	r_f[index] = r_fColl[index];
	r_f[index + 1 * m * n] = r_fColl[(j*n+i-1) + 1 * m * n];
	r_f[index + 2 * m * n] = r_fColl[((j-1) * n + i) + 2 * m * n];
	r_f[index + 5 * m * n] = r_fColl[((j-1) * n + i - 1) + 5 * m * n];

	b_f[index] = b_fColl[index];
	b_f[index + 1 * m * n] = b_fColl[(j*n+i-1) + 1 * m * n];
	b_f[index + 2 * m * n] = b_fColl[((j-1) * n + i) + 2 * m * n];
	b_f[index + 5 * m * n] = b_fColl[((j-1) * n + i - 1) + 5 * m * n];

	//north-west corner
	i=0; j=m-1;
	index = j*n+i;

	r_f[index] = r_fColl[index];
	r_f[index + 2 * m * n] = r_fColl[((j-1) * n + i) + 2 * m * n];
	r_f[index + 3 * m * n] = r_fColl[(j*n + i + 1) + 3 * m * n];
	r_f[index + 6 * m * n] = r_fColl[((j-1) * n + i + 1) + 6 * m * n];

	b_f[index] = b_fColl[index];
	b_f[index + 2 * m * n] = b_fColl[((j-1) * n + i) + 2 * m * n];
	b_f[index + 3 * m * n] = b_fColl[(j*n + i + 1) + 3 * m * n];
	b_f[index + 6 * m * n] = b_fColl[((j-1) * n + i + 1) + 6 * m * n];

	// south-east corner
	i=n-1; j=0;
	index = j*n+i;

	r_f[index] = r_fColl[index];
	r_f[index + 1 * m * n] = r_fColl[(j*n+i-1) + 1 * m * n];
	r_f[index + 4 * m * n] = r_fColl[((j+1) * n + i) + 4 * m * n];
	r_f[index + 8 * m * n] = r_fColl[((j+1) * n + i - 1) + 8 * m * n];

	b_f[index] = b_fColl[index];
	b_f[index + 1 * m * n] = b_fColl[(j*n+i-1) + 1 * m * n];
	b_f[index + 4 * m * n] = b_fColl[((j+1) * n + i) + 4 * m * n];
	b_f[index + 8 * m * n] = b_fColl[((j+1) * n + i - 1) + 8 * m * n];

	// south-west corner
	i=0; j=0;
	index = j*n+i;

	r_f[index] = r_fColl[index];
	r_f[index + 3 * m * n] = r_fColl[(j*n + i + 1) + 3 * m * n];
	r_f[index + 4 * m * n] = r_fColl[((j+1) * n + i) + 4 * m * n];
	r_f[index + 7 * m * n] = r_fColl[((j+1) * n + i + 1) + 7 * m * n];

	b_f[index] = b_fColl[index];
	b_f[index + 3 * m * n] = b_fColl[(j*n + i + 1) + 3 * m * n];
	b_f[index + 4 * m * n] = b_fColl[((j+1) * n + i) + 4 * m * n];
	b_f[index + 7 * m * n] = b_fColl[((j+1) * n + i + 1) + 7 * m * n];

}

void resetArrays(FLOAT_TYPE *color_gradient, int n, int m){
	for(int i = 0; i < m * n *2; i++){
		color_gradient[i] = 0.0;
	}
}

FLOAT_TYPE* convertArray(int n, int m, FLOAT_TYPE *arr){
	FLOAT_TYPE *result = createHostArrayFlt(n*m, ARRAY_NONE);

	for(int i = 0; i < n; i++){
		for(int j = 0; j < m; j++){
			result[j*n+i] = arr[i*m+j];
		}
	}

	return result;
}
